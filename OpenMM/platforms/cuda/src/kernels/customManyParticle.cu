#include "hip/hip_runtime.h"
/**
 * Record the force on an atom to global memory.
 */
inline __device__ void storeForce(int atom, real3 force, unsigned long long* __restrict__ forceBuffers) {
    atomicAdd(&forceBuffers[atom], static_cast<unsigned long long>((long long) (force.x*0x100000000)));
    atomicAdd(&forceBuffers[atom+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (force.y*0x100000000)));
    atomicAdd(&forceBuffers[atom+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (force.z*0x100000000)));
}

/**
 * Convert a real4 to a real3 by removing its last element.
 */
inline __device__ real3 trim(real4 v) {
    return make_real3(v.x, v.y, v.z);
}

/**
 * Compute the difference between two vectors, taking periodic boundary conditions into account
 * and setting the fourth component to the squared magnitude.
 */
inline __device__ real4 delta(real3 vec1, real3 vec2, real4 periodicBoxSize, real4 invPeriodicBoxSize, real4 periodicBoxVecX, real4 periodicBoxVecY, real4 periodicBoxVecZ) {
    real4 result = make_real4(vec1.x-vec2.x, vec1.y-vec2.y, vec1.z-vec2.z, 0.0f);
#ifdef USE_PERIODIC
    APPLY_PERIODIC_TO_DELTA(result)
#endif
    result.w = result.x*result.x + result.y*result.y + result.z*result.z;
    return result;
}

/**
 * Compute the angle between two vectors.  The w component of each vector should contain the squared magnitude.
 */
__device__ real computeAngle(real4 vec1, real4 vec2) {
    real dotProduct = vec1.x*vec2.x + vec1.y*vec2.y + vec1.z*vec2.z;
    real cosine = dotProduct*RSQRT(vec1.w*vec2.w);
    real angle;
    if (cosine > 0.99f || cosine < -0.99f) {
        // We're close to the singularity in acos(), so take the cross product and use asin() instead.

        real3 crossProduct = cross(vec1, vec2);
        real scale = vec1.w*vec2.w;
        angle = ASIN(SQRT(dot(crossProduct, crossProduct)/scale));
        if (cosine < 0.0f)
            angle = M_PI-angle;
    }
    else
       angle = ACOS(cosine);
    return angle;
}

/**
 * Compute the cross product of two vectors, setting the fourth component to the squared magnitude.
 */
inline __device__ real4 computeCross(real4 vec1, real4 vec2) {
    real3 cp = cross(vec1, vec2);
    return make_real4(cp.x, cp.y, cp.z, cp.x*cp.x+cp.y*cp.y+cp.z*cp.z);
}

/**
 * Determine whether a particular interaction is in the list of exclusions.
 */
inline __device__ bool isInteractionExcluded(int atom1, int atom2, const int* __restrict__ exclusions, const int* __restrict__ exclusionStartIndex) {
    int first = exclusionStartIndex[atom1];
    int last = exclusionStartIndex[atom1+1];
    for (int i = last-1; i >= first; i--) {
        int excluded = exclusions[i];
        if (excluded == atom2)
            return true;
        if (excluded <= atom1)
            return false;
    }
    return false;
}

__constant__ float globals[NUM_GLOBALS];

/**
 * Compute the interaction.
 */
extern "C" __global__ void computeInteraction(
        unsigned long long* __restrict__ forceBuffers, mixed* __restrict__ energyBuffer, const real4* __restrict__ posq,
        real4 periodicBoxSize, real4 invPeriodicBoxSize, real4 periodicBoxVecX, real4 periodicBoxVecY, real4 periodicBoxVecZ
#ifdef USE_CUTOFF
        , const int* __restrict__ neighbors, const int* __restrict__ neighborStartIndex
#endif
#ifdef USE_FILTERS
        , int* __restrict__ particleTypes, int* __restrict__ orderIndex, int* __restrict__ particleOrder
#endif
#ifdef USE_EXCLUSIONS
        , int* __restrict__ exclusions, int* __restrict__ exclusionStartIndex
#endif
        PARAMETER_ARGUMENTS) {
    mixed energy = 0;
    
    // Loop over particles to be the first one in the set.
    
    for (int p1 = blockIdx.x; p1 < NUM_ATOMS; p1 += gridDim.x) {
#ifdef USE_CENTRAL_PARTICLE
        const int a1 = p1;
#else
        const int a1 = 0;
#endif
#ifdef USE_CUTOFF
        int firstNeighbor = neighborStartIndex[p1];
        int numNeighbors = neighborStartIndex[p1+1]-firstNeighbor;
#else
  #ifdef USE_CENTRAL_PARTICLE
        int numNeighbors = NUM_ATOMS;
  #else
        int numNeighbors = NUM_ATOMS-p1-1;
  #endif
#endif
        int numCombinations = NUM_CANDIDATE_COMBINATIONS;
        for (int index = threadIdx.x; index < numCombinations; index += blockDim.x) {
            FIND_ATOMS_FOR_COMBINATION_INDEX;
            bool includeInteraction = IS_VALID_COMBINATION;
#ifdef USE_CUTOFF
            if (includeInteraction) {
                VERIFY_CUTOFF;
            }
#endif
#ifdef USE_FILTERS
            int order = orderIndex[COMPUTE_TYPE_INDEX];
            if (order == -1)
                includeInteraction = false;
#endif
#ifdef USE_EXCLUSIONS
            if (includeInteraction) {
                VERIFY_EXCLUSIONS;
            }
#endif
            if (includeInteraction) {
                PERMUTE_ATOMS;
                LOAD_PARTICLE_DATA;
                COMPUTE_INTERACTION;
            }
        }
    }
    energyBuffer[blockIdx.x*blockDim.x+threadIdx.x] += energy;
}

/**
 * Find a bounding box for the atoms in each block.
 */
extern "C" __global__ void findBlockBounds(real4 periodicBoxSize, real4 invPeriodicBoxSize, real4 periodicBoxVecX, real4 periodicBoxVecY, real4 periodicBoxVecZ,
        const real4* __restrict__ posq, real4* __restrict__ blockCenter, real4* __restrict__ blockBoundingBox, int* __restrict__ numNeighborPairs) {
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    int base = index*TILE_SIZE;
    while (base < NUM_ATOMS) {
        real4 pos = posq[base];
#ifdef USE_PERIODIC
        APPLY_PERIODIC_TO_POS(pos)
#endif
        real4 minPos = pos;
        real4 maxPos = pos;
        int last = min(base+TILE_SIZE, NUM_ATOMS);
        for (int i = base+1; i < last; i++) {
            pos = posq[i];
#ifdef USE_PERIODIC
            real4 center = 0.5f*(maxPos+minPos);
            APPLY_PERIODIC_TO_POS_WITH_CENTER(pos, center)
#endif
            minPos = make_real4(min(minPos.x,pos.x), min(minPos.y,pos.y), min(minPos.z,pos.z), 0);
            maxPos = make_real4(max(maxPos.x,pos.x), max(maxPos.y,pos.y), max(maxPos.z,pos.z), 0);
        }
        real4 blockSize = 0.5f*(maxPos-minPos);
        blockBoundingBox[index] = blockSize;
        blockCenter[index] = 0.5f*(maxPos+minPos);
        index += blockDim.x*gridDim.x;
        base = index*TILE_SIZE;
    }
    if (blockIdx.x == 0 && threadIdx.x == 0)
        *numNeighborPairs = 0;
}

/**
 * Find a list of neighbors for each atom.
 */
extern "C" __global__ void findNeighbors(real4 periodicBoxSize, real4 invPeriodicBoxSize, real4 periodicBoxVecX, real4 periodicBoxVecY, real4 periodicBoxVecZ,
        const real4* __restrict__ posq, const real4* __restrict__ blockCenter, const real4* __restrict__ blockBoundingBox, int2* __restrict__ neighborPairs,
        int* __restrict__ numNeighborPairs, int* __restrict__ numNeighborsForAtom, int maxNeighborPairs
#ifdef USE_EXCLUSIONS
        , const int* __restrict__ exclusions, const int* __restrict__ exclusionStartIndex
#endif
        ) {
    __shared__ real3 positionCache[FIND_NEIGHBORS_WORKGROUP_SIZE];
    int indexInWarp = threadIdx.x%32;
    for (int atom1 = blockIdx.x*blockDim.x+threadIdx.x; atom1 < PADDED_NUM_ATOMS; atom1 += blockDim.x*gridDim.x) {
        // Load data for this atom.  Note that all threads in a warp are processing atoms from the same block.
        
        real3 pos1 = trim(posq[atom1]);
        int block1 = atom1/TILE_SIZE;
        real4 blockCenter1 = blockCenter[block1];
        real4 blockSize1 = blockBoundingBox[block1];
        int totalNeighborsForAtom1 = 0;
        
        // Loop over atom blocks to search for neighbors.  The threads in a warp compare block1 against 32
        // other blocks in parallel.

#ifdef USE_CENTRAL_PARTICLE
        int startBlock = 0;
#else
        int startBlock = block1;
#endif
        for (int block2Base = startBlock; block2Base < NUM_BLOCKS; block2Base += 32) {
            int block2 = block2Base+indexInWarp;
            bool includeBlock2 = (block2 < NUM_BLOCKS);
            if (includeBlock2) {
                real4 blockCenter2 = blockCenter[block2];
                real4 blockSize2 = blockBoundingBox[block2];
                real4 blockDelta = blockCenter1-blockCenter2;
#ifdef USE_PERIODIC
                APPLY_PERIODIC_TO_DELTA(blockDelta)
#endif
                blockDelta.x = max(0.0f, fabs(blockDelta.x)-blockSize1.x-blockSize2.x);
                blockDelta.y = max(0.0f, fabs(blockDelta.y)-blockSize1.y-blockSize2.y);
                blockDelta.z = max(0.0f, fabs(blockDelta.z)-blockSize1.z-blockSize2.z);
                includeBlock2 &= (blockDelta.x*blockDelta.x+blockDelta.y*blockDelta.y+blockDelta.z*blockDelta.z < CUTOFF_SQUARED);
            }
            
            // Loop over any blocks we identified as potentially containing neighbors.
            
            int includeBlockFlags = __ballot(includeBlock2);
            while (includeBlockFlags != 0) {
                int i = __ffs(includeBlockFlags)-1;
                includeBlockFlags &= includeBlockFlags-1;
                int block2 = block2Base+i;

                // Loop over atoms in this block.

                int start = block2*TILE_SIZE;
                int included[TILE_SIZE];
                int numIncluded = 0;
                positionCache[threadIdx.x] = trim(posq[start+indexInWarp]);
                if (atom1 < NUM_ATOMS) {
                    for (int j = 0; j < 32; j++) {
                        int atom2 = start+j;
                        real3 pos2 = positionCache[threadIdx.x-indexInWarp+j];

                        // Decide whether to include this atom pair in the neighbor list.

                        real4 atomDelta = delta(pos1, pos2, periodicBoxSize, invPeriodicBoxSize, periodicBoxVecX, periodicBoxVecY, periodicBoxVecZ);
#ifdef USE_CENTRAL_PARTICLE
                        bool includeAtom = (atom2 != atom1 && atom2 < NUM_ATOMS && atomDelta.w < CUTOFF_SQUARED);
#else
                        bool includeAtom = (atom2 > atom1 && atom2 < NUM_ATOMS && atomDelta.w < CUTOFF_SQUARED);
#endif
#ifdef USE_EXCLUSIONS
                        if (includeAtom)
                            includeAtom &= !isInteractionExcluded(atom1, atom2, exclusions, exclusionStartIndex);
#endif
                        if (includeAtom)
                            included[numIncluded++] = atom2;
                    }
                }

                // If we found any neighbors, store them to the neighbor list.

                if (numIncluded > 0) {
                    int baseIndex = atomicAdd(numNeighborPairs, numIncluded);
                    if (baseIndex+numIncluded <= maxNeighborPairs)
                        for (int j = 0; j < numIncluded; j++)
                            neighborPairs[baseIndex+j] = make_int2(atom1, included[j]);
                    totalNeighborsForAtom1 += numIncluded;
                }
            }
        }
        if (atom1 < NUM_ATOMS)
            numNeighborsForAtom[atom1] = totalNeighborsForAtom1;
    }
}

/**
 * Sum the neighbor counts to compute the start position of each atom.  This kernel
 * is executed as a single work group.
 */
extern "C" __global__ void computeNeighborStartIndices(int* __restrict__ numNeighborsForAtom, int* __restrict__ neighborStartIndex,
            int* __restrict__ numNeighborPairs, int maxNeighborPairs) {
    extern __shared__ unsigned int posBuffer[];
    if (*numNeighborPairs > maxNeighborPairs) {
        // There wasn't enough memory for the neighbor list, so we'll need to rebuild it.  Set the neighbor start
        // indices to indicate no neighbors for any atom.
        
        for (int i = threadIdx.x; i <= NUM_ATOMS; i += blockDim.x)
            neighborStartIndex[i] = 0;
        return;
    }
    unsigned int globalOffset = 0;
    for (unsigned int startAtom = 0; startAtom < NUM_ATOMS; startAtom += blockDim.x) {
        // Load the neighbor counts into local memory.

        unsigned int globalIndex = startAtom+threadIdx.x;
        posBuffer[threadIdx.x] = (globalIndex < NUM_ATOMS ? numNeighborsForAtom[globalIndex] : 0);
        __syncthreads();

        // Perform a parallel prefix sum.

        for (unsigned int step = 1; step < blockDim.x; step *= 2) {
            unsigned int add = (threadIdx.x >= step ? posBuffer[threadIdx.x-step] : 0);
            __syncthreads();
            posBuffer[threadIdx.x] += add;
            __syncthreads();
        }

        // Write the results back to global memory.

        if (globalIndex < NUM_ATOMS) {
            neighborStartIndex[globalIndex+1] = posBuffer[threadIdx.x]+globalOffset;
            numNeighborsForAtom[globalIndex] = 0; // Clear this so the next kernel can use it as a counter
        }
        globalOffset += posBuffer[blockDim.x-1];
        __syncthreads();
    }
    if (threadIdx.x == 0)
        neighborStartIndex[0] = 0;
}

/**
 * Assemble the final neighbor list.
 */
extern "C" __global__ void copyPairsToNeighborList(const int2* __restrict__ neighborPairs, int* __restrict__ neighbors, int* __restrict__ numNeighborPairs,
            int maxNeighborPairs, int* __restrict__ numNeighborsForAtom, const int* __restrict__ neighborStartIndex) {
    int actualPairs = *numNeighborPairs;
    if (actualPairs > maxNeighborPairs)
        return; // There wasn't enough memory for the neighbor list, so we'll need to rebuild it.
    for (unsigned int index = blockDim.x*blockIdx.x+threadIdx.x; index < actualPairs; index += blockDim.x*gridDim.x) {
        int2 pair = neighborPairs[index];
        int startIndex = neighborStartIndex[pair.x];
        int offset = atomicAdd(numNeighborsForAtom+pair.x, 1);
        neighbors[startIndex+offset] = pair.y;
    }
}
