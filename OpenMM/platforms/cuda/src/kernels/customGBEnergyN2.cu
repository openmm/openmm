#include "hip/hip_runtime.h"
#define STORE_DERIVATIVE_1(INDEX) atomicAdd(&derivBuffers[offset+(INDEX-1)*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (deriv##INDEX##_1*0x100000000)));
#define STORE_DERIVATIVE_2(INDEX) atomicAdd(&derivBuffers[offset+(INDEX-1)*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].deriv##INDEX*0x100000000)));

typedef struct {
    real3 pos;
    real3 force;
    ATOM_PARAMETER_DATA
#ifdef NEED_PADDING
    float padding;
#endif
} AtomData;

/**
 * Compute a force based on pair interactions.
 */
extern "C" __global__ void computeN2Energy(unsigned long long* __restrict__ forceBuffers, mixed* __restrict__ energyBuffer,
        const real4* __restrict__ posq, const unsigned int* __restrict__ exclusions, const ushort2* __restrict__ exclusionTiles, bool needEnergy,
#ifdef USE_CUTOFF
        const int* __restrict__ tiles, const unsigned int* __restrict__ interactionCount, real4 periodicBoxSize, real4 invPeriodicBoxSize,
        real4 periodicBoxVecX, real4 periodicBoxVecY, real4 periodicBoxVecZ, unsigned int maxTiles, const real4* __restrict__ blockCenter,
        const real4* __restrict__ blockSize, const unsigned int* __restrict__ interactingAtoms
#else
        unsigned int numTiles
#endif
        PARAMETER_ARGUMENTS) {
    const unsigned int totalWarps = (blockDim.x*gridDim.x)/TILE_SIZE;
    const unsigned int warp = (blockIdx.x*blockDim.x+threadIdx.x)/TILE_SIZE;
    const unsigned int tgx = threadIdx.x & (TILE_SIZE-1);
    const unsigned int tbx = threadIdx.x - tgx;
    mixed energy = 0;
    INIT_PARAM_DERIVS
    __shared__ AtomData localData[THREAD_BLOCK_SIZE];

    // First loop: process tiles that contain exclusions.
    
    const unsigned int firstExclusionTile = FIRST_EXCLUSION_TILE+warp*(LAST_EXCLUSION_TILE-FIRST_EXCLUSION_TILE)/totalWarps;
    const unsigned int lastExclusionTile = FIRST_EXCLUSION_TILE+(warp+1)*(LAST_EXCLUSION_TILE-FIRST_EXCLUSION_TILE)/totalWarps;
    for (int pos = firstExclusionTile; pos < lastExclusionTile; pos++) {
        const ushort2 tileIndices = exclusionTiles[pos];
        const unsigned int x = tileIndices.x;
        const unsigned int y = tileIndices.y;
        real3 force = make_real3(0);
        DECLARE_ATOM1_DERIVATIVES
        unsigned int atom1 = x*TILE_SIZE + tgx;
        real4 pos1 = posq[atom1];
        LOAD_ATOM1_PARAMETERS
#ifdef USE_EXCLUSIONS
        unsigned int excl = exclusions[pos*TILE_SIZE+tgx];
#endif
        if (x == y) {
            // This tile is on the diagonal.

            const unsigned int localAtomIndex = threadIdx.x;
            localData[localAtomIndex].pos = make_real3(pos1.x, pos1.y, pos1.z);
            LOAD_LOCAL_PARAMETERS_FROM_1
            for (unsigned int j = 0; j < TILE_SIZE; j++) {
                int atom2 = tbx+j;
                real3 pos2 = localData[atom2].pos;
                real3 delta = make_real3(pos2.x-pos1.x, pos2.y-pos1.y, pos2.z-pos1.z);
#ifdef USE_PERIODIC
                APPLY_PERIODIC_TO_DELTA(delta)
#endif
                real r2 = delta.x*delta.x + delta.y*delta.y + delta.z*delta.z;
#ifdef USE_CUTOFF
                if (r2 < CUTOFF_SQUARED) {
#endif
                    real invR = RSQRT(r2);
                    real r = r2*invR;
                    LOAD_ATOM2_PARAMETERS
                    atom2 = y*TILE_SIZE+j;
                    real dEdR = 0;
                    real tempEnergy = 0;
                    const real interactionScale = 0.5f;
#ifdef USE_EXCLUSIONS
                    bool isExcluded = !(excl & 0x1);
#endif
                    if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS && atom1 != atom2) {
                        COMPUTE_INTERACTION
                        dEdR /= -r;
                    }
                    if (needEnergy)
                        energy += 0.5f*tempEnergy;
                    delta *= dEdR;
                    force.x -= delta.x;
                    force.y -= delta.y;
                    force.z -= delta.z;
#ifdef USE_CUTOFF
                }
#endif
#ifdef USE_EXCLUSIONS
                excl >>= 1;
#endif
            }
        }
        else {
            // This is an off-diagonal tile.

            const unsigned int localAtomIndex = threadIdx.x;
            unsigned int j = y*TILE_SIZE + tgx;
            real4 tempPosq = posq[j];
            localData[localAtomIndex].pos = make_real3(tempPosq.x, tempPosq.y, tempPosq.z);
            LOAD_LOCAL_PARAMETERS_FROM_GLOBAL
            localData[localAtomIndex].force = make_real3(0);
            CLEAR_LOCAL_DERIVATIVES
#ifdef USE_EXCLUSIONS
            excl = (excl >> tgx) | (excl << (TILE_SIZE - tgx));
#endif
            unsigned int tj = tgx;
            for (j = 0; j < TILE_SIZE; j++) {
                int atom2 = tbx+tj;
                real3 pos2 = localData[atom2].pos;
                real3 delta = make_real3(pos2.x-pos1.x, pos2.y-pos1.y, pos2.z-pos1.z);
#ifdef USE_PERIODIC
                APPLY_PERIODIC_TO_DELTA(delta)
#endif
                real r2 = delta.x*delta.x + delta.y*delta.y + delta.z*delta.z;
#ifdef USE_CUTOFF
                if (r2 < CUTOFF_SQUARED) {
#endif
                    real invR = RSQRT(r2);
                    real r = r2*invR;
                    LOAD_ATOM2_PARAMETERS
                    atom2 = y*TILE_SIZE+tj;
                    real dEdR = 0;
                    real tempEnergy = 0;
                    const real interactionScale = 1;
#ifdef USE_EXCLUSIONS
                    bool isExcluded = !(excl & 0x1);
#endif
                    if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS) {
                        COMPUTE_INTERACTION
                        dEdR /= -r;
                    }
                    if (needEnergy)
                        energy += tempEnergy;
                    delta *= dEdR;
                    force.x -= delta.x;
                    force.y -= delta.y;
                    force.z -= delta.z;
                    atom2 = tbx+tj;
                    localData[atom2].force.x += delta.x;
                    localData[atom2].force.y += delta.y;
                    localData[atom2].force.z += delta.z;
                    RECORD_DERIVATIVE_2
#ifdef USE_CUTOFF
                }
#endif
#ifdef USE_EXCLUSIONS
                excl >>= 1;
#endif
                tj = (tj + 1) & (TILE_SIZE - 1);
            }
        }

        // Write results.

        unsigned int offset = x*TILE_SIZE + tgx;
        atomicAdd(&forceBuffers[offset], static_cast<unsigned long long>((long long) (force.x*0x100000000)));
        atomicAdd(&forceBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (force.y*0x100000000)));
        atomicAdd(&forceBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (force.z*0x100000000)));
        STORE_DERIVATIVES_1
        if (x != y) {
            offset = y*TILE_SIZE + tgx;
            atomicAdd(&forceBuffers[offset], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.x*0x100000000)));
            atomicAdd(&forceBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.y*0x100000000)));
            atomicAdd(&forceBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.z*0x100000000)));
            STORE_DERIVATIVES_2
        }
    }

    // Second loop: tiles without exclusions, either from the neighbor list (with cutoff) or just enumerating all
    // of them (no cutoff).

#ifdef USE_CUTOFF
    unsigned int numTiles = interactionCount[0];
    if (numTiles > maxTiles)
        return; // There wasn't enough memory for the neighbor list.
    int pos = (int) (warp*(numTiles > maxTiles ? NUM_BLOCKS*((long long)NUM_BLOCKS+1)/2 : (long)numTiles)/totalWarps);
    int end = (int) ((warp+1)*(numTiles > maxTiles ? NUM_BLOCKS*((long long)NUM_BLOCKS+1)/2 : (long)numTiles)/totalWarps);
#else
    int pos = (int) (warp*(long long)numTiles/totalWarps);
    int end = (int) ((warp+1)*(long long)numTiles/totalWarps);
#endif
    int skipBase = 0;
    int currentSkipIndex = tbx;
    __shared__ int atomIndices[THREAD_BLOCK_SIZE];
    __shared__ volatile int skipTiles[THREAD_BLOCK_SIZE];
    skipTiles[threadIdx.x] = -1;
    
    while (pos < end) {
        const bool isExcluded = false;
        real3 force = make_real3(0);
        DECLARE_ATOM1_DERIVATIVES
        bool includeTile = true;
        
        // Extract the coordinates of this tile.
        
        int x, y;
        bool singlePeriodicCopy = false;
#ifdef USE_CUTOFF
            x = tiles[pos];
            real4 blockSizeX = blockSize[x];
            singlePeriodicCopy = (0.5f*periodicBoxSize.x-blockSizeX.x >= CUTOFF &&
                                  0.5f*periodicBoxSize.y-blockSizeX.y >= CUTOFF &&
                                  0.5f*periodicBoxSize.z-blockSizeX.z >= CUTOFF);
#else
        y = (int) floor(NUM_BLOCKS+0.5f-SQRT((NUM_BLOCKS+0.5f)*(NUM_BLOCKS+0.5f)-2*pos));
        x = (pos-y*NUM_BLOCKS+y*(y+1)/2);
        if (x < y || x >= NUM_BLOCKS) { // Occasionally happens due to roundoff error.
            y += (x < y ? -1 : 1);
            x = (pos-y*NUM_BLOCKS+y*(y+1)/2);
        }

        // Skip over tiles that have exclusions, since they were already processed.

        while (skipTiles[tbx+TILE_SIZE-1] < pos) {
            if (skipBase+tgx < NUM_TILES_WITH_EXCLUSIONS) {
                ushort2 tile = exclusionTiles[skipBase+tgx];
                skipTiles[threadIdx.x] = tile.x + tile.y*NUM_BLOCKS - tile.y*(tile.y+1)/2;
            }
            else
                skipTiles[threadIdx.x] = end;
            skipBase += TILE_SIZE;            
            currentSkipIndex = tbx;
        }
        while (skipTiles[currentSkipIndex] < pos)
            currentSkipIndex++;
        includeTile = (skipTiles[currentSkipIndex] != pos);
#endif
        if (includeTile) {
            unsigned int atom1 = x*TILE_SIZE + tgx;

            // Load atom data for this tile.

            real4 pos1 = posq[atom1];
            LOAD_ATOM1_PARAMETERS
            const unsigned int localAtomIndex = threadIdx.x;
#ifdef USE_CUTOFF
            unsigned int j = interactingAtoms[pos*TILE_SIZE+tgx];
#else
            unsigned int j = y*TILE_SIZE + tgx;
#endif
            atomIndices[threadIdx.x] = j;
            if (j < PADDED_NUM_ATOMS) {
                real4 tempPosq = posq[j];
                localData[localAtomIndex].pos = make_real3(tempPosq.x, tempPosq.y, tempPosq.z);
                LOAD_LOCAL_PARAMETERS_FROM_GLOBAL
                localData[localAtomIndex].force = make_real3(0);
                CLEAR_LOCAL_DERIVATIVES
            }
#ifdef USE_PERIODIC
            if (singlePeriodicCopy) {
                // The box is small enough that we can just translate all the atoms into a single periodic
                // box, then skip having to apply periodic boundary conditions later.

                real4 blockCenterX = blockCenter[x];
                APPLY_PERIODIC_TO_POS_WITH_CENTER(pos1, blockCenterX)
                APPLY_PERIODIC_TO_POS_WITH_CENTER(localData[threadIdx.x].pos, blockCenterX)
                unsigned int tj = tgx;
                for (j = 0; j < TILE_SIZE; j++) {
                    int atom2 = tbx+tj;
                    real3 pos2 = localData[atom2].pos;
                    real3 delta = make_real3(pos2.x-pos1.x, pos2.y-pos1.y, pos2.z-pos1.z);
                    real r2 = delta.x*delta.x + delta.y*delta.y + delta.z*delta.z;
#ifdef USE_CUTOFF
                    if (r2 < CUTOFF_SQUARED) {
#endif
                        real invR = RSQRT(r2);
                        real r = r2*invR;
                        LOAD_ATOM2_PARAMETERS
                        atom2 = atomIndices[tbx+tj];
                        real dEdR = 0;
                        real tempEnergy = 0;
                        const real interactionScale = 1;
                        if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS) {
                            COMPUTE_INTERACTION
                            dEdR /= -r;
                        }
                        if (needEnergy)
                            energy += tempEnergy;
                        delta *= dEdR;
                        force.x -= delta.x;
                        force.y -= delta.y;
                        force.z -= delta.z;
                        atom2 = tbx+tj;
                        localData[atom2].force.x += delta.x;
                        localData[atom2].force.y += delta.y;
                        localData[atom2].force.z += delta.z;
                        RECORD_DERIVATIVE_2
#ifdef USE_CUTOFF
                    }
#endif
                    tj = (tj + 1) & (TILE_SIZE - 1);
                }
            }
            else
#endif
            {
                // We need to apply periodic boundary conditions separately for each interaction.

                unsigned int tj = tgx;
                for (j = 0; j < TILE_SIZE; j++) {
                    int atom2 = tbx+tj;
                    real3 pos2 = localData[atom2].pos;
                    real3 delta = make_real3(pos2.x-pos1.x, pos2.y-pos1.y, pos2.z-pos1.z);
#ifdef USE_PERIODIC
                    APPLY_PERIODIC_TO_DELTA(delta)
#endif
                    real r2 = delta.x*delta.x + delta.y*delta.y + delta.z*delta.z;
#ifdef USE_CUTOFF
                    if (r2 < CUTOFF_SQUARED) {
#endif
                        real invR = RSQRT(r2);
                        real r = r2*invR;
                        LOAD_ATOM2_PARAMETERS
                        atom2 = atomIndices[tbx+tj];
                        real dEdR = 0;
                        real tempEnergy = 0;
                        const real interactionScale = 1;
                        if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS) {
                            COMPUTE_INTERACTION
                            dEdR /= -r;
                        }
                        if (needEnergy)
                            energy += tempEnergy;
                        delta *= dEdR;
                        force.x -= delta.x;
                        force.y -= delta.y;
                        force.z -= delta.z;
                        atom2 = tbx+tj;
                        localData[atom2].force.x += delta.x;
                        localData[atom2].force.y += delta.y;
                        localData[atom2].force.z += delta.z;
                        RECORD_DERIVATIVE_2
#ifdef USE_CUTOFF
                    }
#endif
                    tj = (tj + 1) & (TILE_SIZE - 1);
                }
            }
        
            // Write results.

            atomicAdd(&forceBuffers[atom1], static_cast<unsigned long long>((long long) (force.x*0x100000000)));
            atomicAdd(&forceBuffers[atom1+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (force.y*0x100000000)));
            atomicAdd(&forceBuffers[atom1+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (force.z*0x100000000)));
            unsigned int offset = atom1;
            STORE_DERIVATIVES_1
#ifdef USE_CUTOFF
            unsigned int atom2 = atomIndices[threadIdx.x];
#else
            unsigned int atom2 = y*TILE_SIZE + tgx;
#endif
            if (atom2 < PADDED_NUM_ATOMS) {
                atomicAdd(&forceBuffers[atom2], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.x*0x100000000)));
                atomicAdd(&forceBuffers[atom2+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.y*0x100000000)));
                atomicAdd(&forceBuffers[atom2+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.z*0x100000000)));
                offset = atom2;
                STORE_DERIVATIVES_2
            }
        }
        pos++;
    }
    energyBuffer[blockIdx.x*blockDim.x+threadIdx.x] += energy;
    SAVE_PARAM_DERIVS
}
