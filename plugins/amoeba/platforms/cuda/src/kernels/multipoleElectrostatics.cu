#include "hip/hip_runtime.h"
#define WARPS_PER_GROUP (THREAD_BLOCK_SIZE/TILE_SIZE)

typedef struct {
    real4 posq;
    real3 force, dipole, inducedDipole, inducedDipolePolar;
    real quadrupoleXX, quadrupoleXY, quadrupoleXZ;
    real quadrupoleYY, quadrupoleYZ;
    float thole, damp;
} AtomData;

__device__ void computeOneInteractionF1(AtomData& atom1, volatile AtomData& atom2, float dScale, float pScale, float mScale, real& energy, real3& outputForce);
__device__ void computeOneInteractionT1(AtomData& atom1, volatile AtomData& atom2, float dScale, float pScale, float mScale, real3& outputForce);
__device__ void computeOneInteractionT3(AtomData& atom1, volatile AtomData& atom2, float dScale, float pScale, float mScale, real3& outputForce);

inline __device__ void loadAtomData(AtomData& data, int atom, const real4* __restrict__ posq, const real* __restrict__ labFrameDipole,
        const real* __restrict__ labFrameQuadrupole, const real* __restrict__ inducedDipole, const real* __restrict__ inducedDipolePolar, const float2* __restrict__ dampingAndThole) {
    data.posq = posq[atom];
    data.dipole.x = labFrameDipole[atom*3];
    data.dipole.y = labFrameDipole[atom*3+1];
    data.dipole.z = labFrameDipole[atom*3+2];
    data.quadrupoleXX = labFrameQuadrupole[atom*5];
    data.quadrupoleXY = labFrameQuadrupole[atom*5+1];
    data.quadrupoleXZ = labFrameQuadrupole[atom*5+2];
    data.quadrupoleYY = labFrameQuadrupole[atom*5+3];
    data.quadrupoleYZ = labFrameQuadrupole[atom*5+4];
    data.inducedDipole.x = inducedDipole[atom*3];
    data.inducedDipole.y = inducedDipole[atom*3+1];
    data.inducedDipole.z = inducedDipole[atom*3+2];
    data.inducedDipolePolar.x = inducedDipolePolar[atom*3];
    data.inducedDipolePolar.y = inducedDipolePolar[atom*3+1];
    data.inducedDipolePolar.z = inducedDipolePolar[atom*3+2];
    float2 temp = dampingAndThole[atom];
    data.damp = temp.x;
    data.thole = temp.y;
}

__device__ real computeDScaleFactor(unsigned int polarizationGroup, int index) {
    return (polarizationGroup & 1<<index ? 0 : 1);
}

__device__ float computeMScaleFactor(uint2 covalent, int index) {
    int mask = 1<<index;
    bool x = (covalent.x & mask);
    bool y = (covalent.y & mask);
    return (x ? (y ? 0.0f : 0.4f) : (y ? 0.8f : 1.0f));
}

__device__ float computePScaleFactor(uint2 covalent, unsigned int polarizationGroup, int index) {
    int mask = 1<<index;
    bool x = (covalent.x & mask);
    bool y = (covalent.y & mask);
    bool p = (polarizationGroup & mask);
    return (x && y ? 0.0f : (x && p ? 0.5f : 1.0f));
}

/**
 * Compute electrostatic interactions.
 */
extern "C" __global__ void computeElectrostatics(
        unsigned long long* __restrict__ forceBuffers, unsigned long long* __restrict__ torqueBuffers, real* __restrict__ energyBuffer,
        const real4* __restrict__ posq, const uint2* __restrict__ covalentFlags, const unsigned int* __restrict__ polarizationGroupFlags,
        const ushort2* __restrict__ exclusionTiles, unsigned int startTileIndex, unsigned int numTileIndices,
#ifdef USE_CUTOFF
        const int* __restrict__ tiles, const unsigned int* __restrict__ interactionCount, real4 periodicBoxSize, real4 invPeriodicBoxSize, unsigned int maxTiles, const real4* __restrict__ blockCenter, const unsigned int* __restrict__ interactingAtoms,
#endif
        const real* __restrict__ labFrameDipole, const real* __restrict__ labFrameQuadrupole, const real* __restrict__ inducedDipole,
        const real* __restrict__ inducedDipolePolar, const float2* __restrict__ dampingAndThole) {
    const unsigned int totalWarps = (blockDim.x*gridDim.x)/TILE_SIZE;
    const unsigned int warp = (blockIdx.x*blockDim.x+threadIdx.x)/TILE_SIZE;
    const unsigned int tgx = threadIdx.x & (TILE_SIZE-1);
    const unsigned int tbx = threadIdx.x - tgx;
    real energy = 0;
    __shared__ AtomData localData[THREAD_BLOCK_SIZE];
    

    // First loop: process tiles that contain exclusions.
    
    const unsigned int firstExclusionTile = FIRST_EXCLUSION_TILE+warp*(LAST_EXCLUSION_TILE-FIRST_EXCLUSION_TILE)/totalWarps;
    const unsigned int lastExclusionTile = FIRST_EXCLUSION_TILE+(warp+1)*(LAST_EXCLUSION_TILE-FIRST_EXCLUSION_TILE)/totalWarps;
    for (int pos = firstExclusionTile; pos < lastExclusionTile; pos++) {
        const ushort2 tileIndices = exclusionTiles[pos];
        const unsigned int x = tileIndices.x;
        const unsigned int y = tileIndices.y;
        AtomData data;
        unsigned int atom1 = x*TILE_SIZE + tgx;
        loadAtomData(data, atom1, posq, labFrameDipole, labFrameQuadrupole, inducedDipole, inducedDipolePolar, dampingAndThole);
        data.force = make_real3(0);
        uint2 covalent = covalentFlags[pos*TILE_SIZE+tgx];
        unsigned int polarizationGroup = polarizationGroupFlags[pos*TILE_SIZE+tgx];
        if (x == y) {
            // This tile is on the diagonal.

            localData[threadIdx.x].posq = data.posq;
            localData[threadIdx.x].dipole = data.dipole;
            localData[threadIdx.x].quadrupoleXX = data.quadrupoleXX;
            localData[threadIdx.x].quadrupoleXY = data.quadrupoleXY;
            localData[threadIdx.x].quadrupoleXZ = data.quadrupoleXZ;
            localData[threadIdx.x].quadrupoleYY = data.quadrupoleYY;
            localData[threadIdx.x].quadrupoleYZ = data.quadrupoleYZ;
            localData[threadIdx.x].inducedDipole = data.inducedDipole;
            localData[threadIdx.x].inducedDipolePolar = data.inducedDipolePolar;
            localData[threadIdx.x].thole = data.thole;
            localData[threadIdx.x].damp = data.damp;

            // Compute forces.

            for (unsigned int j = 0; j < TILE_SIZE; j++) {
                int atom2 = y*TILE_SIZE+j;
                if (atom1 != atom2 && atom1 < NUM_ATOMS && atom2 < NUM_ATOMS) {
                    real3 tempForce;
                    real tempEnergy;
                    float d = computeDScaleFactor(polarizationGroup, j);
                    float p = computePScaleFactor(covalent, polarizationGroup, j);
                    float m = computeMScaleFactor(covalent, j);
                    computeOneInteractionF1(data, localData[tbx+j], d, p, m, tempEnergy, tempForce);
                    data.force += tempForce;
                    energy += 0.5f*tempEnergy;
                }
            }
            data.force *= ENERGY_SCALE_FACTOR;
            atomicAdd(&forceBuffers[atom1], static_cast<unsigned long long>((long long) (data.force.x*0x100000000)));
            atomicAdd(&forceBuffers[atom1+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.y*0x100000000)));
            atomicAdd(&forceBuffers[atom1+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.z*0x100000000)));

            // Compute torques.

            data.force = make_real3(0);
            for (unsigned int j = 0; j < TILE_SIZE; j++) {
                int atom2 = y*TILE_SIZE+j;
                if (atom1 != atom2 && atom1 < NUM_ATOMS && atom2 < NUM_ATOMS) {
                    real3 tempForce;
                    float d = computeDScaleFactor(polarizationGroup, j);
                    float p = computePScaleFactor(covalent, polarizationGroup, j);
                    float m = computeMScaleFactor(covalent, j);
                    computeOneInteractionT1(data, localData[tbx+j], d, p, m, tempForce);
                    data.force += tempForce;
                }
            }
            data.force *= ENERGY_SCALE_FACTOR;
            atomicAdd(&torqueBuffers[atom1], static_cast<unsigned long long>((long long) (data.force.x*0x100000000)));
            atomicAdd(&torqueBuffers[atom1+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.y*0x100000000)));
            atomicAdd(&torqueBuffers[atom1+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.z*0x100000000)));
        }
        else {
            // This is an off-diagonal tile.

            unsigned int j = y*TILE_SIZE + tgx;
            loadAtomData(localData[threadIdx.x], j, posq, labFrameDipole, labFrameQuadrupole, inducedDipole, inducedDipolePolar, dampingAndThole);
            localData[threadIdx.x].force = make_real3(0);
            unsigned int tj = tgx;
            for (j = 0; j < TILE_SIZE; j++) {
                int atom2 = y*TILE_SIZE+tj;
                if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS) {
                    real3 tempForce;
                    real tempEnergy;
                    float d = computeDScaleFactor(polarizationGroup, tj);
                    float p = computePScaleFactor(covalent, polarizationGroup, tj);
                    float m = computeMScaleFactor(covalent, tj);
                    computeOneInteractionF1(data, localData[tbx+tj], d, p, m, tempEnergy, tempForce);
                    data.force += tempForce;
                    localData[tbx+tj].force -= tempForce;
                    energy += tempEnergy;
                }
                tj = (tj + 1) & (TILE_SIZE - 1);
            }
            data.force *= ENERGY_SCALE_FACTOR;
            localData[threadIdx.x].force *= ENERGY_SCALE_FACTOR;
            unsigned int offset = x*TILE_SIZE + tgx;
            atomicAdd(&forceBuffers[offset], static_cast<unsigned long long>((long long) (data.force.x*0x100000000)));
            atomicAdd(&forceBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.y*0x100000000)));
            atomicAdd(&forceBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.z*0x100000000)));
            offset = y*TILE_SIZE + tgx;
            atomicAdd(&forceBuffers[offset], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.x*0x100000000)));
            atomicAdd(&forceBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.y*0x100000000)));
            atomicAdd(&forceBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.z*0x100000000)));

            // Compute torques.

            data.force = make_real3(0);
            localData[threadIdx.x].force = make_real3(0);
            for (j = 0; j < TILE_SIZE; j++) {
                int atom2 = y*TILE_SIZE+tj;
                if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS) {
                    real3 tempForce;
                    float d = computeDScaleFactor(polarizationGroup, tj);
                    float p = computePScaleFactor(covalent, polarizationGroup, tj);
                    float m = computeMScaleFactor(covalent, tj);
                    computeOneInteractionT1(data, localData[tbx+tj], d, p, m, tempForce);
                    data.force += tempForce;
                    computeOneInteractionT3(data, localData[tbx+tj], d, p, m, tempForce);
                    localData[tbx+tj].force += tempForce;
                }
                tj = (tj + 1) & (TILE_SIZE - 1);
            }
            data.force *= ENERGY_SCALE_FACTOR;
            localData[threadIdx.x].force *= ENERGY_SCALE_FACTOR;
            offset = x*TILE_SIZE + tgx;
            atomicAdd(&torqueBuffers[offset], static_cast<unsigned long long>((long long) (data.force.x*0x100000000)));
            atomicAdd(&torqueBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.y*0x100000000)));
            atomicAdd(&torqueBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.z*0x100000000)));
            offset = y*TILE_SIZE + tgx;
            atomicAdd(&torqueBuffers[offset], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.x*0x100000000)));
            atomicAdd(&torqueBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.y*0x100000000)));
            atomicAdd(&torqueBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.z*0x100000000)));
        }
    }

    // Second loop: tiles without exclusions, either from the neighbor list (with cutoff) or just enumerating all
    // of them (no cutoff).

#ifdef USE_CUTOFF
    const unsigned int numTiles = interactionCount[0];
    int pos = (numTiles > maxTiles ? startTileIndex+warp*numTileIndices/totalWarps : warp*numTiles/totalWarps);
    int end = (numTiles > maxTiles ? startTileIndex+(warp+1)*numTileIndices/totalWarps : (warp+1)*numTiles/totalWarps);
#else
    const unsigned int numTiles = numTileIndices;
    int pos = startTileIndex+warp*numTiles/totalWarps;
    int end = startTileIndex+(warp+1)*numTiles/totalWarps;
#endif
    int skipBase = 0;
    int currentSkipIndex = tbx;
    __shared__ int atomIndices[THREAD_BLOCK_SIZE];
    __shared__ volatile int skipTiles[THREAD_BLOCK_SIZE];
    skipTiles[threadIdx.x] = -1;
    
    while (pos < end) {
        bool includeTile = true;

        // Extract the coordinates of this tile.
        
        unsigned int x, y;
#ifdef USE_CUTOFF
        if (numTiles <= maxTiles)
            x = tiles[pos];
        else
#endif
        {
            y = (unsigned int) floor(NUM_BLOCKS+0.5f-SQRT((NUM_BLOCKS+0.5f)*(NUM_BLOCKS+0.5f)-2*pos));
            x = (pos-y*NUM_BLOCKS+y*(y+1)/2);
            if (x < y || x >= NUM_BLOCKS) { // Occasionally happens due to roundoff error.
                y += (x < y ? -1 : 1);
                x = (pos-y*NUM_BLOCKS+y*(y+1)/2);
            }

            // Skip over tiles that have exclusions, since they were already processed.

            while (skipTiles[tbx+TILE_SIZE-1] < pos) {
                if (skipBase+tgx < NUM_TILES_WITH_EXCLUSIONS) {
                    ushort2 tile = exclusionTiles[skipBase+tgx];
                    skipTiles[threadIdx.x] = tile.x + tile.y*NUM_BLOCKS - tile.y*(tile.y+1)/2;
                }
                else
                    skipTiles[threadIdx.x] = end;
                skipBase += TILE_SIZE;            
                currentSkipIndex = tbx;
            }
            while (skipTiles[currentSkipIndex] < pos)
                currentSkipIndex++;
            includeTile = (skipTiles[currentSkipIndex] != pos);
        }
        if (includeTile) {
            unsigned int atom1 = x*TILE_SIZE + tgx;

            // Load atom data for this tile.

            AtomData data;
            loadAtomData(data, atom1, posq, labFrameDipole, labFrameQuadrupole, inducedDipole, inducedDipolePolar, dampingAndThole);
            data.force = make_real3(0);
#ifdef USE_CUTOFF
            unsigned int j = (numTiles <= maxTiles ? interactingAtoms[pos*TILE_SIZE+tgx] : y*TILE_SIZE + tgx);
#else
            unsigned int j = y*TILE_SIZE + tgx;
#endif
            atomIndices[threadIdx.x] = j;
            loadAtomData(localData[threadIdx.x], j, posq, labFrameDipole, labFrameQuadrupole, inducedDipole, inducedDipolePolar, dampingAndThole);
            localData[threadIdx.x].force = make_real3(0);

            // Compute forces.

            unsigned int tj = tgx;
            for (j = 0; j < TILE_SIZE; j++) {
                int atom2 = atomIndices[tbx+tj];
                if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS) {
                    real3 tempForce;
                    real tempEnergy;
                    computeOneInteractionF1(data, localData[tbx+tj], 1, 1, 1, tempEnergy, tempForce);
                    data.force += tempForce;
                    localData[tbx+tj].force -= tempForce;
                    energy += tempEnergy;
                }
                tj = (tj + 1) & (TILE_SIZE - 1);
            }
            data.force *= ENERGY_SCALE_FACTOR;
            localData[threadIdx.x].force *= ENERGY_SCALE_FACTOR;
            unsigned int offset = x*TILE_SIZE + tgx;
            atomicAdd(&forceBuffers[offset], static_cast<unsigned long long>((long long) (data.force.x*0x100000000)));
            atomicAdd(&forceBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.y*0x100000000)));
            atomicAdd(&forceBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.z*0x100000000)));
#ifdef USE_CUTOFF
            offset = atomIndices[threadIdx.x];
#else
            offset = y*TILE_SIZE + tgx;
#endif
            atomicAdd(&forceBuffers[offset], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.x*0x100000000)));
            atomicAdd(&forceBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.y*0x100000000)));
            atomicAdd(&forceBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.z*0x100000000)));

            // Compute torques.

            data.force = make_real3(0);
            localData[threadIdx.x].force = make_real3(0);
            for (j = 0; j < TILE_SIZE; j++) {
                int atom2 = y*TILE_SIZE+tj;
                if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS) {
                    real3 tempForce;
                    computeOneInteractionT1(data, localData[tbx+tj], 1, 1, 1, tempForce);
                    data.force += tempForce;
                    computeOneInteractionT3(data, localData[tbx+tj], 1, 1, 1, tempForce);
                    localData[tbx+tj].force += tempForce;
                }
                tj = (tj + 1) & (TILE_SIZE - 1);
            }
            data.force *= ENERGY_SCALE_FACTOR;
            localData[threadIdx.x].force *= ENERGY_SCALE_FACTOR;
            offset = x*TILE_SIZE + tgx;
            atomicAdd(&torqueBuffers[offset], static_cast<unsigned long long>((long long) (data.force.x*0x100000000)));
            atomicAdd(&torqueBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.y*0x100000000)));
            atomicAdd(&torqueBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.z*0x100000000)));
#ifdef USE_CUTOFF
            offset = atomIndices[threadIdx.x];
#else
            offset = y*TILE_SIZE + tgx;
#endif
            atomicAdd(&torqueBuffers[offset], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.x*0x100000000)));
            atomicAdd(&torqueBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.y*0x100000000)));
            atomicAdd(&torqueBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.z*0x100000000)));
        }
        pos++;
    }
    energyBuffer[blockIdx.x*blockDim.x+threadIdx.x] += energy*ENERGY_SCALE_FACTOR;
}
