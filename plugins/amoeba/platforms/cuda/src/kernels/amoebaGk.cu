#include "hip/hip_runtime.h"
#define TILE_SIZE 32

/**
 * Reduce the Born sums to compute the Born radii.
 */
extern "C" __global__ void reduceBornSum(const long long* __restrict__ bornSum, const float2* __restrict__ params, real* __restrict__ bornRadii) {
    for (unsigned int index = blockIdx.x*blockDim.x+threadIdx.x; index < NUM_ATOMS; index += blockDim.x*gridDim.x) {
        // Get summed Born data

        real sum = RECIP(0x100000000)*bornSum[index];

        // Now calculate Born radius.

        float radius = params[index].x;
        radius = RECIP(radius*radius*radius);
        sum = radius-sum;
        sum = (sum <= 0 ? (real) 1000 : POW(sum, -1/(real) 3));
        bornRadii[index] = sum;
    }
}

#ifdef SURFACE_AREA_FACTOR
/**
 * Apply the surface area term to the force and energy.
 */
extern "C" __global__ void computeSurfaceAreaForce(long long* __restrict__ bornForce, mixed* __restrict__ energyBuffer, const float2* __restrict__ params, const real* __restrict__ bornRadii) {
    mixed energy = 0;
    for (unsigned int index = blockIdx.x*blockDim.x+threadIdx.x; index < NUM_ATOMS; index += blockDim.x*gridDim.x) {
        real bornRadius = bornRadii[index];
        float radius = params[index].x;
        real r = radius + DIELECTRIC_OFFSET + PROBE_RADIUS;
        real ratio6 = (radius+DIELECTRIC_OFFSET)/bornRadius;
        ratio6 = ratio6*ratio6*ratio6;
        ratio6 = ratio6*ratio6;
        real saTerm = SURFACE_AREA_FACTOR * r * r * ratio6;
        bornForce[index] += (long long) (saTerm*0x100000000/bornRadius);
        energy += saTerm;
    }
    energyBuffer[blockIdx.x*blockDim.x+threadIdx.x] -= energy/6;
}
#endif

/**
 * Data structure used by computeBornSum().
 */
typedef struct {
    real3 pos;
    real bornSum;
    float radius, scaledRadius, padding;
} AtomData1;

__device__ void computeBornSumOneInteraction(AtomData1& atom1, AtomData1& atom2) {
    if (atom1.radius <= 0)
        return; // Ignore this interaction
    real3 delta = atom2.pos - atom1.pos;
    real r2 = dot(delta, delta);
    real r = SQRT(r2);
    float sk = atom2.scaledRadius;

    if (atom1.radius > r + sk)
        return; // No descreening due to atom1 engulfing atom2.

    real sk2 = sk*sk;
    if (atom1.radius+r < sk) {
        real lik = atom1.radius;
        real uik = sk - r; 
        atom1.bornSum -= RECIP(uik*uik*uik) - RECIP(lik*lik*lik);
    }
    real uik = r+sk;
    real lik;
    if (atom1.radius+r < sk)
        lik = sk-r;
    else if (r < atom1.radius+sk)
        lik = atom1.radius;
    else
        lik = r-sk;
    real l2 = lik*lik; 
    real l4 = l2*l2;
    real lr = lik*r;
    real l4r = l4*r; 
    real u2 = uik*uik;
    real u4 = u2*u2;
    real ur = uik*r; 
    real u4r = u4*r;
    real term = (3*(r2-sk2)+6*u2-8*ur)/u4r - (3*(r2-sk2)+6*l2-8*lr)/l4r;
    atom1.bornSum += term/16;
}

/**
 * Compute the Born sum.
 */
extern "C" __global__ void computeBornSum(unsigned long long* __restrict__ bornSum, const real4* __restrict__ posq,
        const float2* __restrict__ params, unsigned int numTiles) {
    unsigned int totalWarps = (blockDim.x*gridDim.x)/TILE_SIZE;
    unsigned int warp = (blockIdx.x*blockDim.x+threadIdx.x)/TILE_SIZE;
    unsigned int pos = (unsigned int) (warp*(long long)numTiles/totalWarps);
    unsigned int end = (unsigned int) ((warp+1)*(long long)numTiles/totalWarps);
    unsigned int lasty = 0xFFFFFFFF;
    __shared__ AtomData1 localData[BORN_SUM_THREAD_BLOCK_SIZE];
    do {
        // Extract the coordinates of this tile
        const unsigned int tgx = threadIdx.x & (TILE_SIZE-1);
        const unsigned int tbx = threadIdx.x - tgx;
        int x, y;
        AtomData1 data;
        data.bornSum = 0;
        if (pos < end) {
            y = (int) floor(NUM_BLOCKS+0.5f-sqrt((NUM_BLOCKS+0.5f)*(NUM_BLOCKS+0.5f)-2*pos));
            x = (pos-y*NUM_BLOCKS+y*(y+1)/2);
            if (x < y || x >= NUM_BLOCKS) { // Occasionally happens due to roundoff error.
                y += (x < y ? -1 : 1);
                x = (pos-y*NUM_BLOCKS+y*(y+1)/2);
            }
            unsigned int atom1 = x*TILE_SIZE + tgx;
            data.pos = trimTo3(posq[atom1]);
            float2 params1 = params[atom1];
            data.radius = params1.x;
            data.scaledRadius = params1.y;
            if (pos >= end)
                ; // This warp is done.
            else if (x == y) {
                // This tile is on the diagonal.

                localData[threadIdx.x].pos = data.pos;
                localData[threadIdx.x].radius = params1.x;
                localData[threadIdx.x].scaledRadius = params1.y;
                for (unsigned int j = 0; j < TILE_SIZE; j++) {
                    int atom2 = y*TILE_SIZE+j;
                    if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS && atom1 != atom2)
                        computeBornSumOneInteraction(data, localData[tbx+j]);
                }
            }
            else {
                // This is an off-diagonal tile.

                if (lasty != y) {
                    unsigned int j = y*TILE_SIZE + tgx;
                    real4 tempPosq = posq[j];
                    localData[threadIdx.x].pos = trimTo3(tempPosq);
                    float2 tempParams = params[j];
                    localData[threadIdx.x].radius = tempParams.x;
                    localData[threadIdx.x].scaledRadius = tempParams.y;
                }
                localData[threadIdx.x].bornSum = 0;
                
                // Compute the full set of interactions in this tile.

                unsigned int tj = tgx;
                for (unsigned int j = 0; j < TILE_SIZE; j++) {
                    int atom2 = y*TILE_SIZE+tj;
                    if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS) {
                        computeBornSumOneInteraction(data, localData[tbx+tj]);
                        computeBornSumOneInteraction(localData[tbx+tj], data);
                    }
                    tj = (tj + 1) & (TILE_SIZE - 1);
                }
            }
        }
        
        // Write results.
        
        if (pos < end) {
            const unsigned int offset = x*TILE_SIZE + tgx;
            atomicAdd(&bornSum[offset], static_cast<unsigned long long>((long long) (data.bornSum*0x100000000)));
        }
        if (pos < end && x != y) {
            const unsigned int offset = y*TILE_SIZE + tgx;
            atomicAdd(&bornSum[offset], static_cast<unsigned long long>((long long) (localData[threadIdx.x].bornSum*0x100000000)));
        }
        lasty = y;
        pos++;
    } while (pos < end);
}

/**
 * Data structure used by computeGKForces().
 */
typedef struct {
    real3 pos, force, dipole, inducedDipole, inducedDipolePolar;
    real quadrupoleXX, quadrupoleXY, quadrupoleXZ;
    real quadrupoleYY, quadrupoleYZ, quadrupoleZZ;
    real q, bornRadius, bornForce;
} AtomData2;

__device__ void computeOneInteractionF1(AtomData2& atom1, volatile AtomData2& atom2, real& outputEnergy, real3& force);
__device__ void computeOneInteractionF2(AtomData2& atom1, volatile AtomData2& atom2, real& outputEnergy, real3& force);
__device__ void computeOneInteractionT1(AtomData2& atom1, volatile AtomData2& atom2, real3& torque);
__device__ void computeOneInteractionT2(AtomData2& atom1, volatile AtomData2& atom2, real3& torque);
__device__ void computeOneInteractionB1B2(AtomData2& atom1, volatile AtomData2& atom2);

inline __device__ void loadAtomData2(AtomData2& data, int atom, const real4* __restrict__ posq, const real* __restrict__ labFrameDipole,
        const real* __restrict__ labFrameQuadrupole, const real3* __restrict__ inducedDipole, const real3* __restrict__ inducedDipolePolar, const real* __restrict__ bornRadius) {
    real4 atomPosq = posq[atom];
    data.pos = trimTo3(atomPosq);
    data.q = atomPosq.w;
    data.dipole.x = labFrameDipole[atom*3];
    data.dipole.y = labFrameDipole[atom*3+1];
    data.dipole.z = labFrameDipole[atom*3+2];
    data.quadrupoleXX = labFrameQuadrupole[atom*5];
    data.quadrupoleXY = labFrameQuadrupole[atom*5+1];
    data.quadrupoleXZ = labFrameQuadrupole[atom*5+2];
    data.quadrupoleYY = labFrameQuadrupole[atom*5+3];
    data.quadrupoleYZ = labFrameQuadrupole[atom*5+4];
    data.quadrupoleZZ = -(data.quadrupoleXX+data.quadrupoleYY);
    data.inducedDipole = inducedDipole[atom];
    data.inducedDipolePolar = inducedDipolePolar[atom];
    data.bornRadius = bornRadius[atom];
}

inline __device__ void zeroAtomData(AtomData2& data) {
    data.force = make_real3(0);
    data.bornForce = 0;
}

/**
 * Compute electrostatic interactions.
 */
extern "C" __global__ void computeGKForces(
        unsigned long long* __restrict__ forceBuffers, unsigned long long* __restrict__ torqueBuffers, mixed* __restrict__ energyBuffer,
        const real4* __restrict__ posq, unsigned int startTileIndex, unsigned int numTileIndices, const real* __restrict__ labFrameDipole,
        const real* __restrict__ labFrameQuadrupole, const real3* __restrict__ inducedDipole, const real3* __restrict__ inducedDipolePolar,
        const real* __restrict__ bornRadii, unsigned long long* __restrict__ bornForce) {
    unsigned int totalWarps = (blockDim.x*gridDim.x)/TILE_SIZE;
    unsigned int warp = (blockIdx.x*blockDim.x+threadIdx.x)/TILE_SIZE;
    const unsigned int numTiles = numTileIndices;
    unsigned int pos = (unsigned int) (startTileIndex+warp*(long long)numTiles/totalWarps);
    unsigned int end = (unsigned int) (startTileIndex+(warp+1)*(long long)numTiles/totalWarps);
    mixed energy = 0;
    __shared__ AtomData2 localData[GK_FORCE_THREAD_BLOCK_SIZE];
    
    do {
        // Extract the coordinates of this tile
        const unsigned int tgx = threadIdx.x & (TILE_SIZE-1);
        const unsigned int tbx = threadIdx.x - tgx;
        int x, y;
        AtomData2 data;
        if (pos < end) {
            y = (int) floor(NUM_BLOCKS+0.5f-SQRT((NUM_BLOCKS+0.5f)*(NUM_BLOCKS+0.5f)-2*pos));
            x = (pos-y*NUM_BLOCKS+y*(y+1)/2);
            if (x < y || x >= NUM_BLOCKS) { // Occasionally happens due to roundoff error.
                y += (x < y ? -1 : 1);
                x = (pos-y*NUM_BLOCKS+y*(y+1)/2);
            }
            unsigned int atom1 = x*TILE_SIZE + tgx;
            loadAtomData2(data, atom1, posq, labFrameDipole, labFrameQuadrupole, inducedDipole, inducedDipolePolar, bornRadii);
            zeroAtomData(data);
            if (pos >= end)
                ; // This warp is done.
            else if (x == y) {
                // This tile is on the diagonal.

                localData[threadIdx.x].pos = data.pos;
                localData[threadIdx.x].q = data.q;
                localData[threadIdx.x].dipole = data.dipole;
                localData[threadIdx.x].quadrupoleXX = data.quadrupoleXX;
                localData[threadIdx.x].quadrupoleXY = data.quadrupoleXY;
                localData[threadIdx.x].quadrupoleXZ = data.quadrupoleXZ;
                localData[threadIdx.x].quadrupoleYY = data.quadrupoleYY;
                localData[threadIdx.x].quadrupoleYZ = data.quadrupoleYZ;
                localData[threadIdx.x].quadrupoleZZ = data.quadrupoleZZ;
                localData[threadIdx.x].inducedDipole = data.inducedDipole;
                localData[threadIdx.x].inducedDipolePolar = data.inducedDipolePolar;
                localData[threadIdx.x].bornRadius = data.bornRadius;
                
                // Compute forces.
                
                for (unsigned int j = 0; j < TILE_SIZE; j++) {
                    int atom2 = y*TILE_SIZE+j;
                    if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS) {
                        real3 tempForce;
                        real tempEnergy;
                        computeOneInteractionF1(data, localData[tbx+j], tempEnergy, tempForce);
                        computeOneInteractionF2(data, localData[tbx+j], tempEnergy, tempForce);
                        data.force += tempForce;
                        energy += 0.5f*tempEnergy;
                    }
                }
                data.force *= 0.5f;
                atomicAdd(&forceBuffers[atom1], static_cast<unsigned long long>((long long) (data.force.x*0x100000000)));
                atomicAdd(&forceBuffers[atom1+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.y*0x100000000)));
                atomicAdd(&forceBuffers[atom1+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.z*0x100000000)));
                
                // Compute torques.
                
                zeroAtomData(data);
                for (unsigned int j = 0; j < TILE_SIZE; j++) {
                    int atom2 = y*TILE_SIZE+j;
                    if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS) {
                        real3 tempTorque;
                        computeOneInteractionT1(data, localData[tbx+j], tempTorque);
                        computeOneInteractionT2(data, localData[tbx+j], tempTorque);
                        data.force += tempTorque;
                    }
                }
                atomicAdd(&torqueBuffers[atom1], static_cast<unsigned long long>((long long) (data.force.x*0x100000000)));
                atomicAdd(&torqueBuffers[atom1+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.y*0x100000000)));
                atomicAdd(&torqueBuffers[atom1+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.z*0x100000000)));
                
                // Compute chain rule terms.
                
                zeroAtomData(data);
                for (unsigned int j = 0; j < TILE_SIZE; j++) {
                    int atom2 = y*TILE_SIZE+j;
                    if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS)
                        computeOneInteractionB1B2(data, localData[tbx+j]);
                }
                atomicAdd(&bornForce[atom1], static_cast<unsigned long long>((long long) (data.bornForce*0x100000000)));
            }
            else {
                // This is an off-diagonal tile.

                unsigned int j = y*TILE_SIZE + tgx;
                loadAtomData2(localData[threadIdx.x], j, posq, labFrameDipole, labFrameQuadrupole, inducedDipole, inducedDipolePolar, bornRadii);
                zeroAtomData(localData[threadIdx.x]);
                unsigned int tj = tgx;
                for (j = 0; j < TILE_SIZE; j++) {
                    int atom2 = y*TILE_SIZE+tj;
                    if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS) {
                        real3 tempForce;
                        real tempEnergy;
                        computeOneInteractionF1(data, localData[tbx+tj], tempEnergy, tempForce);
                        computeOneInteractionF2(data, localData[tbx+tj], tempEnergy, tempForce);
                        data.force += tempForce;
                        localData[tbx+tj].force -= tempForce;
                        energy += tempEnergy;
                    }
                    tj = (tj + 1) & (TILE_SIZE - 1);
                }
                data.force *= 0.5f;
                localData[threadIdx.x].force *= 0.5f;
                if (pos < end) {
                    unsigned int offset = x*TILE_SIZE + tgx;
                    atomicAdd(&forceBuffers[offset], static_cast<unsigned long long>((long long) (data.force.x*0x100000000)));
                    atomicAdd(&forceBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.y*0x100000000)));
                    atomicAdd(&forceBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.z*0x100000000)));
                    offset = y*TILE_SIZE + tgx;
                    atomicAdd(&forceBuffers[offset], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.x*0x100000000)));
                    atomicAdd(&forceBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.y*0x100000000)));
                    atomicAdd(&forceBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.z*0x100000000)));
                }

                // Compute torques.

                zeroAtomData(data);
                zeroAtomData(localData[threadIdx.x]);
                for (j = 0; j < TILE_SIZE; j++) {
                    int atom2 = y*TILE_SIZE+tj;
                    if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS) {
                        real3 tempTorque;
                        computeOneInteractionT1(data, localData[tbx+tj], tempTorque);
                        computeOneInteractionT2(data, localData[tbx+tj], tempTorque);
                        data.force += tempTorque;
                        computeOneInteractionT1(localData[tbx+tj], data, tempTorque);
                        computeOneInteractionT2(localData[tbx+tj], data, tempTorque);
                        localData[tbx+tj].force += tempTorque;
                    }
                    tj = (tj + 1) & (TILE_SIZE - 1);
                }
                if (pos < end) {
                    unsigned int offset = x*TILE_SIZE + tgx;
                    atomicAdd(&torqueBuffers[offset], static_cast<unsigned long long>((long long) (data.force.x*0x100000000)));
                    atomicAdd(&torqueBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.y*0x100000000)));
                    atomicAdd(&torqueBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.z*0x100000000)));
                    offset = y*TILE_SIZE + tgx;
                    atomicAdd(&torqueBuffers[offset], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.x*0x100000000)));
                    atomicAdd(&torqueBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.y*0x100000000)));
                    atomicAdd(&torqueBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.z*0x100000000)));
                }

                // Compute chain rule terms.

                zeroAtomData(data);
                zeroAtomData(localData[threadIdx.x]);
                for (j = 0; j < TILE_SIZE; j++) {
                    int atom2 = y*TILE_SIZE+tj;
                    if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS)
                        computeOneInteractionB1B2(data, localData[tbx+tj]);
                    tj = (tj + 1) & (TILE_SIZE - 1);
                }
                if (pos < end) {
                    unsigned int offset = x*TILE_SIZE + tgx;
                    atomicAdd(&bornForce[offset], static_cast<unsigned long long>((long long) (data.bornForce*0x100000000)));
                    offset = y*TILE_SIZE + tgx;
                    atomicAdd(&bornForce[offset], static_cast<unsigned long long>((long long) (localData[threadIdx.x].bornForce*0x100000000)));
                }
            }
        }
        pos++;
    } while (pos < end);
    energyBuffer[blockIdx.x*blockDim.x+threadIdx.x] += energy*0.5f;
}


/**
 * Data structure used by computeChainRuleForce().
 */
typedef struct {
    real3 pos, force;
    real radius, scaledRadius, bornRadius, bornForce;
} AtomData3;

inline __device__ void loadAtomData3(AtomData3& data, int atom, const real4* __restrict__ posq, const float2* __restrict__ params, const real* __restrict__ bornRadius, const long long* __restrict__ bornForce) {
    data.pos = trimTo3(posq[atom]);
    data.bornRadius = bornRadius[atom];
    float2 params1 = params[atom];
    data.radius = params1.x;
    data.scaledRadius = params1.y;
    data.bornForce = bornForce[atom]/(real) 0x100000000;
}

__device__ void computeBornChainRuleInteraction(AtomData3& atom1, AtomData3& atom2, real3& force) {
    real third = 1/(real) 3;
    real pi43 = 4*third*M_PI;
    real factor = -POW(M_PI, third)*POW((real) 6, 2/(real) 3)/9;
    real term = pi43/(atom1.bornRadius*atom1.bornRadius*atom1.bornRadius);
    term = factor/POW(term, 4/(real) 3);

    real3 delta = atom2.pos-atom1.pos;

    float sk = atom2.scaledRadius;
    real sk2 = sk*sk;
    real r2 = dot(delta, delta);
    real r = SQRT(r2);
    real de = 0;

    if (atom1.radius > r + sk)
        return; // No descreening due to atom1 engulfing atom2.

    if (atom1.radius+r < sk) {
        real uik = sk-r;
        real uik4 = uik*uik;
        uik4 = uik4*uik4;
        de = -4*M_PI/uik4;
        real lik = sk - r;
        real lik4 = lik*lik;
        lik4 = lik4*lik4;
        de += 0.25f*M_PI*(sk2-4*sk*r+17*r2)/(r2*lik4);
    }
    else if (r < atom1.radius+sk) {
        real lik = atom1.radius;
        real lik4 = lik*lik;
        lik4 = lik4*lik4;
        de += 0.25f*M_PI*(2*atom1.radius*atom1.radius-sk2-r2)/(r2*lik4);
    }
    else {
        real lik = r-sk;
        real lik4 = lik*lik;
        lik4 = lik4*lik4;
        de += 0.25f*M_PI*(sk2-4*sk*r+r2)/(r2*lik4);
    }
    real uik = r+sk;
    real uik4 = uik*uik;
    uik4 = uik4*uik4;
    de -= 0.25f*M_PI*(sk2+4*sk*r+r2)/(r2*uik4);
    real dbr = term*de/r;
    de = dbr*atom1.bornForce;
    force = delta*de;
}

/**
 * Compute chain rule terms.
 */
extern "C" __global__ void computeChainRuleForce(
        unsigned long long* __restrict__ forceBuffers, const real4* __restrict__ posq, unsigned int startTileIndex, unsigned int numTileIndices,
        const float2* __restrict__ params, const real* __restrict__ bornRadii, const long long* __restrict__ bornForce) {
    unsigned int totalWarps = (blockDim.x*gridDim.x)/TILE_SIZE;
    unsigned int warp = (blockIdx.x*blockDim.x+threadIdx.x)/TILE_SIZE;
    const unsigned int numTiles = numTileIndices;
    unsigned int pos = startTileIndex+warp*numTiles/totalWarps;
    unsigned int end = startTileIndex+(warp+1)*numTiles/totalWarps;
    __shared__ AtomData3 localData[CHAIN_RULE_THREAD_BLOCK_SIZE];
    
    do {
        // Extract the coordinates of this tile
        const unsigned int tgx = threadIdx.x & (TILE_SIZE-1);
        const unsigned int tbx = threadIdx.x - tgx;
        int x, y;
        AtomData3 data;
        if (pos < end) {
            y = (int) floor(NUM_BLOCKS+0.5f-SQRT((NUM_BLOCKS+0.5f)*(NUM_BLOCKS+0.5f)-2*pos));
            x = (pos-y*NUM_BLOCKS+y*(y+1)/2);
            if (x < y || x >= NUM_BLOCKS) { // Occasionally happens due to roundoff error.
                y += (x < y ? -1 : 1);
                x = (pos-y*NUM_BLOCKS+y*(y+1)/2);
            }
            unsigned int atom1 = x*TILE_SIZE + tgx;
            loadAtomData3(data, atom1, posq, params, bornRadii, bornForce);
            data.force = make_real3(0);
            if (pos >= end)
                ; // This warp is done.
            else if (x == y) {
                // This tile is on the diagonal.

                localData[threadIdx.x].pos = data.pos;
                localData[threadIdx.x].radius = data.radius;
                localData[threadIdx.x].scaledRadius = data.scaledRadius;
                localData[threadIdx.x].bornRadius = data.bornRadius;
                localData[threadIdx.x].bornForce = data.bornForce;
                localData[threadIdx.x].force = make_real3(0);
                
                // Compute forces.
                
                for (unsigned int j = (tgx+1)&(TILE_SIZE-1); j != tgx; j = (j+1)&(TILE_SIZE-1)) {
                    int atom2 = y*TILE_SIZE+j;
                    if (atom1 != atom2 && atom1 < NUM_ATOMS && atom2 < NUM_ATOMS) {
                        real3 tempForce;
                        computeBornChainRuleInteraction(data, localData[tbx+j], tempForce);
                        data.force -= tempForce;
                        localData[tbx+j].force += tempForce;
                    }
                }
                atomicAdd(&forceBuffers[atom1], static_cast<unsigned long long>((long long) ((data.force.x+localData[threadIdx.x].force.x)*0x100000000)));
                atomicAdd(&forceBuffers[atom1+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) ((data.force.y+localData[threadIdx.x].force.y)*0x100000000)));
                atomicAdd(&forceBuffers[atom1+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) ((data.force.z+localData[threadIdx.x].force.z)*0x100000000)));
            }
            else {
                // This is an off-diagonal tile.

                unsigned int j = y*TILE_SIZE + tgx;
                loadAtomData3(localData[threadIdx.x], j, posq, params, bornRadii, bornForce);
                localData[threadIdx.x].force = make_real3(0);
                unsigned int tj = tgx;
                for (j = 0; j < TILE_SIZE; j++) {
                    int atom2 = y*TILE_SIZE+tj;
                    if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS) {
                        real3 tempForce;
                        computeBornChainRuleInteraction(data, localData[tbx+tj], tempForce);
                        data.force -= tempForce;
                        localData[tbx+tj].force += tempForce;
                        computeBornChainRuleInteraction(localData[tbx+tj], data, tempForce);
                        data.force += tempForce;
                        localData[tbx+tj].force -= tempForce;
                    }
                    tj = (tj + 1) & (TILE_SIZE - 1);
                }
                if (pos < end) {
                    unsigned int offset = x*TILE_SIZE + tgx;
                    atomicAdd(&forceBuffers[offset], static_cast<unsigned long long>((long long) (data.force.x*0x100000000)));
                    atomicAdd(&forceBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.y*0x100000000)));
                    atomicAdd(&forceBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.z*0x100000000)));
                    offset = y*TILE_SIZE + tgx;
                    atomicAdd(&forceBuffers[offset], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.x*0x100000000)));
                    atomicAdd(&forceBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.y*0x100000000)));
                    atomicAdd(&forceBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.z*0x100000000)));
                }
            }
        }
        pos++;
    } while (pos < end);
}

typedef struct {
    real3 pos, force, dipole, inducedDipole, inducedDipolePolar, inducedDipoleS, inducedDipolePolarS;
    real q, quadrupoleXX, quadrupoleXY, quadrupoleXZ;
    real quadrupoleYY, quadrupoleYZ, quadrupoleZZ;
    float thole, damp;
} AtomData4;

__device__ void computeOneEDiffInteractionF1(AtomData4& atom1, volatile AtomData4& atom2, float dScale, float pScale, real& outputEnergy, real3& outputForce);
__device__ void computeOneEDiffInteractionT1(AtomData4& atom1, volatile AtomData4& atom2, float dScale, float pScale, real3& outputForce);
__device__ void computeOneEDiffInteractionT3(AtomData4& atom1, volatile AtomData4& atom2, float dScale, float pScale, real3& outputForce);

inline __device__ void loadAtomData4(AtomData4& data, int atom, const real4* __restrict__ posq, const real* __restrict__ labFrameDipole,
        const real* __restrict__ labFrameQuadrupole, const real3* __restrict__ inducedDipole, const real3* __restrict__ inducedDipolePolar,
        const real3* __restrict__ inducedDipoleS, const real3* __restrict__ inducedDipolePolarS, const float2* __restrict__ dampingAndThole) {
    real4 atomPosq = posq[atom];
    data.pos = make_real3(atomPosq.x, atomPosq.y, atomPosq.z);
    data.q = atomPosq.w;
    data.dipole.x = labFrameDipole[atom*3];
    data.dipole.y = labFrameDipole[atom*3+1];
    data.dipole.z = labFrameDipole[atom*3+2];
    data.quadrupoleXX = labFrameQuadrupole[atom*5];
    data.quadrupoleXY = labFrameQuadrupole[atom*5+1];
    data.quadrupoleXZ = labFrameQuadrupole[atom*5+2];
    data.quadrupoleYY = labFrameQuadrupole[atom*5+3];
    data.quadrupoleYZ = labFrameQuadrupole[atom*5+4];
    data.quadrupoleZZ = -(data.quadrupoleXX+data.quadrupoleYY);
    data.inducedDipole = inducedDipole[atom];
    data.inducedDipolePolar = inducedDipolePolar[atom];
    data.inducedDipoleS = inducedDipoleS[atom];
    data.inducedDipolePolarS = inducedDipolePolarS[atom];
    float2 temp = dampingAndThole[atom];
    data.damp = temp.x;
    data.thole = temp.y;
}

__device__ real computeDScaleFactor(unsigned int polarizationGroup, int index) {
    return (polarizationGroup & 1<<index ? 0 : 1);
}

__device__ float computePScaleFactor(uint2 covalent, unsigned int polarizationGroup, int index) {
    int mask = 1<<index;
    bool x = (covalent.x & mask);
    bool y = (covalent.y & mask);
    bool p = (polarizationGroup & mask);
    return (x && y ? 0.0f : (x && p ? 0.5f : 1.0f));
}

/**
 * Compute electrostatic interactions.
 */
extern "C" __global__ void computeEDiffForce(
        unsigned long long* __restrict__ forceBuffers, unsigned long long* __restrict__ torqueBuffers, mixed* __restrict__ energyBuffer,
        const real4* __restrict__ posq, const uint2* __restrict__ covalentFlags, const unsigned int* __restrict__ polarizationGroupFlags,
        const int2* __restrict__ exclusionTiles, unsigned int startTileIndex, unsigned int numTileIndices,
        const real* __restrict__ labFrameDipole, const real* __restrict__ labFrameQuadrupole, const real3* __restrict__ inducedDipole,
        const real3* __restrict__ inducedDipolePolar, const real3* __restrict__ inducedDipoleS, const real3* __restrict__ inducedDipolePolarS,
        const float2* __restrict__ dampingAndThole) {
    const unsigned int totalWarps = (blockDim.x*gridDim.x)/TILE_SIZE;
    const unsigned int warp = (blockIdx.x*blockDim.x+threadIdx.x)/TILE_SIZE;
    const unsigned int tgx = threadIdx.x & (TILE_SIZE-1);
    const unsigned int tbx = threadIdx.x - tgx;
    mixed energy = 0;
    __shared__ AtomData4 localData[EDIFF_THREAD_BLOCK_SIZE];

    // First loop: process tiles that contain exclusions.
    
    const unsigned int firstExclusionTile = FIRST_EXCLUSION_TILE+warp*(LAST_EXCLUSION_TILE-FIRST_EXCLUSION_TILE)/totalWarps;
    const unsigned int lastExclusionTile = FIRST_EXCLUSION_TILE+(warp+1)*(LAST_EXCLUSION_TILE-FIRST_EXCLUSION_TILE)/totalWarps;
    for (int pos = firstExclusionTile; pos < lastExclusionTile; pos++) {
        const int2 tileIndices = exclusionTiles[pos];
        const unsigned int x = tileIndices.x;
        const unsigned int y = tileIndices.y;
        AtomData4 data;
        data.force = make_real3(0);
        unsigned int atom1 = x*TILE_SIZE + tgx;
        loadAtomData4(data, atom1, posq, labFrameDipole, labFrameQuadrupole, inducedDipole, inducedDipolePolar, inducedDipoleS, inducedDipolePolarS, dampingAndThole);
        uint2 covalent = covalentFlags[pos*TILE_SIZE+tgx];
        unsigned int polarizationGroup = polarizationGroupFlags[pos*TILE_SIZE+tgx];
        if (x == y) {
            // This tile is on the diagonal.

            localData[threadIdx.x].pos = data.pos;
            localData[threadIdx.x].q = data.q;
            localData[threadIdx.x].dipole = data.dipole;
            localData[threadIdx.x].quadrupoleXX = data.quadrupoleXX;
            localData[threadIdx.x].quadrupoleXY = data.quadrupoleXY;
            localData[threadIdx.x].quadrupoleXZ = data.quadrupoleXZ;
            localData[threadIdx.x].quadrupoleYY = data.quadrupoleYY;
            localData[threadIdx.x].quadrupoleYZ = data.quadrupoleYZ;
            localData[threadIdx.x].quadrupoleZZ = data.quadrupoleZZ;
            localData[threadIdx.x].inducedDipole = data.inducedDipole;
            localData[threadIdx.x].inducedDipolePolar = data.inducedDipolePolar;
            localData[threadIdx.x].inducedDipoleS = data.inducedDipoleS;
            localData[threadIdx.x].inducedDipolePolarS = data.inducedDipolePolarS;
            localData[threadIdx.x].thole = data.thole;
            localData[threadIdx.x].damp = data.damp;

            // Compute forces.

            for (unsigned int j = 0; j < TILE_SIZE; j++) {
                int atom2 = y*TILE_SIZE+j;
                if (atom1 != atom2 && atom1 < NUM_ATOMS && atom2 < NUM_ATOMS) {
                    real3 tempForce;
                    real tempEnergy;
                    float d = computeDScaleFactor(polarizationGroup, j);
                    float p = computePScaleFactor(covalent, polarizationGroup, j);
                    computeOneEDiffInteractionF1(data, localData[tbx+j], d, p, tempEnergy, tempForce);
                    energy += 0.25f*tempEnergy;
                    data.force += tempForce;
                }
            }
            data.force *= ENERGY_SCALE_FACTOR;
            atomicAdd(&forceBuffers[atom1], static_cast<unsigned long long>((long long) (data.force.x*0x100000000)));
            atomicAdd(&forceBuffers[atom1+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.y*0x100000000)));
            atomicAdd(&forceBuffers[atom1+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.z*0x100000000)));

            // Compute torques.

            data.force = make_real3(0);
            for (unsigned int j = 0; j < TILE_SIZE; j++) {
                int atom2 = y*TILE_SIZE+j;
                if (atom1 != atom2 && atom1 < NUM_ATOMS && atom2 < NUM_ATOMS) {
                    real3 tempTorque;
                    float d = computeDScaleFactor(polarizationGroup, j);
                    float p = computePScaleFactor(covalent, polarizationGroup, j);
                    computeOneEDiffInteractionT1(data, localData[tbx+j], d, p, tempTorque);
                    data.force += tempTorque;
                }
            }
            data.force *= ENERGY_SCALE_FACTOR;
            atomicAdd(&torqueBuffers[atom1], static_cast<unsigned long long>((long long) (data.force.x*0x100000000)));
            atomicAdd(&torqueBuffers[atom1+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.y*0x100000000)));
            atomicAdd(&torqueBuffers[atom1+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.z*0x100000000)));
        }
        else {
            // This is an off-diagonal tile.

            unsigned int j = y*TILE_SIZE + tgx;
            loadAtomData4(localData[threadIdx.x], j, posq, labFrameDipole, labFrameQuadrupole, inducedDipole, inducedDipolePolar, inducedDipoleS, inducedDipolePolarS, dampingAndThole);
            localData[threadIdx.x].force = make_real3(0);

            // Compute forces.

            unsigned int tj = tgx;
            for (j = 0; j < TILE_SIZE; j++) {
                int atom2 = y*TILE_SIZE+tj;
                if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS) {
                    real3 tempForce;
                    real tempEnergy;
                    float d = computeDScaleFactor(polarizationGroup, tj);
                    float p = computePScaleFactor(covalent, polarizationGroup, tj);
                    computeOneEDiffInteractionF1(data, localData[tbx+tj], d, p, tempEnergy, tempForce);
                    energy += 0.5f*tempEnergy;
                    data.force += tempForce;
                    localData[tbx+tj].force -= tempForce;
                }
                tj = (tj + 1) & (TILE_SIZE - 1);
            }
            data.force *= ENERGY_SCALE_FACTOR;
            localData[threadIdx.x].force *= ENERGY_SCALE_FACTOR;
            unsigned int offset = x*TILE_SIZE + tgx;
            atomicAdd(&forceBuffers[offset], static_cast<unsigned long long>((long long) (data.force.x*0x100000000)));
            atomicAdd(&forceBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.y*0x100000000)));
            atomicAdd(&forceBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.z*0x100000000)));
            offset = y*TILE_SIZE + tgx;
            atomicAdd(&forceBuffers[offset], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.x*0x100000000)));
            atomicAdd(&forceBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.y*0x100000000)));
            atomicAdd(&forceBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.z*0x100000000)));

            // Compute torques.

            data.force = make_real3(0);
            localData[threadIdx.x].force = make_real3(0);
            for (j = 0; j < TILE_SIZE; j++) {
                int atom2 = y*TILE_SIZE+tj;
                if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS) {
                    real3 tempTorque;
                    float d = computeDScaleFactor(polarizationGroup, tj);
                    float p = computePScaleFactor(covalent, polarizationGroup, tj);
                    computeOneEDiffInteractionT1(data, localData[tbx+tj], d, p, tempTorque);
                    data.force += tempTorque;
                    computeOneEDiffInteractionT3(data, localData[tbx+tj], d, p, tempTorque);
                    localData[tbx+tj].force += tempTorque;
                }
                tj = (tj + 1) & (TILE_SIZE - 1);
            }
            data.force *= ENERGY_SCALE_FACTOR;
            localData[threadIdx.x].force *= ENERGY_SCALE_FACTOR;
            offset = x*TILE_SIZE + tgx;
            atomicAdd(&torqueBuffers[offset], static_cast<unsigned long long>((long long) (data.force.x*0x100000000)));
            atomicAdd(&torqueBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.y*0x100000000)));
            atomicAdd(&torqueBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.z*0x100000000)));
            offset = y*TILE_SIZE + tgx;
            atomicAdd(&torqueBuffers[offset], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.x*0x100000000)));
            atomicAdd(&torqueBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.y*0x100000000)));
            atomicAdd(&torqueBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.z*0x100000000)));
        }
    }

    // Second loop: tiles without exclusions (by enumerating all of them, since there's no cutoff).

    const unsigned int numTiles = numTileIndices;
    int pos = startTileIndex+warp*numTiles/totalWarps;
    int end = startTileIndex+(warp+1)*numTiles/totalWarps;
    int skipBase = 0;
    int currentSkipIndex = tbx;
    __shared__ volatile int skipTiles[EDIFF_THREAD_BLOCK_SIZE];
    skipTiles[threadIdx.x] = -1;

    while (pos < end) {
        // Extract the coordinates of this tile.

        int x, y;
        y = (int) floor(NUM_BLOCKS+0.5f-SQRT((NUM_BLOCKS+0.5f)*(NUM_BLOCKS+0.5f)-2*pos));
        x = (pos-y*NUM_BLOCKS+y*(y+1)/2);
        if (x < y || x >= NUM_BLOCKS) { // Occasionally happens due to roundoff error.
            y += (x < y ? -1 : 1);
            x = (pos-y*NUM_BLOCKS+y*(y+1)/2);
        }

        // Skip over tiles that have exclusions, since they were already processed.

        while (skipTiles[tbx+TILE_SIZE-1] < pos) {
            if (skipBase+tgx < NUM_TILES_WITH_EXCLUSIONS) {
                int2 tile = exclusionTiles[skipBase+tgx];
                skipTiles[threadIdx.x] = tile.x + tile.y*NUM_BLOCKS - tile.y*(tile.y+1)/2;
            }
            else
                skipTiles[threadIdx.x] = end;
            skipBase += TILE_SIZE;            
            currentSkipIndex = tbx;
        }
        while (skipTiles[currentSkipIndex] < pos)
            currentSkipIndex++;
        bool includeTile = (skipTiles[currentSkipIndex] != pos);
        if (includeTile) {
            unsigned int atom1 = x*TILE_SIZE + tgx;

            // Load atom data for this tile.

            AtomData4 data;
            data.force = make_real3(0);
            loadAtomData4(data, atom1, posq, labFrameDipole, labFrameQuadrupole, inducedDipole, inducedDipolePolar, inducedDipoleS, inducedDipolePolarS, dampingAndThole);
            loadAtomData4(localData[threadIdx.x], atom1, posq, labFrameDipole, labFrameQuadrupole, inducedDipole, inducedDipolePolar, inducedDipoleS, inducedDipolePolarS, dampingAndThole);
            unsigned int j = y*TILE_SIZE + tgx;
            loadAtomData4(localData[threadIdx.x], j, posq, labFrameDipole, labFrameQuadrupole, inducedDipole, inducedDipolePolar, inducedDipoleS, inducedDipolePolarS, dampingAndThole);
            localData[threadIdx.x].force = make_real3(0);

            // Compute forces.

            unsigned int tj = tgx;
            for (unsigned int j = 0; j < TILE_SIZE; j++) {
                int atom2 = y*TILE_SIZE+tj;
                if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS) {
                    real3 tempForce;
                    real tempEnergy;
                    computeOneEDiffInteractionF1(data, localData[tbx+tj], 1, 1, tempEnergy, tempForce);
                    energy += 0.5f*tempEnergy;
                    data.force += tempForce;
                    localData[tbx+tj].force -= tempForce;
                }
                tj = (tj + 1) & (TILE_SIZE - 1);
            }
            data.force *= ENERGY_SCALE_FACTOR;
            localData[threadIdx.x].force *= ENERGY_SCALE_FACTOR;
            unsigned int offset = x*TILE_SIZE + tgx;
            atomicAdd(&forceBuffers[offset], static_cast<unsigned long long>((long long) (data.force.x*0x100000000)));
            atomicAdd(&forceBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.y*0x100000000)));
            atomicAdd(&forceBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.z*0x100000000)));
            offset = y*TILE_SIZE + tgx;
            atomicAdd(&forceBuffers[offset], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.x*0x100000000)));
            atomicAdd(&forceBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.y*0x100000000)));
            atomicAdd(&forceBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.z*0x100000000)));

            // Compute torques.

            data.force = make_real3(0);
            localData[threadIdx.x].force = make_real3(0);
            for (unsigned int j = 0; j < TILE_SIZE; j++) {
                int atom2 = y*TILE_SIZE+tj;
                if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS) {
                    real3 tempTorque;
                    computeOneEDiffInteractionT1(data, localData[tbx+tj], 1, 1, tempTorque);
                    data.force += tempTorque;
                    computeOneEDiffInteractionT3(data, localData[tbx+tj], 1, 1, tempTorque);
                    localData[tbx+tj].force += tempTorque;
                }
                tj = (tj + 1) & (TILE_SIZE - 1);
            }
            data.force *= ENERGY_SCALE_FACTOR;
            localData[threadIdx.x].force *= ENERGY_SCALE_FACTOR;
            offset = x*TILE_SIZE + tgx;
            atomicAdd(&torqueBuffers[offset], static_cast<unsigned long long>((long long) (data.force.x*0x100000000)));
            atomicAdd(&torqueBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.y*0x100000000)));
            atomicAdd(&torqueBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.z*0x100000000)));
            offset = y*TILE_SIZE + tgx;
            atomicAdd(&torqueBuffers[offset], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.x*0x100000000)));
            atomicAdd(&torqueBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.y*0x100000000)));
            atomicAdd(&torqueBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.z*0x100000000)));
        }
        pos++;
    }
    energyBuffer[blockIdx.x*blockDim.x+threadIdx.x] += energy*ENERGY_SCALE_FACTOR;
}
