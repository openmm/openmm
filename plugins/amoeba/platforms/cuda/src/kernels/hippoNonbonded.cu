#include "hip/hip_runtime.h"
// This is a modified version of the standard nonbonded kernel for computing HippoNonbondedForce.
// This is needed because of two ways in which it differs from most nonbonded interactions:
// the force between two atoms doesn't always point along the line between them, and we need
// to accumulate torques as well as forces.

#define WARPS_PER_GROUP (THREAD_BLOCK_SIZE/TILE_SIZE)

#ifndef ENABLE_SHUFFLE
typedef struct {
    real x, y, z;
    real q;
    real fx, fy, fz;
    real tx, ty, tz;
    ATOM_PARAMETER_DATA
#ifndef PARAMETER_SIZE_IS_EVEN
    real padding;
#endif
} AtomData;
#endif

#ifdef ENABLE_SHUFFLE
//support for 64 bit shuffles
static __inline__ __device__ float real_shfl(float var, int srcLane) {
    return SHFL(var, srcLane);
}

static __inline__ __device__ double real_shfl(double var, int srcLane) {
    int hi, lo;
    asm volatile("mov.b64 { %0, %1 }, %2;" : "=r"(lo), "=r"(hi) : "d"(var));
    hi = SHFL(hi, srcLane);
    lo = SHFL(lo, srcLane);
    return __hiloint2double( hi, lo );
}

static __inline__ __device__ long long real_shfl(long long var, int srcLane) {
    int hi, lo;
    asm volatile("mov.b64 { %0, %1 }, %2;" : "=r"(lo), "=r"(hi) : "l"(var));
    hi = SHFL(hi, srcLane);
    lo = SHFL(lo, srcLane);
    // unforunately there isn't an __nv_hiloint2long(hi,lo) intrinsic cast
    int2 fuse; fuse.x = lo; fuse.y = hi;
    return *reinterpret_cast<long long*>(&fuse);
}
#endif

extern "C" __global__ void computeNonbonded(
        unsigned long long* __restrict__ forceBuffers, mixed* __restrict__ energyBuffer, const real4* __restrict__ posq, const tileflags* __restrict__ exclusions,
        const ushort2* __restrict__ exclusionTiles, unsigned int startTileIndex, unsigned int numTileIndices
#ifdef USE_CUTOFF
        , const int* __restrict__ tiles, const unsigned int* __restrict__ interactionCount, real4 periodicBoxSize, real4 invPeriodicBoxSize, 
        real4 periodicBoxVecX, real4 periodicBoxVecY, real4 periodicBoxVecZ, unsigned int maxTiles, const real4* __restrict__ blockCenter,
        const real4* __restrict__ blockSize, const unsigned int* __restrict__ interactingAtoms, unsigned int maxSinglePairs,
        const int2* __restrict__ singlePairs
#endif
        PARAMETER_ARGUMENTS) {
    const unsigned int totalWarps = (blockDim.x*gridDim.x)/TILE_SIZE;
    const unsigned int warp = (blockIdx.x*blockDim.x+threadIdx.x)/TILE_SIZE; // global warpIndex
    const unsigned int tgx = threadIdx.x & (TILE_SIZE-1); // index within the warp
    const unsigned int tbx = threadIdx.x - tgx;           // block warpIndex
    mixed energy = 0;
    // used shared memory if the device cannot shuffle
#ifndef ENABLE_SHUFFLE
    __shared__ AtomData localData[THREAD_BLOCK_SIZE];
#endif

    // First loop: process tiles that contain exclusions.

    const unsigned int firstExclusionTile = FIRST_EXCLUSION_TILE+warp*(LAST_EXCLUSION_TILE-FIRST_EXCLUSION_TILE)/totalWarps;
    const unsigned int lastExclusionTile = FIRST_EXCLUSION_TILE+(warp+1)*(LAST_EXCLUSION_TILE-FIRST_EXCLUSION_TILE)/totalWarps;
    for (int pos = firstExclusionTile; pos < lastExclusionTile; pos++) {
        const ushort2 tileIndices = exclusionTiles[pos];
        const unsigned int x = tileIndices.x;
        const unsigned int y = tileIndices.y;
        real3 force = make_real3(0);
        real3 torque = make_real3(0);
        unsigned int atom1 = x*TILE_SIZE + tgx;
        real4 posq1 = posq[atom1];
        LOAD_ATOM1_PARAMETERS
        tileflags excl = exclusions[pos*TILE_SIZE+tgx];
        const bool hasExclusions = true;
        if (x == y) {
            // This tile is on the diagonal.
#ifdef ENABLE_SHUFFLE
            real4 shflPosq = posq1;
#else
            localData[threadIdx.x].x = posq1.x;
            localData[threadIdx.x].y = posq1.y;
            localData[threadIdx.x].z = posq1.z;
            localData[threadIdx.x].q = posq1.w;
            LOAD_LOCAL_PARAMETERS_FROM_1
#endif

            // we do not need to fetch parameters from global since this is a symmetric tile
            // instead we can broadcast the values using shuffle
            for (unsigned int j = 0; j < TILE_SIZE; j++) {
                int atom2 = tbx+j;
                real4 posq2;
#ifdef ENABLE_SHUFFLE
                BROADCAST_WARP_DATA
#else   
                posq2 = make_real4(localData[atom2].x, localData[atom2].y, localData[atom2].z, localData[atom2].q);
#endif
                real3 delta = make_real3(posq2.x-posq1.x, posq2.y-posq1.y, posq2.z-posq1.z);
#ifdef USE_PERIODIC
                APPLY_PERIODIC_TO_DELTA(delta)
#endif
                real r2 = delta.x*delta.x + delta.y*delta.y + delta.z*delta.z;
                real rInv = RSQRT(r2);
                real r = r2*rInv;
                LOAD_ATOM2_PARAMETERS
                atom2 = y*TILE_SIZE+j;
                real3 tempForce = make_real3(0);
                real3 tempTorque1 = make_real3(0);
                real3 tempTorque2 = make_real3(0);
                bool isExcluded = (atom1 >= NUM_ATOMS || atom2 >= NUM_ATOMS || !(excl & 0x1));
                real tempEnergy = 0.0f;
                const real interactionScale = 0.5f;
                COMPUTE_INTERACTION
                energy += 0.5f*tempEnergy;
                force += tempForce;
                torque += tempTorque1;
                excl >>= 1;
            }
        }
        else {
            // This is an off-diagonal tile.
            unsigned int j = y*TILE_SIZE + tgx;
            real4 shflPosq = posq[j];
#ifdef ENABLE_SHUFFLE
            real3 shflForce = make_real3(0);
            real3 shflTorque = make_real3(0);
#else
            localData[threadIdx.x].x = shflPosq.x;
            localData[threadIdx.x].y = shflPosq.y;
            localData[threadIdx.x].z = shflPosq.z;
            localData[threadIdx.x].q = shflPosq.w;
            localData[threadIdx.x].fx = 0.0f;
            localData[threadIdx.x].fy = 0.0f;
            localData[threadIdx.x].fz = 0.0f;
            localData[threadIdx.x].tx = 0.0f;
            localData[threadIdx.x].ty = 0.0f;
            localData[threadIdx.x].tz = 0.0f;
#endif
            DECLARE_LOCAL_PARAMETERS
            LOAD_LOCAL_PARAMETERS_FROM_GLOBAL
            excl = (excl >> tgx) | (excl << (TILE_SIZE - tgx));
            unsigned int tj = tgx;
            for (j = 0; j < TILE_SIZE; j++) {
                int atom2 = tbx+tj;
#ifdef ENABLE_SHUFFLE
                real4 posq2 = shflPosq;
#else
                real4 posq2 = make_real4(localData[atom2].x, localData[atom2].y, localData[atom2].z, localData[atom2].q);
#endif
                real3 delta = make_real3(posq2.x-posq1.x, posq2.y-posq1.y, posq2.z-posq1.z);
#ifdef USE_PERIODIC
                APPLY_PERIODIC_TO_DELTA(delta)
#endif
                real r2 = delta.x*delta.x + delta.y*delta.y + delta.z*delta.z;
                real rInv = RSQRT(r2);
                real r = r2*rInv;
                LOAD_ATOM2_PARAMETERS
                atom2 = y*TILE_SIZE+tj;
                real3 tempForce = make_real3(0);
                real3 tempTorque1 = make_real3(0);
                real3 tempTorque2 = make_real3(0);
                bool isExcluded = (atom1 >= NUM_ATOMS || atom2 >= NUM_ATOMS || !(excl & 0x1));
                real tempEnergy = 0.0f;
                const real interactionScale = 1.0f;
                COMPUTE_INTERACTION
                energy += tempEnergy;
                force += tempForce;
                torque += tempTorque1;
#ifdef ENABLE_SHUFFLE
                shflForce -= tempForce;
                shflTorque += tempTorque2;
                SHUFFLE_WARP_DATA
                shflTorque.x = real_shfl(shflTorque.x, tgx+1);
                shflTorque.y = real_shfl(shflTorque.y, tgx+1);
                shflTorque.z = real_shfl(shflTorque.z, tgx+1);
#else
                localData[tbx+tj].fx -= tempForce.x;
                localData[tbx+tj].fy -= tempForce.y;
                localData[tbx+tj].fz -= tempForce.z;
                localData[tbx+tj].tx += tempTorque2.x;
                localData[tbx+tj].ty += tempTorque2.y;
                localData[tbx+tj].tz += tempTorque2.z;
#endif
                excl >>= 1;
                // cycles the indices
                // 0 1 2 3 4 5 6 7 -> 1 2 3 4 5 6 7 0
                tj = (tj + 1) & (TILE_SIZE - 1);
            }
            const unsigned int offset = y*TILE_SIZE + tgx;
            // write results for off diagonal tiles
#ifdef ENABLE_SHUFFLE
            atomicAdd(&forceBuffers[offset], static_cast<unsigned long long>((long long) (shflForce.x*0x100000000)));
            atomicAdd(&forceBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (shflForce.y*0x100000000)));
            atomicAdd(&forceBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (shflForce.z*0x100000000)));
            atomicAdd(&torqueBuffers[offset], static_cast<unsigned long long>((long long) (shflTorque.x*0x100000000)));
            atomicAdd(&torqueBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (shflTorque.y*0x100000000)));
            atomicAdd(&torqueBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (shflTorque.z*0x100000000)));
#else
            atomicAdd(&forceBuffers[offset], static_cast<unsigned long long>((long long) (localData[threadIdx.x].fx*0x100000000)));
            atomicAdd(&forceBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].fy*0x100000000)));
            atomicAdd(&forceBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].fz*0x100000000)));
            atomicAdd(&torqueBuffers[offset], static_cast<unsigned long long>((long long) (localData[threadIdx.x].tx*0x100000000)));
            atomicAdd(&torqueBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].ty*0x100000000)));
            atomicAdd(&torqueBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].tz*0x100000000)));
#endif
        }
        // Write results for on and off diagonal tiles

        const unsigned int offset = x*TILE_SIZE + tgx;
        atomicAdd(&forceBuffers[offset], static_cast<unsigned long long>((long long) (force.x*0x100000000)));
        atomicAdd(&forceBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (force.y*0x100000000)));
        atomicAdd(&forceBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (force.z*0x100000000)));
        atomicAdd(&torqueBuffers[offset], static_cast<unsigned long long>((long long) (torque.x*0x100000000)));
        atomicAdd(&torqueBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (torque.y*0x100000000)));
        atomicAdd(&torqueBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (torque.z*0x100000000)));
    }

    // Second loop: tiles without exclusions, either from the neighbor list (with cutoff) or just enumerating all
    // of them (no cutoff).

#ifdef USE_CUTOFF
    const unsigned int numTiles = interactionCount[0];
    if (numTiles > maxTiles)
        return; // There wasn't enough memory for the neighbor list.
    int pos = (int) (numTiles > maxTiles ? startTileIndex+warp*(long long)numTileIndices/totalWarps : warp*(long long)numTiles/totalWarps);
    int end = (int) (numTiles > maxTiles ? startTileIndex+(warp+1)*(long long)numTileIndices/totalWarps : (warp+1)*(long long)numTiles/totalWarps);
#else
    const unsigned int numTiles = numTileIndices;
    int pos = (int) (startTileIndex+warp*(long long)numTiles/totalWarps);
    int end = (int) (startTileIndex+(warp+1)*(long long)numTiles/totalWarps);
#endif
    int skipBase = 0;
    int currentSkipIndex = tbx;
    // atomIndices can probably be shuffled as well
    // but it probably wouldn't make things any faster
    __shared__ int atomIndices[THREAD_BLOCK_SIZE];
    __shared__ volatile int skipTiles[THREAD_BLOCK_SIZE];
    skipTiles[threadIdx.x] = -1;
    
    while (pos < end) {
        const bool hasExclusions = false;
        real3 force = make_real3(0);
        real3 torque = make_real3(0);
        bool includeTile = true;

        // Extract the coordinates of this tile.
        int x, y;
        bool singlePeriodicCopy = false;
#ifdef USE_CUTOFF
        x = tiles[pos];
        real4 blockSizeX = blockSize[x];
        singlePeriodicCopy = (0.5f*periodicBoxSize.x-blockSizeX.x >= MAX_CUTOFF &&
                              0.5f*periodicBoxSize.y-blockSizeX.y >= MAX_CUTOFF &&
                              0.5f*periodicBoxSize.z-blockSizeX.z >= MAX_CUTOFF);
#else
        y = (int) floor(NUM_BLOCKS+0.5f-SQRT((NUM_BLOCKS+0.5f)*(NUM_BLOCKS+0.5f)-2*pos));
        x = (pos-y*NUM_BLOCKS+y*(y+1)/2);
        if (x < y || x >= NUM_BLOCKS) { // Occasionally happens due to roundoff error.
            y += (x < y ? -1 : 1);
            x = (pos-y*NUM_BLOCKS+y*(y+1)/2);
        }

        // Skip over tiles that have exclusions, since they were already processed.

        while (skipTiles[tbx+TILE_SIZE-1] < pos) {
            if (skipBase+tgx < NUM_TILES_WITH_EXCLUSIONS) {
                ushort2 tile = exclusionTiles[skipBase+tgx];
                skipTiles[threadIdx.x] = tile.x + tile.y*NUM_BLOCKS - tile.y*(tile.y+1)/2;
            }
            else
                skipTiles[threadIdx.x] = end;
            skipBase += TILE_SIZE;            
            currentSkipIndex = tbx;
        }
        while (skipTiles[currentSkipIndex] < pos)
            currentSkipIndex++;
        includeTile = (skipTiles[currentSkipIndex] != pos);
#endif
        if (includeTile) {
            unsigned int atom1 = x*TILE_SIZE + tgx;
            // Load atom data for this tile.
            real4 posq1 = posq[atom1];
            LOAD_ATOM1_PARAMETERS
            //const unsigned int localAtomIndex = threadIdx.x;
#ifdef USE_CUTOFF
            unsigned int j = interactingAtoms[pos*TILE_SIZE+tgx];
#else
            unsigned int j = y*TILE_SIZE + tgx;
#endif
            atomIndices[threadIdx.x] = j;
#ifdef ENABLE_SHUFFLE
            DECLARE_LOCAL_PARAMETERS
            real4 shflPosq;
            real3 shflForce = make_real3(0);
            real3 shflTorque = make_real3(0);
#endif
            if (j < PADDED_NUM_ATOMS) {
                // Load position of atom j from from global memory
#ifdef ENABLE_SHUFFLE
                shflPosq = posq[j];
#else
                localData[threadIdx.x].x = posq[j].x;
                localData[threadIdx.x].y = posq[j].y;
                localData[threadIdx.x].z = posq[j].z;
                localData[threadIdx.x].q = posq[j].w;
                localData[threadIdx.x].fx = 0.0f;
                localData[threadIdx.x].fy = 0.0f;
                localData[threadIdx.x].fz = 0.0f;
#endif                
                LOAD_LOCAL_PARAMETERS_FROM_GLOBAL
            }
            else {
#ifdef ENABLE_SHUFFLE
                shflPosq = make_real4(0, 0, 0, 0);
#else
                localData[threadIdx.x].x = 0;
                localData[threadIdx.x].y = 0;
                localData[threadIdx.x].z = 0;
#endif
            }
#ifdef USE_PERIODIC
            if (singlePeriodicCopy) {
                // The box is small enough that we can just translate all the atoms into a single periodic
                // box, then skip having to apply periodic boundary conditions later.
                real4 blockCenterX = blockCenter[x];
                APPLY_PERIODIC_TO_POS_WITH_CENTER(posq1, blockCenterX)
#ifdef ENABLE_SHUFFLE
                APPLY_PERIODIC_TO_POS_WITH_CENTER(shflPosq, blockCenterX)
#else
                APPLY_PERIODIC_TO_POS_WITH_CENTER(localData[threadIdx.x], blockCenterX)
#endif
                unsigned int tj = tgx;
                for (j = 0; j < TILE_SIZE; j++) {
                    int atom2 = tbx+tj;
#ifdef ENABLE_SHUFFLE
                    real4 posq2 = shflPosq; 
#else
                    real4 posq2 = make_real4(localData[atom2].x, localData[atom2].y, localData[atom2].z, localData[atom2].q);
#endif
                    real3 delta = make_real3(posq2.x-posq1.x, posq2.y-posq1.y, posq2.z-posq1.z);
                    real r2 = delta.x*delta.x + delta.y*delta.y + delta.z*delta.z;
                    real rInv = RSQRT(r2);
                    real r = r2*rInv;
                    LOAD_ATOM2_PARAMETERS
                    atom2 = atomIndices[tbx+tj];
                    real3 tempForce = make_real3(0);
                    real3 tempTorque1 = make_real3(0);
                    real3 tempTorque2 = make_real3(0);
                    bool isExcluded = (atom1 >= NUM_ATOMS || atom2 >= NUM_ATOMS);
                    real tempEnergy = 0.0f;
                    const real interactionScale = 1.0f;
                    COMPUTE_INTERACTION
                    energy += tempEnergy;
                    force += tempForce;
                    torque += tempTorque1;
#ifdef ENABLE_SHUFFLE
                    shflForce -= tempForce;
                    shflTorque += tempTorque2;
                    SHUFFLE_WARP_DATA
                    shflTorque.x = real_shfl(shflTorque.x, tgx+1);
                    shflTorque.y = real_shfl(shflTorque.y, tgx+1);
                    shflTorque.z = real_shfl(shflTorque.z, tgx+1);
#else
                    localData[tbx+tj].fx -= tempForce.x;
                    localData[tbx+tj].fy -= tempForce.y;
                    localData[tbx+tj].fz -= tempForce.z;
                    localData[tbx+tj].tx += tempTorque2.x;
                    localData[tbx+tj].ty += tempTorque2.y;
                    localData[tbx+tj].tz += tempTorque2.z;
#endif
                    tj = (tj + 1) & (TILE_SIZE - 1);
                }
            }
            else
#endif
            {
                // We need to apply periodic boundary conditions separately for each interaction.
                unsigned int tj = tgx;
                for (j = 0; j < TILE_SIZE; j++) {
                    int atom2 = tbx+tj;
#ifdef ENABLE_SHUFFLE
                    real4 posq2 = shflPosq;
#else
                    real4 posq2 = make_real4(localData[atom2].x, localData[atom2].y, localData[atom2].z, localData[atom2].q);
#endif
                    real3 delta = make_real3(posq2.x-posq1.x, posq2.y-posq1.y, posq2.z-posq1.z);
#ifdef USE_PERIODIC
                    APPLY_PERIODIC_TO_DELTA(delta)
#endif
                    real r2 = delta.x*delta.x + delta.y*delta.y + delta.z*delta.z;
                    real rInv = RSQRT(r2);
                    real r = r2*rInv;
                    LOAD_ATOM2_PARAMETERS
                    atom2 = atomIndices[tbx+tj];
                    real3 tempForce = make_real3(0);
                    real3 tempTorque1 = make_real3(0);
                    real3 tempTorque2 = make_real3(0);
                    bool isExcluded = (atom1 >= NUM_ATOMS || atom2 >= NUM_ATOMS);
                    real tempEnergy = 0.0f;
                    const real interactionScale = 1.0f;
                    COMPUTE_INTERACTION
                    energy += tempEnergy;
                    force += tempForce;
                    torque += tempTorque1;
#ifdef ENABLE_SHUFFLE
                    shflForce -= tempForce;
                    shflTorque += tempTorque2;
                    SHUFFLE_WARP_DATA
                    shflTorque.x = real_shfl(shflTorque.x, tgx+1);
                    shflTorque.y = real_shfl(shflTorque.y, tgx+1);
                    shflTorque.z = real_shfl(shflTorque.z, tgx+1);
#else
                    localData[tbx+tj].fx -= tempForce.x;
                    localData[tbx+tj].fy -= tempForce.y;
                    localData[tbx+tj].fz -= tempForce.z;
                    localData[tbx+tj].tx += tempTorque.x;
                    localData[tbx+tj].ty += tempTorque.y;
                    localData[tbx+tj].tz += tempTorque.z;
#endif
                    tj = (tj + 1) & (TILE_SIZE - 1);
                }
            }

            // Write results.

            atomicAdd(&forceBuffers[atom1], static_cast<unsigned long long>((long long) (force.x*0x100000000)));
            atomicAdd(&forceBuffers[atom1+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (force.y*0x100000000)));
            atomicAdd(&forceBuffers[atom1+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (force.z*0x100000000)));
            atomicAdd(&torqueBuffers[atom1], static_cast<unsigned long long>((long long) (torque.x*0x100000000)));
            atomicAdd(&torqueBuffers[atom1+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (torque.y*0x100000000)));
            atomicAdd(&torqueBuffers[atom1+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (torque.z*0x100000000)));
#ifdef USE_CUTOFF
            unsigned int atom2 = atomIndices[threadIdx.x];
#else
            unsigned int atom2 = y*TILE_SIZE + tgx;
#endif
            if (atom2 < PADDED_NUM_ATOMS) {
#ifdef ENABLE_SHUFFLE
                atomicAdd(&forceBuffers[atom2], static_cast<unsigned long long>((long long) (shflForce.x*0x100000000)));
                atomicAdd(&forceBuffers[atom2+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (shflForce.y*0x100000000)));
                atomicAdd(&forceBuffers[atom2+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (shflForce.z*0x100000000)));
                atomicAdd(&torqueBuffers[atom2], static_cast<unsigned long long>((long long) (shflTorque.x*0x100000000)));
                atomicAdd(&torqueBuffers[atom2+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (shflTorque.y*0x100000000)));
                atomicAdd(&torqueBuffers[atom2+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (shflTorque.z*0x100000000)));
#else
                atomicAdd(&forceBuffers[atom2], static_cast<unsigned long long>((long long) (localData[threadIdx.x].fx*0x100000000)));
                atomicAdd(&forceBuffers[atom2+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].fy*0x100000000)));
                atomicAdd(&forceBuffers[atom2+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].fz*0x100000000)));
                atomicAdd(&torqueBuffers[atom2], static_cast<unsigned long long>((long long) (localData[threadIdx.x].tx*0x100000000)));
                atomicAdd(&torqueBuffers[atom2+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].ty*0x100000000)));
                atomicAdd(&torqueBuffers[atom2+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].tz*0x100000000)));
#endif
            }
        }
        pos++;
    }
#ifdef INCLUDE_ENERGY
    energyBuffer[blockIdx.x*blockDim.x+threadIdx.x] += energy;
#endif
}