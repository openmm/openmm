#include "hip/hip_runtime.h"
#define WARPS_PER_GROUP (THREAD_BLOCK_SIZE/TILE_SIZE)

#ifndef ENABLE_SHUFFLE
typedef struct {
    real x, y, z;
    real q;
    real fx, fy, fz;
    ATOM_PARAMETER_DATA
#ifndef PARAMETER_SIZE_IS_EVEN
    real padding;
#endif
} AtomData;
#endif

#ifdef ENABLE_SHUFFLE
//support for 64 bit shuffles
static __inline__ __device__ float real_shfl(float var, int srcLane) {
    return __shfl(var, srcLane);
}

static __inline__ __device__ double real_shfl(double var, int srcLane) {
    int hi, lo;
    asm volatile("mov.b64 { %0, %1 }, %2;" : "=r"(lo), "=r"(hi) : "d"(var));
    hi = __shfl(hi, srcLane);
    lo = __shfl(lo, srcLane);
    return __hiloint2double( hi, lo );
}

static __inline__ __device__ long long real_shfl(long long var, int srcLane) {
    int hi, lo;
    asm volatile("mov.b64 { %0, %1 }, %2;" : "=r"(lo), "=r"(hi) : "l"(var));
    hi = __shfl(hi, srcLane);
    lo = __shfl(lo, srcLane);
    // unforunately there isn't an __nv_hiloint2long(hi,lo) intrinsic cast
    int2 fuse; fuse.x = lo; fuse.y = hi;
    return *reinterpret_cast<long long*>(&fuse);
}
#endif

/**
 * Compute nonbonded interactions. The kernel is separated into two parts,
 * tiles with exclusions and tiles without exclusions. It relies heavily on 
 * implicit warp-level synchronization. A tile is defined by two atom blocks 
 * each of warpsize. Each warp computes a range of tiles.
 * 
 * Tiles with exclusions compute the entire set of interactions across
 * atom blocks, equal to warpsize*warpsize. In order to avoid access conflicts 
 * the forces are computed and accumulated diagonally in the manner shown below
 * where, suppose
 *
 * [a-h] comprise atom block 1, [i-p] comprise atom block 2
 *
 * 1 denotes the first set of calculations within the warp
 * 2 denotes the second set of calculations within the warp
 * ... etc.
 * 
 *        threads
 *     0 1 2 3 4 5 6 7
 *         atom1 
 * L    a b c d e f g h 
 * o  i 1 2 3 4 5 6 7 8
 * c  j 8 1 2 3 4 5 6 7
 * a  k 7 8 1 2 3 4 5 6
 * l  l 6 7 8 1 2 3 4 5
 * D  m 5 6 7 8 1 2 3 4 
 * a  n 4 5 6 7 8 1 2 3
 * t  o 3 4 5 6 7 8 1 2
 * a  p 2 3 4 5 6 7 8 1
 *
 * Tiles without exclusions read off directly from the neighbourlist interactingAtoms
 * and follows the same force accumulation method. If more there are more interactingTiles
 * than the size of the neighbourlist initially allocated, the neighbourlist is rebuilt
 * and the full tileset is computed. This should happen on the first step, and very rarely 
 * afterwards.
 *
 * On CUDA devices that support the shuffle intrinsic, on diagonal exclusion tiles use
 * __shfl to broadcast. For all other types of tiles __shfl is used to pass around the 
 * forces, positions, and parameters when computing the forces. 
 *
 * [out]forceBuffers    - forces on each atom to eventually be accumulated
 * [out]energyBuffer    - energyBuffer to eventually be accumulated
 * [in]posq             - x,y,z,charge 
 * [in]exclusions       - 1024-bit flags denoting atom-atom exclusions for each tile
 * [in]exclusionTiles   - x,y denotes the indices of tiles that have an exclusion
 * [in]startTileIndex   - index into first tile to be processed
 * [in]numTileIndices   - number of tiles this context is responsible for processing
 * [in]int tiles        - the atom block for each tile
 * [in]interactionCount - total number of tiles that have an interaction
 * [in]maxTiles         - stores the size of the neighbourlist in case it needs 
 *                      - to be expanded
 * [in]periodicBoxSize  - size of the Periodic Box, last dimension (w) not used
 * [in]invPeriodicBox   - inverse of the periodicBoxSize, pre-computed for speed
 * [in]blockCenter      - the center of each block in euclidean coordinates
 * [in]blockSize        - size of the each block, radiating from the center
 *                      - x is half the distance of total length
 *                      - y is half the distance of total width
 *                      - z is half the distance of total height
 *                      - w is not used
 * [in]interactingAtoms - a list of interactions within a given tile     
 *
 */
extern "C" __global__ void computeNonbonded(
        unsigned long long* __restrict__ forceBuffers, mixed* __restrict__ energyBuffer, const real4* __restrict__ posq, const tileflags* __restrict__ exclusions,
        const ushort2* __restrict__ exclusionTiles, unsigned int startTileIndex, unsigned int numTileIndices
#ifdef USE_CUTOFF
        , const int* __restrict__ tiles, const unsigned int* __restrict__ interactionCount, real4 periodicBoxSize, real4 invPeriodicBoxSize, 
        real4 periodicBoxVecX, real4 periodicBoxVecY, real4 periodicBoxVecZ, unsigned int maxTiles, const real4* __restrict__ blockCenter,
        const real4* __restrict__ blockSize, const unsigned int* __restrict__ interactingAtoms, unsigned int maxSinglePairs,
        const int2* __restrict__ singlePairs
#endif
        PARAMETER_ARGUMENTS) {
    const unsigned int totalWarps = (blockDim.x*gridDim.x)/TILE_SIZE;
    const unsigned int warp = (blockIdx.x*blockDim.x+threadIdx.x)/TILE_SIZE; // global warpIndex
    const unsigned int tgx = threadIdx.x & (TILE_SIZE-1); // index within the warp
    const unsigned int tbx = threadIdx.x - tgx;           // block warpIndex
    mixed energy = 0;
    INIT_DERIVATIVES
    // used shared memory if the device cannot shuffle
#ifndef ENABLE_SHUFFLE
    __shared__ AtomData localData[THREAD_BLOCK_SIZE];
#endif

    // First loop: process tiles that contain exclusions.

    const unsigned int firstExclusionTile = FIRST_EXCLUSION_TILE+warp*(LAST_EXCLUSION_TILE-FIRST_EXCLUSION_TILE)/totalWarps;
    const unsigned int lastExclusionTile = FIRST_EXCLUSION_TILE+(warp+1)*(LAST_EXCLUSION_TILE-FIRST_EXCLUSION_TILE)/totalWarps;
    for (int pos = firstExclusionTile; pos < lastExclusionTile; pos++) {
        const ushort2 tileIndices = exclusionTiles[pos];
        const unsigned int x = tileIndices.x;
        const unsigned int y = tileIndices.y;
        real3 force = make_real3(0);
        unsigned int atom1 = x*TILE_SIZE + tgx;
        real4 posq1 = posq[atom1];
        LOAD_ATOM1_PARAMETERS
#ifdef USE_EXCLUSIONS
        tileflags excl = exclusions[pos*TILE_SIZE+tgx];
#endif
        const bool hasExclusions = true;
        if (x == y) {
            // This tile is on the diagonal.
#ifdef ENABLE_SHUFFLE
            real4 shflPosq = posq1;
#else
            localData[threadIdx.x].x = posq1.x;
            localData[threadIdx.x].y = posq1.y;
            localData[threadIdx.x].z = posq1.z;
            localData[threadIdx.x].q = posq1.w;
            LOAD_LOCAL_PARAMETERS_FROM_1
#endif

            // we do not need to fetch parameters from global since this is a symmetric tile
            // instead we can broadcast the values using shuffle
            for (unsigned int j = 0; j < TILE_SIZE; j++) {
                int atom2 = tbx+j;
                real4 posq2;
#ifdef ENABLE_SHUFFLE
                BROADCAST_WARP_DATA
#else   
                posq2 = make_real4(localData[atom2].x, localData[atom2].y, localData[atom2].z, localData[atom2].q);
#endif
                real3 delta = make_real3(posq2.x-posq1.x, posq2.y-posq1.y, posq2.z-posq1.z);
#ifdef USE_PERIODIC
                APPLY_PERIODIC_TO_DELTA(delta)
#endif
                real r2 = delta.x*delta.x + delta.y*delta.y + delta.z*delta.z;
                real invR = RSQRT(r2);
                real r = r2*invR;
                LOAD_ATOM2_PARAMETERS
                atom2 = y*TILE_SIZE+j;
#ifdef USE_SYMMETRIC
                real dEdR = 0.0f;
#else
                real3 dEdR1 = make_real3(0);
                real3 dEdR2 = make_real3(0);
#endif
#ifdef USE_EXCLUSIONS
                bool isExcluded = (atom1 >= NUM_ATOMS || atom2 >= NUM_ATOMS || !(excl & 0x1));
#endif
                real tempEnergy = 0.0f;
                const real interactionScale = 0.5f;
                COMPUTE_INTERACTION
                energy += 0.5f*tempEnergy;
#ifdef INCLUDE_FORCES
#ifdef USE_SYMMETRIC
                force.x -= delta.x*dEdR;
                force.y -= delta.y*dEdR;
                force.z -= delta.z*dEdR;
#else
                force.x -= dEdR1.x;
                force.y -= dEdR1.y;
                force.z -= dEdR1.z;
#endif
#endif
#ifdef USE_EXCLUSIONS
                excl >>= 1;
#endif
            }
        }
        else {
            // This is an off-diagonal tile.
            unsigned int j = y*TILE_SIZE + tgx;
            real4 shflPosq = posq[j];
#ifdef ENABLE_SHUFFLE
            real3 shflForce;
            shflForce.x = 0.0f;
            shflForce.y = 0.0f;
            shflForce.z = 0.0f;
#else
            localData[threadIdx.x].x = shflPosq.x;
            localData[threadIdx.x].y = shflPosq.y;
            localData[threadIdx.x].z = shflPosq.z;
            localData[threadIdx.x].q = shflPosq.w;
            localData[threadIdx.x].fx = 0.0f;
            localData[threadIdx.x].fy = 0.0f;
            localData[threadIdx.x].fz = 0.0f;
#endif
            DECLARE_LOCAL_PARAMETERS
            LOAD_LOCAL_PARAMETERS_FROM_GLOBAL
#ifdef USE_EXCLUSIONS
            excl = (excl >> tgx) | (excl << (TILE_SIZE - tgx));
#endif
            unsigned int tj = tgx;
            for (j = 0; j < TILE_SIZE; j++) {
                int atom2 = tbx+tj;
#ifdef ENABLE_SHUFFLE
                real4 posq2 = shflPosq;
#else
                real4 posq2 = make_real4(localData[atom2].x, localData[atom2].y, localData[atom2].z, localData[atom2].q);
#endif
                real3 delta = make_real3(posq2.x-posq1.x, posq2.y-posq1.y, posq2.z-posq1.z);
#ifdef USE_PERIODIC
                APPLY_PERIODIC_TO_DELTA(delta)
#endif
                real r2 = delta.x*delta.x + delta.y*delta.y + delta.z*delta.z;
                real invR = RSQRT(r2);
                real r = r2*invR;
                LOAD_ATOM2_PARAMETERS
                atom2 = y*TILE_SIZE+tj;
#ifdef USE_SYMMETRIC
                real dEdR = 0.0f;
#else
                real3 dEdR1 = make_real3(0);
                real3 dEdR2 = make_real3(0);
#endif
#ifdef USE_EXCLUSIONS
                bool isExcluded = (atom1 >= NUM_ATOMS || atom2 >= NUM_ATOMS || !(excl & 0x1));
#endif
                real tempEnergy = 0.0f;
                const real interactionScale = 1.0f;
                COMPUTE_INTERACTION
                energy += tempEnergy;
#ifdef INCLUDE_FORCES
#ifdef USE_SYMMETRIC
                delta *= dEdR;
                force.x -= delta.x;
                force.y -= delta.y;
                force.z -= delta.z;
#ifdef ENABLE_SHUFFLE
                shflForce.x += delta.x;
                shflForce.y += delta.y;
                shflForce.z += delta.z;

#else
                localData[tbx+tj].fx += delta.x;
                localData[tbx+tj].fy += delta.y;
                localData[tbx+tj].fz += delta.z;
#endif
#else // !USE_SYMMETRIC
                force.x -= dEdR1.x;
                force.y -= dEdR1.y;
                force.z -= dEdR1.z;
#ifdef ENABLE_SHUFFLE
                shflForce.x += dEdR2.x;
                shflForce.y += dEdR2.y;
                shflForce.z += dEdR2.z;
#else
                localData[tbx+tj].fx += dEdR2.x;
                localData[tbx+tj].fy += dEdR2.y;
                localData[tbx+tj].fz += dEdR2.z;
#endif 
#endif // end USE_SYMMETRIC
#endif
#ifdef ENABLE_SHUFFLE
                SHUFFLE_WARP_DATA
#endif
#ifdef USE_EXCLUSIONS
                excl >>= 1;
#endif
                // cycles the indices
                // 0 1 2 3 4 5 6 7 -> 1 2 3 4 5 6 7 0
                tj = (tj + 1) & (TILE_SIZE - 1);
            }
            const unsigned int offset = y*TILE_SIZE + tgx;
            // write results for off diagonal tiles
#ifdef INCLUDE_FORCES
#ifdef ENABLE_SHUFFLE
            atomicAdd(&forceBuffers[offset], static_cast<unsigned long long>((long long) (shflForce.x*0x100000000)));
            atomicAdd(&forceBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (shflForce.y*0x100000000)));
            atomicAdd(&forceBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (shflForce.z*0x100000000)));
#else
            atomicAdd(&forceBuffers[offset], static_cast<unsigned long long>((long long) (localData[threadIdx.x].fx*0x100000000)));
            atomicAdd(&forceBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].fy*0x100000000)));
            atomicAdd(&forceBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].fz*0x100000000)));
#endif
#endif
        }
        // Write results for on and off diagonal tiles
#ifdef INCLUDE_FORCES
        const unsigned int offset = x*TILE_SIZE + tgx;
        atomicAdd(&forceBuffers[offset], static_cast<unsigned long long>((long long) (force.x*0x100000000)));
        atomicAdd(&forceBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (force.y*0x100000000)));
        atomicAdd(&forceBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (force.z*0x100000000)));
#endif
    }

    // Second loop: tiles without exclusions, either from the neighbor list (with cutoff) or just enumerating all
    // of them (no cutoff).

#ifdef USE_CUTOFF
    const unsigned int numTiles = interactionCount[0];
    if (numTiles > maxTiles)
        return; // There wasn't enough memory for the neighbor list.
    int pos = (int) (numTiles > maxTiles ? startTileIndex+warp*(long long)numTileIndices/totalWarps : warp*(long long)numTiles/totalWarps);
    int end = (int) (numTiles > maxTiles ? startTileIndex+(warp+1)*(long long)numTileIndices/totalWarps : (warp+1)*(long long)numTiles/totalWarps);
#else
    const unsigned int numTiles = numTileIndices;
    int pos = (int) (startTileIndex+warp*(long long)numTiles/totalWarps);
    int end = (int) (startTileIndex+(warp+1)*(long long)numTiles/totalWarps);
#endif
    int skipBase = 0;
    int currentSkipIndex = tbx;
    // atomIndices can probably be shuffled as well
    // but it probably wouldn't make things any faster
    __shared__ int atomIndices[THREAD_BLOCK_SIZE];
    __shared__ volatile int skipTiles[THREAD_BLOCK_SIZE];
    skipTiles[threadIdx.x] = -1;
    
    while (pos < end) {
        const bool hasExclusions = false;
        real3 force = make_real3(0);
        bool includeTile = true;

        // Extract the coordinates of this tile.
        int x, y;
        bool singlePeriodicCopy = false;
#ifdef USE_CUTOFF
        x = tiles[pos];
        real4 blockSizeX = blockSize[x];
        singlePeriodicCopy = (0.5f*periodicBoxSize.x-blockSizeX.x >= MAX_CUTOFF &&
                              0.5f*periodicBoxSize.y-blockSizeX.y >= MAX_CUTOFF &&
                              0.5f*periodicBoxSize.z-blockSizeX.z >= MAX_CUTOFF);
#else
        y = (int) floor(NUM_BLOCKS+0.5f-SQRT((NUM_BLOCKS+0.5f)*(NUM_BLOCKS+0.5f)-2*pos));
        x = (pos-y*NUM_BLOCKS+y*(y+1)/2);
        if (x < y || x >= NUM_BLOCKS) { // Occasionally happens due to roundoff error.
            y += (x < y ? -1 : 1);
            x = (pos-y*NUM_BLOCKS+y*(y+1)/2);
        }

        // Skip over tiles that have exclusions, since they were already processed.

        while (skipTiles[tbx+TILE_SIZE-1] < pos) {
            if (skipBase+tgx < NUM_TILES_WITH_EXCLUSIONS) {
                ushort2 tile = exclusionTiles[skipBase+tgx];
                skipTiles[threadIdx.x] = tile.x + tile.y*NUM_BLOCKS - tile.y*(tile.y+1)/2;
            }
            else
                skipTiles[threadIdx.x] = end;
            skipBase += TILE_SIZE;            
            currentSkipIndex = tbx;
        }
        while (skipTiles[currentSkipIndex] < pos)
            currentSkipIndex++;
        includeTile = (skipTiles[currentSkipIndex] != pos);
#endif
        if (includeTile) {
            unsigned int atom1 = x*TILE_SIZE + tgx;
            // Load atom data for this tile.
            real4 posq1 = posq[atom1];
            LOAD_ATOM1_PARAMETERS
            //const unsigned int localAtomIndex = threadIdx.x;
#ifdef USE_CUTOFF
            unsigned int j = interactingAtoms[pos*TILE_SIZE+tgx];
#else
            unsigned int j = y*TILE_SIZE + tgx;
#endif
            atomIndices[threadIdx.x] = j;
#ifdef ENABLE_SHUFFLE
            DECLARE_LOCAL_PARAMETERS
            real4 shflPosq;
            real3 shflForce;
            shflForce.x = 0.0f;
            shflForce.y = 0.0f;
            shflForce.z = 0.0f;
#endif
            if (j < PADDED_NUM_ATOMS) {
                // Load position of atom j from from global memory
#ifdef ENABLE_SHUFFLE
                shflPosq = posq[j];
#else
                localData[threadIdx.x].x = posq[j].x;
                localData[threadIdx.x].y = posq[j].y;
                localData[threadIdx.x].z = posq[j].z;
                localData[threadIdx.x].q = posq[j].w;
                localData[threadIdx.x].fx = 0.0f;
                localData[threadIdx.x].fy = 0.0f;
                localData[threadIdx.x].fz = 0.0f;
#endif                
                LOAD_LOCAL_PARAMETERS_FROM_GLOBAL
            }
            else {
#ifdef ENABLE_SHUFFLE
                shflPosq = make_real4(0, 0, 0, 0);
#else
                localData[threadIdx.x].x = 0;
                localData[threadIdx.x].y = 0;
                localData[threadIdx.x].z = 0;
#endif
            }
#ifdef USE_PERIODIC
            if (singlePeriodicCopy) {
                // The box is small enough that we can just translate all the atoms into a single periodic
                // box, then skip having to apply periodic boundary conditions later.
                real4 blockCenterX = blockCenter[x];
                APPLY_PERIODIC_TO_POS_WITH_CENTER(posq1, blockCenterX)
#ifdef ENABLE_SHUFFLE
                APPLY_PERIODIC_TO_POS_WITH_CENTER(shflPosq, blockCenterX)
#else
                APPLY_PERIODIC_TO_POS_WITH_CENTER(localData[threadIdx.x], blockCenterX)
#endif
                unsigned int tj = tgx;
                for (j = 0; j < TILE_SIZE; j++) {
                    int atom2 = tbx+tj;
#ifdef ENABLE_SHUFFLE
                    real4 posq2 = shflPosq; 
#else
                    real4 posq2 = make_real4(localData[atom2].x, localData[atom2].y, localData[atom2].z, localData[atom2].q);
#endif
                    real3 delta = make_real3(posq2.x-posq1.x, posq2.y-posq1.y, posq2.z-posq1.z);
                    real r2 = delta.x*delta.x + delta.y*delta.y + delta.z*delta.z;
                    real invR = RSQRT(r2);
                    real r = r2*invR;
                    LOAD_ATOM2_PARAMETERS
                    atom2 = atomIndices[tbx+tj];
#ifdef USE_SYMMETRIC
                    real dEdR = 0.0f;
#else
                    real3 dEdR1 = make_real3(0);
                    real3 dEdR2 = make_real3(0);
#endif
#ifdef USE_EXCLUSIONS
                    bool isExcluded = (atom1 >= NUM_ATOMS || atom2 >= NUM_ATOMS);
#endif
                    real tempEnergy = 0.0f;
                    const real interactionScale = 1.0f;
                    COMPUTE_INTERACTION
                    energy += tempEnergy;
#ifdef INCLUDE_FORCES
#ifdef USE_SYMMETRIC
                    delta *= dEdR;
                    force.x -= delta.x;
                    force.y -= delta.y;
                    force.z -= delta.z;
#ifdef ENABLE_SHUFFLE
                    shflForce.x += delta.x;
                    shflForce.y += delta.y;
                    shflForce.z += delta.z;

#else
                    localData[tbx+tj].fx += delta.x;
                    localData[tbx+tj].fy += delta.y;
                    localData[tbx+tj].fz += delta.z;
#endif
#else // !USE_SYMMETRIC
                    force.x -= dEdR1.x;
                    force.y -= dEdR1.y;
                    force.z -= dEdR1.z;
#ifdef ENABLE_SHUFFLE
                    shflForce.x += dEdR2.x;
                    shflForce.y += dEdR2.y;
                    shflForce.z += dEdR2.z;
#else
                    localData[tbx+tj].fx += dEdR2.x;
                    localData[tbx+tj].fy += dEdR2.y;
                    localData[tbx+tj].fz += dEdR2.z;
#endif 
#endif // end USE_SYMMETRIC
#endif
#ifdef ENABLE_SHUFFLE
                    SHUFFLE_WARP_DATA
#endif
                    tj = (tj + 1) & (TILE_SIZE - 1);
                }
            }
            else
#endif
            {
                // We need to apply periodic boundary conditions separately for each interaction.
                unsigned int tj = tgx;
                for (j = 0; j < TILE_SIZE; j++) {
                    int atom2 = tbx+tj;
#ifdef ENABLE_SHUFFLE
                    real4 posq2 = shflPosq;
#else
                    real4 posq2 = make_real4(localData[atom2].x, localData[atom2].y, localData[atom2].z, localData[atom2].q);
#endif
                    real3 delta = make_real3(posq2.x-posq1.x, posq2.y-posq1.y, posq2.z-posq1.z);
#ifdef USE_PERIODIC
                    APPLY_PERIODIC_TO_DELTA(delta)
#endif
                    real r2 = delta.x*delta.x + delta.y*delta.y + delta.z*delta.z;
                    real invR = RSQRT(r2);
                    real r = r2*invR;
                    LOAD_ATOM2_PARAMETERS
                    atom2 = atomIndices[tbx+tj];
#ifdef USE_SYMMETRIC
                    real dEdR = 0.0f;
#else
                    real3 dEdR1 = make_real3(0);
                    real3 dEdR2 = make_real3(0);
#endif
#ifdef USE_EXCLUSIONS
                    bool isExcluded = (atom1 >= NUM_ATOMS || atom2 >= NUM_ATOMS);
#endif
                    real tempEnergy = 0.0f;
                    const real interactionScale = 1.0f;
                    COMPUTE_INTERACTION
                    energy += tempEnergy;
#ifdef INCLUDE_FORCES
#ifdef USE_SYMMETRIC
                    delta *= dEdR;
                    force.x -= delta.x;
                    force.y -= delta.y;
                    force.z -= delta.z;
#ifdef ENABLE_SHUFFLE
                    shflForce.x += delta.x;
                    shflForce.y += delta.y;
                    shflForce.z += delta.z;

#else
                    localData[tbx+tj].fx += delta.x;
                    localData[tbx+tj].fy += delta.y;
                    localData[tbx+tj].fz += delta.z;
#endif
#else // !USE_SYMMETRIC
                    force.x -= dEdR1.x;
                    force.y -= dEdR1.y;
                    force.z -= dEdR1.z;
#ifdef ENABLE_SHUFFLE
                    shflForce.x += dEdR2.x;
                    shflForce.y += dEdR2.y;
                    shflForce.z += dEdR2.z;
#else
                    localData[tbx+tj].fx += dEdR2.x;
                    localData[tbx+tj].fy += dEdR2.y;
                    localData[tbx+tj].fz += dEdR2.z;
#endif 
#endif // end USE_SYMMETRIC
#endif
#ifdef ENABLE_SHUFFLE
                    SHUFFLE_WARP_DATA
#endif
                    tj = (tj + 1) & (TILE_SIZE - 1);
                }
            }

            // Write results.
#ifdef INCLUDE_FORCES
            atomicAdd(&forceBuffers[atom1], static_cast<unsigned long long>((long long) (force.x*0x100000000)));
            atomicAdd(&forceBuffers[atom1+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (force.y*0x100000000)));
            atomicAdd(&forceBuffers[atom1+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (force.z*0x100000000)));
#ifdef USE_CUTOFF
            unsigned int atom2 = atomIndices[threadIdx.x];
#else
            unsigned int atom2 = y*TILE_SIZE + tgx;
#endif
            if (atom2 < PADDED_NUM_ATOMS) {
#ifdef ENABLE_SHUFFLE
                atomicAdd(&forceBuffers[atom2], static_cast<unsigned long long>((long long) (shflForce.x*0x100000000)));
                atomicAdd(&forceBuffers[atom2+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (shflForce.y*0x100000000)));
                atomicAdd(&forceBuffers[atom2+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (shflForce.z*0x100000000)));
#else
                atomicAdd(&forceBuffers[atom2], static_cast<unsigned long long>((long long) (localData[threadIdx.x].fx*0x100000000)));
                atomicAdd(&forceBuffers[atom2+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].fy*0x100000000)));
                atomicAdd(&forceBuffers[atom2+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].fz*0x100000000)));
#endif
            }
#endif
        }
        pos++;
    }
    
    // Third loop: single pairs that aren't part of a tile.
    
#if USE_CUTOFF
    const unsigned int numPairs = interactionCount[1];
    if (numPairs > maxSinglePairs)
        return; // There wasn't enough memory for the neighbor list.
    for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < numPairs; i += blockDim.x*gridDim.x) {
        int2 pair = singlePairs[i];
        int atom1 = pair.x;
        int atom2 = pair.y;
        real4 posq1 = posq[atom1];
        real4 posq2 = posq[atom2];
        LOAD_ATOM1_PARAMETERS
        int j = atom2;
atom2 = threadIdx.x;
        DECLARE_LOCAL_PARAMETERS
        LOAD_LOCAL_PARAMETERS_FROM_GLOBAL
        LOAD_ATOM2_PARAMETERS
atom2 = pair.y;
        real3 delta = make_real3(posq2.x-posq1.x, posq2.y-posq1.y, posq2.z-posq1.z);
#ifdef USE_PERIODIC
        APPLY_PERIODIC_TO_DELTA(delta)
#endif
        real r2 = delta.x*delta.x + delta.y*delta.y + delta.z*delta.z;
        real invR = RSQRT(r2);
        real r = r2*invR;
#ifdef USE_SYMMETRIC
        real dEdR = 0.0f;
#else
        real3 dEdR1 = make_real3(0);
        real3 dEdR2 = make_real3(0);
#endif
        bool hasExclusions = false;
        bool isExcluded = false;
        real tempEnergy = 0.0f;
        const real interactionScale = 1.0f;
        COMPUTE_INTERACTION
        energy += tempEnergy;
#ifdef INCLUDE_FORCES
#ifdef USE_SYMMETRIC
        real3 dEdR1 = delta*dEdR;
        real3 dEdR2 = -dEdR1;
#endif
        atomicAdd(&forceBuffers[atom1], static_cast<unsigned long long>((long long) (-dEdR1.x*0x100000000)));
        atomicAdd(&forceBuffers[atom1+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (-dEdR1.y*0x100000000)));
        atomicAdd(&forceBuffers[atom1+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (-dEdR1.z*0x100000000)));
        atomicAdd(&forceBuffers[atom2], static_cast<unsigned long long>((long long) (-dEdR2.x*0x100000000)));
        atomicAdd(&forceBuffers[atom2+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (-dEdR2.y*0x100000000)));
        atomicAdd(&forceBuffers[atom2+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (-dEdR2.z*0x100000000)));
#endif
    }
#endif
#ifdef INCLUDE_ENERGY
    energyBuffer[blockIdx.x*blockDim.x+threadIdx.x] += energy;
#endif
    SAVE_DERIVATIVES
}