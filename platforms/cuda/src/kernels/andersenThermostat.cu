#include "hip/hip_runtime.h"
/**
 * Apply the Andersen thermostat to adjust particle velocities.
 */

extern "C" __global__ void applyAndersenThermostat(int numAtoms, float collisionFrequency, float kT, mixed4* velm, const mixed4* __restrict__ stepSize, const float4* __restrict__ random,
        unsigned int randomIndex, const int* __restrict__ atomGroups) {
    float collisionProbability = 1.0f-expf(-(float) (collisionFrequency*stepSize[0].y));
    float randomRange = erff(collisionProbability/sqrtf(2.0f));
    for (int index = blockIdx.x*blockDim.x+threadIdx.x; index < numAtoms; index += blockDim.x*gridDim.x) {
        mixed4 velocity = velm[index];
        float4 selectRand = random[randomIndex+atomGroups[index]];
        float4 velRand = random[randomIndex+index];
        real scale = (selectRand.w > -randomRange && selectRand.w < randomRange ? 0 : 1);
        real add = (1-scale)*SQRT(kT*velocity.w);
        velocity.x = scale*velocity.x + add*velRand.x;
        velocity.y = scale*velocity.y + add*velRand.y;
        velocity.z = scale*velocity.z + add*velRand.z;
        velm[index] = velocity;
    }
}
