
#include <hip/hip_runtime.h>
/**
 * This file defines vector operations to simplify code elsewhere.
 */

// Versions of make_x() that take a single value and set all components to that.

inline __device__ int2 make_int2(int a) {
    return make_int2(a, a);
}

inline __device__ int3 make_int3(int a) {
    return make_int3(a, a, a);
}

inline __device__ int4 make_int4(int a) {
    return make_int4(a, a, a, a);
}

inline __device__ float2 make_float2(float a) {
    return make_float2(a, a);
}

inline __device__ float3 make_float3(float a) {
    return make_float3(a, a, a);
}

inline __device__ float4 make_float4(float a) {
    return make_float4(a, a, a, a);
}

inline __device__ double2 make_double2(double a) {
    return make_double2(a, a);
}

inline __device__ double3 make_double3(double a) {
    return make_double3(a, a, a);
}

inline __device__ double4 make_double4(double a) {
    return make_double4(a, a, a, a);
}

// Negate a vector.

inline __device__ int2 operator-(int2 a) {
    return make_int2(-a.x, -a.y);
}

inline __device__ int3 operator-(int3 a) {
    return make_int3(-a.x, -a.y, -a.z);
}

inline __device__ int4 operator-(int4 a) {
    return make_int4(-a.x, -a.y, -a.z, -a.w);
}

inline __device__ float2 operator-(float2 a) {
    return make_float2(-a.x, -a.y);
}

inline __device__ float3 operator-(float3 a) {
    return make_float3(-a.x, -a.y, -a.z);
}

inline __device__ float4 operator-(float4 a) {
    return make_float4(-a.x, -a.y, -a.z, -a.w);
}

inline __device__ double2 operator-(double2 a) {
    return make_double2(-a.x, -a.y);
}

inline __device__ double3 operator-(double3 a) {
    return make_double3(-a.x, -a.y, -a.z);
}

inline __device__ double4 operator-(double4 a) {
    return make_double4(-a.x, -a.y, -a.z, -a.w);
}

// Add two vectors.

inline __device__ int2 operator+(int2 a, int2 b) {
    return make_int2(a.x+b.x, a.y+b.y);
}

inline __device__ int3 operator+(int3 a, int3 b) {
    return make_int3(a.x+b.x, a.y+b.y, a.z+b.z);
}

inline __device__ int4 operator+(int4 a, int4 b) {
    return make_int4(a.x+b.x, a.y+b.y, a.z+b.z, a.w+b.w);
}

inline __device__ float2 operator+(float2 a, float2 b) {
    return make_float2(a.x+b.x, a.y+b.y);
}

inline __device__ float3 operator+(float3 a, float3 b) {
    return make_float3(a.x+b.x, a.y+b.y, a.z+b.z);
}

inline __device__ float4 operator+(float4 a, float4 b) {
    return make_float4(a.x+b.x, a.y+b.y, a.z+b.z, a.w+b.w);
}

inline __device__ double2 operator+(double2 a, double2 b) {
    return make_double2(a.x+b.x, a.y+b.y);
}

inline __device__ double3 operator+(double3 a, double3 b) {
    return make_double3(a.x+b.x, a.y+b.y, a.z+b.z);
}

inline __device__ double4 operator+(double4 a, double4 b) {
    return make_double4(a.x+b.x, a.y+b.y, a.z+b.z, a.w+b.w);
}

// Subtract two vectors.

inline __device__ int2 operator-(int2 a, int2 b) {
    return make_int2(a.x-b.x, a.y-b.y);
}

inline __device__ int3 operator-(int3 a, int3 b) {
    return make_int3(a.x-b.x, a.y-b.y, a.z-b.z);
}

inline __device__ int4 operator-(int4 a, int4 b) {
    return make_int4(a.x-b.x, a.y-b.y, a.z-b.z, a.w-b.w);
}

inline __device__ float2 operator-(float2 a, float2 b) {
    return make_float2(a.x-b.x, a.y-b.y);
}

inline __device__ float3 operator-(float3 a, float3 b) {
    return make_float3(a.x-b.x, a.y-b.y, a.z-b.z);
}

inline __device__ float4 operator-(float4 a, float4 b) {
    return make_float4(a.x-b.x, a.y-b.y, a.z-b.z, a.w-b.w);
}

inline __device__ double2 operator-(double2 a, double2 b) {
    return make_double2(a.x-b.x, a.y-b.y);
}

inline __device__ double3 operator-(double3 a, double3 b) {
    return make_double3(a.x-b.x, a.y-b.y, a.z-b.z);
}

inline __device__ double4 operator-(double4 a, double4 b) {
    return make_double4(a.x-b.x, a.y-b.y, a.z-b.z, a.w-b.w);
}

// Multiply two vectors.

inline __device__ int2 operator*(int2 a, int2 b) {
    return make_int2(a.x*b.x, a.y*b.y);
}

inline __device__ int3 operator*(int3 a, int3 b) {
    return make_int3(a.x*b.x, a.y*b.y, a.z*b.z);
}

inline __device__ int4 operator*(int4 a, int4 b) {
    return make_int4(a.x*b.x, a.y*b.y, a.z*b.z, a.w*b.w);
}

inline __device__ float2 operator*(float2 a, float2 b) {
    return make_float2(a.x*b.x, a.y*b.y);
}

inline __device__ float3 operator*(float3 a, float3 b) {
    return make_float3(a.x*b.x, a.y*b.y, a.z*b.z);
}

inline __device__ float4 operator*(float4 a, float4 b) {
    return make_float4(a.x*b.x, a.y*b.y, a.z*b.z, a.w*b.w);
}

inline __device__ double2 operator*(double2 a, double2 b) {
    return make_double2(a.x*b.x, a.y*b.y);
}

inline __device__ double3 operator*(double3 a, double3 b) {
    return make_double3(a.x*b.x, a.y*b.y, a.z*b.z);
}

inline __device__ double4 operator*(double4 a, double4 b) {
    return make_double4(a.x*b.x, a.y*b.y, a.z*b.z, a.w*b.w);
}

// Divide two vectors.

inline __device__ int2 operator/(int2 a, int2 b) {
    return make_int2(a.x/b.x, a.y/b.y);
}

inline __device__ int3 operator/(int3 a, int3 b) {
    return make_int3(a.x/b.x, a.y/b.y, a.z/b.z);
}

inline __device__ int4 operator/(int4 a, int4 b) {
    return make_int4(a.x/b.x, a.y/b.y, a.z/b.z, a.w/b.w);
}

inline __device__ float2 operator/(float2 a, float2 b) {
    return make_float2(a.x/b.x, a.y/b.y);
}

inline __device__ float3 operator/(float3 a, float3 b) {
    return make_float3(a.x/b.x, a.y/b.y, a.z/b.z);
}

inline __device__ float4 operator/(float4 a, float4 b) {
    return make_float4(a.x/b.x, a.y/b.y, a.z/b.z, a.w/b.w);
}

inline __device__ double2 operator/(double2 a, double2 b) {
    return make_double2(a.x/b.x, a.y/b.y);
}

inline __device__ double3 operator/(double3 a, double3 b) {
    return make_double3(a.x/b.x, a.y/b.y, a.z/b.z);
}

inline __device__ double4 operator/(double4 a, double4 b) {
    return make_double4(a.x/b.x, a.y/b.y, a.z/b.z, a.w/b.w);
}

// += operator

inline __device__ void operator+=(int2& a, int2 b) {
    a.x += b.x; a.y += b.y;
}

inline __device__ void operator+=(int3& a, int3 b) {
    a.x += b.x; a.y += b.y; a.z += b.z;
}

inline __device__ void operator+=(int4& a, int4 b) {
    a.x += b.x; a.y += b.y; a.z += b.z; a.w += b.w;
}

inline __device__ void operator+=(float2& a, float2 b) {
    a.x += b.x; a.y += b.y;
}

inline __device__ void operator+=(float3& a, float3 b) {
    a.x += b.x; a.y += b.y; a.z += b.z;
}

inline __device__ void operator+=(float4& a, float4 b) {
    a.x += b.x; a.y += b.y; a.z += b.z; a.w += b.w;
}

inline __device__ void operator+=(double2& a, double2 b) {
    a.x += b.x; a.y += b.y;
}

inline __device__ void operator+=(double3& a, double3 b) {
    a.x += b.x; a.y += b.y; a.z += b.z;
}

inline __device__ void operator+=(double4& a, double4 b) {
    a.x += b.x; a.y += b.y; a.z += b.z; a.w += b.w;
}

// -= operator

inline __device__ void operator-=(int2& a, int2 b) {
    a.x -= b.x; a.y -= b.y;
}

inline __device__ void operator-=(int3& a, int3 b) {
    a.x -= b.x; a.y -= b.y; a.z -= b.z;
}

inline __device__ void operator-=(int4& a, int4 b) {
    a.x -= b.x; a.y -= b.y; a.z -= b.z; a.w -= b.w;
}

inline __device__ void operator-=(float2& a, float2 b) {
    a.x -= b.x; a.y -= b.y;
}

inline __device__ void operator-=(float3& a, float3 b) {
    a.x -= b.x; a.y -= b.y; a.z -= b.z;
}

inline __device__ void operator-=(float4& a, float4 b) {
    a.x -= b.x; a.y -= b.y; a.z -= b.z; a.w -= b.w;
}

inline __device__ void operator-=(double2& a, double2 b) {
    a.x -= b.x; a.y -= b.y;
}

inline __device__ void operator-=(double3& a, double3 b) {
    a.x -= b.x; a.y -= b.y; a.z -= b.z;
}

inline __device__ void operator-=(double4& a, double4 b) {
    a.x -= b.x; a.y -= b.y; a.z -= b.z; a.w -= b.w;
}

// *= operator

inline __device__ void operator*=(int2& a, int2 b) {
    a.x *= b.x; a.y *= b.y;
}

inline __device__ void operator*=(int3& a, int3 b) {
    a.x *= b.x; a.y *= b.y; a.z *= b.z;
}

inline __device__ void operator*=(int4& a, int4 b) {
    a.x *= b.x; a.y *= b.y; a.z *= b.z; a.w *= b.w;
}

inline __device__ void operator*=(float2& a, float2 b) {
    a.x *= b.x; a.y *= b.y;
}

inline __device__ void operator*=(float3& a, float3 b) {
    a.x *= b.x; a.y *= b.y; a.z *= b.z;
}

inline __device__ void operator*=(float4& a, float4 b) {
    a.x *= b.x; a.y *= b.y; a.z *= b.z; a.w *= b.w;
}

inline __device__ void operator*=(double2& a, double2 b) {
    a.x *= b.x; a.y *= b.y;
}

inline __device__ void operator*=(double3& a, double3 b) {
    a.x *= b.x; a.y *= b.y; a.z *= b.z;
}

inline __device__ void operator*=(double4& a, double4 b) {
    a.x *= b.x; a.y *= b.y; a.z *= b.z; a.w *= b.w;
}

// /= operator

inline __device__ void operator/=(int2& a, int2 b) {
    a.x /= b.x; a.y /= b.y;
}

inline __device__ void operator/=(int3& a, int3 b) {
    a.x /= b.x; a.y /= b.y; a.z /= b.z;
}

inline __device__ void operator/=(int4& a, int4 b) {
    a.x /= b.x; a.y /= b.y; a.z /= b.z; a.w /= b.w;
}

inline __device__ void operator/=(float2& a, float2 b) {
    a.x /= b.x; a.y /= b.y;
}

inline __device__ void operator/=(float3& a, float3 b) {
    a.x /= b.x; a.y /= b.y; a.z /= b.z;
}

inline __device__ void operator/=(float4& a, float4 b) {
    a.x /= b.x; a.y /= b.y; a.z /= b.z; a.w /= b.w;
}

inline __device__ void operator/=(double2& a, double2 b) {
    a.x /= b.x; a.y /= b.y;
}

inline __device__ void operator/=(double3& a, double3 b) {
    a.x /= b.x; a.y /= b.y; a.z /= b.z;
}

inline __device__ void operator/=(double4& a, double4 b) {
    a.x /= b.x; a.y /= b.y; a.z /= b.z; a.w /= b.w;
}

// Multiply a vector by a constant.

inline __device__ int2 operator*(int2 a, int b) {
    return make_int2(a.x*b, a.y*b);
}

inline __device__ int3 operator*(int3 a, int b) {
    return make_int3(a.x*b, a.y*b, a.z*b);
}

inline __device__ int4 operator*(int4 a, int b) {
    return make_int4(a.x*b, a.y*b, a.z*b, a.w*b);
}

inline __device__ int2 operator*(int a, int2 b) {
    return make_int2(a*b.x, a*b.y);
}

inline __device__ int3 operator*(int a, int3 b) {
    return make_int3(a*b.x, a*b.y, a*b.z);
}

inline __device__ int4 operator*(int a, int4 b) {
    return make_int4(a*b.x, a*b.y, a*b.z, a*b.w);
}

inline __device__ float2 operator*(float2 a, float b) {
    return make_float2(a.x*b, a.y*b);
}

inline __device__ float3 operator*(float3 a, float b) {
    return make_float3(a.x*b, a.y*b, a.z*b);
}

inline __device__ float4 operator*(float4 a, float b) {
    return make_float4(a.x*b, a.y*b, a.z*b, a.w*b);
}

inline __device__ float2 operator*(float a, float2 b) {
    return make_float2(a*b.x, a*b.y);
}

inline __device__ float3 operator*(float a, float3 b) {
    return make_float3(a*b.x, a*b.y, a*b.z);
}

inline __device__ float4 operator*(float a, float4 b) {
    return make_float4(a*b.x, a*b.y, a*b.z, a*b.w);
}

inline __device__ double2 operator*(double2 a, double b) {
    return make_double2(a.x*b, a.y*b);
}

inline __device__ double3 operator*(double3 a, double b) {
    return make_double3(a.x*b, a.y*b, a.z*b);
}

inline __device__ double4 operator*(double4 a, double b) {
    return make_double4(a.x*b, a.y*b, a.z*b, a.w*b);
}

inline __device__ double2 operator*(double a, double2 b) {
    return make_double2(a*b.x, a*b.y);
}

inline __device__ double3 operator*(double a, double3 b) {
    return make_double3(a*b.x, a*b.y, a*b.z);
}

inline __device__ double4 operator*(double a, double4 b) {
    return make_double4(a*b.x, a*b.y, a*b.z, a*b.w);
}

// Divide a vector by a constant.

inline __device__ int2 operator/(int2 a, int b) {
    return make_int2(a.x/b, a.y/b);
}

inline __device__ int3 operator/(int3 a, int b) {
    return make_int3(a.x/b, a.y/b, a.z/b);
}

inline __device__ int4 operator/(int4 a, int b) {
    return make_int4(a.x/b, a.y/b, a.z/b, a.w/b);
}

inline __device__ float2 operator/(float2 a, float b) {
    float scale = 1.0f/b;
    return a*scale;
}

inline __device__ float3 operator/(float3 a, float b) {
    float scale = 1.0f/b;
    return a*scale;
}

inline __device__ float4 operator/(float4 a, float b) {
    float scale = 1.0f/b;
    return a*scale;
}

inline __device__ double2 operator/(double2 a, double b) {
    double scale = 1.0/b;
    return a*scale;
}

inline __device__ double3 operator/(double3 a, double b) {
    double scale = 1.0/b;
    return a*scale;
}

inline __device__ double4 operator/(double4 a, double b) {
    double scale = 1.0/b;
    return a*scale;
}

// *= operator (multiply vector by constant)

inline __device__ void operator*=(int2& a, int b) {
    a.x *= b; a.y *= b;
}

inline __device__ void operator*=(int3& a, int b) {
    a.x *= b; a.y *= b; a.z *= b;
}

inline __device__ void operator*=(int4& a, int b) {
    a.x *= b; a.y *= b; a.z *= b; a.w *= b;
}

inline __device__ void operator*=(float2& a, float b) {
    a.x *= b; a.y *= b;
}

inline __device__ void operator*=(float3& a, float b) {
    a.x *= b; a.y *= b; a.z *= b;
}

inline __device__ void operator*=(float4& a, float b) {
    a.x *= b; a.y *= b; a.z *= b; a.w *= b;
}

inline __device__ void operator*=(double2& a, double b) {
    a.x *= b; a.y *= b;
}

inline __device__ void operator*=(double3& a, double b) {
    a.x *= b; a.y *= b; a.z *= b;
}

inline __device__ void operator*=(double4& a, double b) {
    a.x *= b; a.y *= b; a.z *= b; a.w *= b;
}

// Dot product

inline __device__ float dot(float3 a, float3 b) {
    return a.x*b.x+a.y*b.y+a.z*b.z;
}

inline __device__ double dot(double3 a, double3 b) {
    return a.x*b.x+a.y*b.y+a.z*b.z;
}

// Cross product

inline __device__ float3 cross(float3 a, float3 b) {
    return make_float3(a.y*b.z-a.z*b.y, a.z*b.x-a.x*b.z, a.x*b.y-a.y*b.x);
}

inline __device__ float4 cross(float4 a, float4 b) {
    return make_float4(a.y*b.z-a.z*b.y, a.z*b.x-a.x*b.z, a.x*b.y-a.y*b.x, 0.0f);
}

inline __device__ double3 cross(double3 a, double3 b) {
    return make_double3(a.y*b.z-a.z*b.y, a.z*b.x-a.x*b.z, a.x*b.y-a.y*b.x);
}

inline __device__ double4 cross(double4 a, double4 b) {
    return make_double4(a.y*b.z-a.z*b.y, a.z*b.x-a.x*b.z, a.x*b.y-a.y*b.x, 0.0);
}

// Normalize a vector

inline __device__ float2 normalize(float2 a) {
    return a*rsqrtf(a.x*a.x+a.y*a.y);
}

inline __device__ float3 normalize(float3 a) {
    return a*rsqrtf(a.x*a.x+a.y*a.y+a.z*a.z);
}

inline __device__ float4 normalize(float4 a) {
    return a*rsqrtf(a.x*a.x+a.y*a.y+a.z*a.z+a.w*a.w);
}

inline __device__ double2 normalize(double2 a) {
    return a*rsqrt(a.x*a.x+a.y*a.y);
}

inline __device__ double3 normalize(double3 a) {
    return a*rsqrt(a.x*a.x+a.y*a.y+a.z*a.z);
}

inline __device__ double4 normalize(double4 a) {
    return a*rsqrt(a.x*a.x+a.y*a.y+a.z*a.z+a.w*a.w);
}

// Strip off the fourth component of a vector.

inline __device__ short3 trimTo3(short4 v) {
    return make_short3(v.x, v.y, v.z);
}

inline __device__ int3 trimTo3(int4 v) {
    return make_int3(v.x, v.y, v.z);
}

inline __device__ float3 trimTo3(float4 v) {
    return make_float3(v.x, v.y, v.z);
}

inline __device__ double3 trimTo3(double4 v) {
    return make_double3(v.x, v.y, v.z);
}
