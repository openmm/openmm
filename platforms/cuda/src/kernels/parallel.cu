
#include <hip/hip_runtime.h>
/**
 * Sum the forces computed by different contexts.
 */

extern "C" __global__ void sumForces(long long* __restrict__ force, long long* __restrict__ buffer, int bufferSize, int numBuffers) {
    long long totalSize = bufferSize*numBuffers;
    for (int index = blockDim.x*blockIdx.x+threadIdx.x; index < bufferSize; index += blockDim.x*gridDim.x) {
        long long sum = force[index];
        for (long long i = index; i < totalSize; i += bufferSize)
            sum += buffer[i];
        force[index] = sum;
    }
}
