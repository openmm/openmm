#include "hip/hip_runtime.h"
typedef struct {
    real3 pos;
    real value;
    ATOM_PARAMETER_DATA
#ifdef NEED_PADDING
    float padding;
#endif
} AtomData;

/**
 * Compute a value based on pair interactions.
 */
extern "C" __global__ void computeN2Value(const real4* __restrict__ posq, const unsigned int* __restrict__ exclusions,
        const ushort2* __restrict__ exclusionTiles, unsigned long long* __restrict__ global_value,
#ifdef USE_CUTOFF
        const int* __restrict__ tiles, const unsigned int* __restrict__ interactionCount, real4 periodicBoxSize, real4 invPeriodicBoxSize,
        real4 periodicBoxVecX, real4 periodicBoxVecY, real4 periodicBoxVecZ, unsigned int maxTiles, const real4* __restrict__ blockCenter,
        const real4* __restrict__ blockSize, const unsigned int* __restrict__ interactingAtoms
#else
        unsigned int numTiles
#endif
        PARAMETER_ARGUMENTS) {
    const unsigned int totalWarps = (blockDim.x*gridDim.x)/TILE_SIZE;
    const unsigned int warp = (blockIdx.x*blockDim.x+threadIdx.x)/TILE_SIZE;
    const unsigned int tgx = threadIdx.x & (TILE_SIZE-1);
    const unsigned int tbx = threadIdx.x - tgx;
    __shared__ AtomData localData[THREAD_BLOCK_SIZE];

    // First loop: process tiles that contain exclusions.
    
    const unsigned int firstExclusionTile = FIRST_EXCLUSION_TILE+warp*(LAST_EXCLUSION_TILE-FIRST_EXCLUSION_TILE)/totalWarps;
    const unsigned int lastExclusionTile = FIRST_EXCLUSION_TILE+(warp+1)*(LAST_EXCLUSION_TILE-FIRST_EXCLUSION_TILE)/totalWarps;
    for (int pos = firstExclusionTile; pos < lastExclusionTile; pos++) {
        const ushort2 tileIndices = exclusionTiles[pos];
        const unsigned int x = tileIndices.x;
        const unsigned int y = tileIndices.y;
        real value = 0;
        unsigned int atom1 = x*TILE_SIZE + tgx;
        real4 pos1 = posq[atom1];
        LOAD_ATOM1_PARAMETERS
#ifdef USE_EXCLUSIONS
        unsigned int excl = exclusions[pos*TILE_SIZE+tgx];
#endif
        if (x == y) {
            // This tile is on the diagonal.

            const unsigned int localAtomIndex = threadIdx.x;
            localData[localAtomIndex].pos = make_real3(pos1.x, pos1.y, pos1.z);
            LOAD_LOCAL_PARAMETERS_FROM_1
            for (unsigned int j = 0; j < TILE_SIZE; j++) {
                int atom2 = tbx+j;
                real3 pos2 = localData[atom2].pos;
                real3 delta = make_real3(pos2.x-pos1.x, pos2.y-pos1.y, pos2.z-pos1.z);
#ifdef USE_PERIODIC
                APPLY_PERIODIC_TO_DELTA(delta)
#endif
                real r2 = delta.x*delta.x + delta.y*delta.y + delta.z*delta.z;
#ifdef USE_CUTOFF
                if (r2 < CUTOFF_SQUARED) {
#endif
                    real invR = RSQRT(r2);
                    real r = r2*invR;
                    LOAD_ATOM2_PARAMETERS
                    atom2 = y*TILE_SIZE+j;
                    real tempValue1 = 0;
                    real tempValue2 = 0;
#ifdef USE_EXCLUSIONS
                    bool isExcluded = (atom1 >= NUM_ATOMS || atom2 >= NUM_ATOMS || !(excl & 0x1));
                    if (!isExcluded && atom1 != atom2) {
#else
                    if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS && atom1 != atom2) {
#endif
                        COMPUTE_VALUE
                    }
                    value += tempValue1;
#ifdef USE_CUTOFF
                }
#endif
#ifdef USE_EXCLUSIONS
                excl >>= 1;
#endif
            }
        }
        else {
            // This is an off-diagonal tile.

            const unsigned int localAtomIndex = threadIdx.x;
            unsigned int j = y*TILE_SIZE + tgx;
            real4 tempPosq = posq[j];
            localData[localAtomIndex].pos = make_real3(tempPosq.x, tempPosq.y, tempPosq.z);
            LOAD_LOCAL_PARAMETERS_FROM_GLOBAL
            localData[localAtomIndex].value = 0;
#ifdef USE_EXCLUSIONS
            excl = (excl >> tgx) | (excl << (TILE_SIZE - tgx));
#endif
            unsigned int tj = tgx;
            for (j = 0; j < TILE_SIZE; j++) {
                int atom2 = tbx+tj;
                real3 pos2 = localData[atom2].pos;
                real3 delta = make_real3(pos2.x-pos1.x, pos2.y-pos1.y, pos2.z-pos1.z);
#ifdef USE_PERIODIC
                APPLY_PERIODIC_TO_DELTA(delta)
#endif
                real r2 = delta.x*delta.x + delta.y*delta.y + delta.z*delta.z;
#ifdef USE_CUTOFF
                if (r2 < CUTOFF_SQUARED) {
#endif
                    real invR = RSQRT(r2);
                    real r = r2*invR;
                    LOAD_ATOM2_PARAMETERS
                    atom2 = y*TILE_SIZE+tj;
                    real tempValue1 = 0;
                    real tempValue2 = 0;
#ifdef USE_EXCLUSIONS
                    bool isExcluded = (atom1 >= NUM_ATOMS || atom2 >= NUM_ATOMS || !(excl & 0x1));
                    if (!isExcluded) {
#else
                    if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS) {
#endif
                        COMPUTE_VALUE
                    }
                    value += tempValue1;
                    localData[tbx+tj].value += tempValue2;
#ifdef USE_CUTOFF
                }
#endif
#ifdef USE_EXCLUSIONS
                excl >>= 1;
#endif
                tj = (tj + 1) & (TILE_SIZE - 1);
            }
        }

        // Write results.

        unsigned int offset = x*TILE_SIZE + tgx;
        atomicAdd(&global_value[offset], static_cast<unsigned long long>((long long) (value*0x100000000)));
        if (x != y) {
            offset = y*TILE_SIZE + tgx;
            atomicAdd(&global_value[offset], static_cast<unsigned long long>((long long) (localData[threadIdx.x].value*0x100000000)));
        }
    }

    // Second loop: tiles without exclusions, either from the neighbor list (with cutoff) or just enumerating all
    // of them (no cutoff).

#ifdef USE_CUTOFF
    unsigned int numTiles = interactionCount[0];
    if (numTiles > maxTiles)
        return; // There wasn't enough memory for the neighbor list.
    int pos = (int) (warp*(numTiles > maxTiles ? NUM_BLOCKS*((long long)NUM_BLOCKS+1)/2 : (long)numTiles)/totalWarps);
    int end = (int) ((warp+1)*(numTiles > maxTiles ? NUM_BLOCKS*((long long)NUM_BLOCKS+1)/2 : (long)numTiles)/totalWarps);
#else
    int pos = (int) (warp*(long long)numTiles/totalWarps);
    int end = (int) ((warp+1)*(long long)numTiles/totalWarps);
#endif
    int skipBase = 0;
    int currentSkipIndex = tbx;
    __shared__ int atomIndices[THREAD_BLOCK_SIZE];
    __shared__ volatile int skipTiles[THREAD_BLOCK_SIZE];
    skipTiles[threadIdx.x] = -1;
    
    while (pos < end) {
        real value = 0;
        bool includeTile = true;
        
        // Extract the coordinates of this tile.
        
        int x, y;
        bool singlePeriodicCopy = false;
#ifdef USE_CUTOFF
        x = tiles[pos];
        real4 blockSizeX = blockSize[x];
        singlePeriodicCopy = (0.5f*periodicBoxSize.x-blockSizeX.x >= CUTOFF &&
                              0.5f*periodicBoxSize.y-blockSizeX.y >= CUTOFF &&
                              0.5f*periodicBoxSize.z-blockSizeX.z >= CUTOFF);
#else
        y = (int) floor(NUM_BLOCKS+0.5f-SQRT((NUM_BLOCKS+0.5f)*(NUM_BLOCKS+0.5f)-2*pos));
        x = (pos-y*NUM_BLOCKS+y*(y+1)/2);
        if (x < y || x >= NUM_BLOCKS) { // Occasionally happens due to roundoff error.
            y += (x < y ? -1 : 1);
            x = (pos-y*NUM_BLOCKS+y*(y+1)/2);
        }

        // Skip over tiles that have exclusions, since they were already processed.

        while (skipTiles[tbx+TILE_SIZE-1] < pos) {
            if (skipBase+tgx < NUM_TILES_WITH_EXCLUSIONS) {
                ushort2 tile = exclusionTiles[skipBase+tgx];
                skipTiles[threadIdx.x] = tile.x + tile.y*NUM_BLOCKS - tile.y*(tile.y+1)/2;
            }
            else
                skipTiles[threadIdx.x] = end;
            skipBase += TILE_SIZE;            
            currentSkipIndex = tbx;
        }
        while (skipTiles[currentSkipIndex] < pos)
            currentSkipIndex++;
        includeTile = (skipTiles[currentSkipIndex] != pos);
#endif
        if (includeTile) {
            unsigned int atom1 = x*TILE_SIZE + tgx;

            // Load atom data for this tile.
            
            real4 pos1 = posq[atom1];
            LOAD_ATOM1_PARAMETERS
            const unsigned int localAtomIndex = threadIdx.x;
#ifdef USE_CUTOFF
            unsigned int j = (numTiles <= maxTiles ? interactingAtoms[pos*TILE_SIZE+tgx] : y*TILE_SIZE + tgx);
#else
            unsigned int j = y*TILE_SIZE + tgx;
#endif
            atomIndices[threadIdx.x] = j;
            if (j < PADDED_NUM_ATOMS) {
                real4 tempPosq = posq[j];
                localData[localAtomIndex].pos = make_real3(tempPosq.x, tempPosq.y, tempPosq.z);
                LOAD_LOCAL_PARAMETERS_FROM_GLOBAL
                localData[localAtomIndex].value = 0;
            }
#ifdef USE_PERIODIC
            if (singlePeriodicCopy) {
                // The box is small enough that we can just translate all the atoms into a single periodic
                // box, then skip having to apply periodic boundary conditions later.

                real4 blockCenterX = blockCenter[x];
                APPLY_PERIODIC_TO_POS_WITH_CENTER(pos1, blockCenterX)
                APPLY_PERIODIC_TO_POS_WITH_CENTER(localData[threadIdx.x].pos, blockCenterX)
                unsigned int tj = tgx;
                for (unsigned int j = 0; j < TILE_SIZE; j++) {
                    int atom2 = tbx+tj;
                    real3 pos2 = localData[atom2].pos;
                    real3 delta = make_real3(pos2.x-pos1.x, pos2.y-pos1.y, pos2.z-pos1.z);
                    real r2 = delta.x*delta.x + delta.y*delta.y + delta.z*delta.z;
                    if (r2 < CUTOFF_SQUARED) {
                        real invR = RSQRT(r2);
                        real r = r2*invR;
                        LOAD_ATOM2_PARAMETERS
                        atom2 = atomIndices[tbx+tj];
                        real tempValue1 = 0;
                        real tempValue2 = 0;
                        if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS) {
                            COMPUTE_VALUE
                        }
                        value += tempValue1;
                        localData[tbx+tj].value += tempValue2;
                    }
                    tj = (tj + 1) & (TILE_SIZE - 1);
                }
            }
            else
#endif
            {
                // We need to apply periodic boundary conditions separately for each interaction.

                unsigned int tj = tgx;
                for (unsigned int j = 0; j < TILE_SIZE; j++) {
                    int atom2 = tbx+tj;
                    real3 pos2 = localData[atom2].pos;
                    real3 delta = make_real3(pos2.x-pos1.x, pos2.y-pos1.y, pos2.z-pos1.z);
#ifdef USE_PERIODIC
                    APPLY_PERIODIC_TO_DELTA(delta)
#endif
                    real r2 = delta.x*delta.x + delta.y*delta.y + delta.z*delta.z;
#ifdef USE_CUTOFF
                    if (r2 < CUTOFF_SQUARED) {
#endif
                        real invR = RSQRT(r2);
                        real r = r2*invR;
                        LOAD_ATOM2_PARAMETERS
                        atom2 = atomIndices[tbx+tj];
                        real tempValue1 = 0;
                        real tempValue2 = 0;
                        if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS) {
                            COMPUTE_VALUE
                        }
                        value += tempValue1;
                        localData[tbx+tj].value += tempValue2;
#ifdef USE_CUTOFF
                    }
#endif
                    tj = (tj + 1) & (TILE_SIZE - 1);
                }
            }
        
            // Write results.

            atomicAdd(&global_value[atom1], static_cast<unsigned long long>((long long) (value*0x100000000)));
#ifdef USE_CUTOFF
            unsigned int atom2 = atomIndices[threadIdx.x];
#else
            unsigned int atom2 = y*TILE_SIZE + tgx;
#endif
            if (atom2 < PADDED_NUM_ATOMS)
                atomicAdd(&global_value[atom2], static_cast<unsigned long long>((long long) (localData[threadIdx.x].value*0x100000000)));
        }
        pos++;
    }
}
