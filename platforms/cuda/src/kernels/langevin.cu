#include "hip/hip_runtime.h"
enum {VelScale, ForceScale, NoiseScale, MaxParams};

/**
 * Perform the first step of Langevin integration.
 */

extern "C" __global__ void integrateLangevinPart1(int numAtoms, int paddedNumAtoms, mixed4* __restrict__ velm, const long long* __restrict__ force, mixed4* __restrict__ posDelta,
        const mixed* __restrict__ paramBuffer, const mixed2* __restrict__ dt, const float4* __restrict__ random, unsigned int randomIndex) {
    mixed vscale = paramBuffer[VelScale];
    mixed fscale = paramBuffer[ForceScale]/(mixed) 0x100000000;
    mixed noisescale = paramBuffer[NoiseScale];
    mixed stepSize = dt[0].y;
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    randomIndex += index;
    while (index < numAtoms) {
        mixed4 velocity = velm[index];
        if (velocity.w != 0) {
            mixed sqrtInvMass = SQRT(velocity.w);
            velocity.x = vscale*velocity.x + fscale*velocity.w*force[index] + noisescale*sqrtInvMass*random[randomIndex].x;
            velocity.y = vscale*velocity.y + fscale*velocity.w*force[index+paddedNumAtoms] + noisescale*sqrtInvMass*random[randomIndex].y;
            velocity.z = vscale*velocity.z + fscale*velocity.w*force[index+paddedNumAtoms*2] + noisescale*sqrtInvMass*random[randomIndex].z;
            velm[index] = velocity;
            posDelta[index] = make_mixed4(stepSize*velocity.x, stepSize*velocity.y, stepSize*velocity.z, 0);
        }
        randomIndex += blockDim.x*gridDim.x;
        index += blockDim.x*gridDim.x;
    }
}

/**
 * Perform the second step of Langevin integration.
 */

extern "C" __global__ void integrateLangevinPart2(int numAtoms, real4* __restrict__ posq, real4* __restrict__ posqCorrection, const mixed4* __restrict__ posDelta, mixed4* __restrict__ velm, const mixed2* __restrict__ dt) {
#if __CUDA_ARCH__ >= 130
    double invStepSize = 1.0/dt[0].y;
#else
    float invStepSize = 1.0f/dt[0].y;
    float correction = (1.0f-invStepSize*dt[0].y)/dt[0].y;
#endif
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    while (index < numAtoms) {
        mixed4 vel = velm[index];
        if (vel.w != 0) {
#ifdef USE_MIXED_PRECISION
            real4 pos1 = posq[index];
            real4 pos2 = posqCorrection[index];
            mixed4 pos = make_mixed4(pos1.x+(mixed)pos2.x, pos1.y+(mixed)pos2.y, pos1.z+(mixed)pos2.z, pos1.w);
#else
            real4 pos = posq[index];
#endif
            mixed4 delta = posDelta[index];
            pos.x += delta.x;
            pos.y += delta.y;
            pos.z += delta.z;
#if __CUDA_ARCH__ >= 130
            vel.x = (mixed) (invStepSize*delta.x);
            vel.y = (mixed) (invStepSize*delta.y);
            vel.z = (mixed) (invStepSize*delta.z);
#else
            vel.x = invStepSize*delta.x + correction*delta.x;
            vel.y = invStepSize*delta.y + correction*delta.x;
            vel.z = invStepSize*delta.z + correction*delta.x;
#endif
#ifdef USE_MIXED_PRECISION
            posq[index] = make_real4((real) pos.x, (real) pos.y, (real) pos.z, (real) pos.w);
            posqCorrection[index] = make_real4(pos.x-(real) pos.x, pos.y-(real) pos.y, pos.z-(real) pos.z, 0);
#else
            posq[index] = pos;
#endif
            velm[index] = vel;
        }
        index += blockDim.x*gridDim.x;
    }
}

/**
 * Select the step size to use for the next step.
 */

extern "C" __global__ void selectLangevinStepSize(int numAtoms, int paddedNumAtoms, mixed maxStepSize, mixed errorTol, mixed tau, mixed kT, mixed2* __restrict__ dt,
        const mixed4* __restrict__ velm, const long long* __restrict__ force, mixed* __restrict__ paramBuffer) {
    // Calculate the error.

    extern __shared__ mixed params[];
    mixed* error = &params[MaxParams];
    mixed err = 0;
    unsigned int index = threadIdx.x;
    const mixed scale = RECIP((mixed) 0x100000000);
    while (index < numAtoms) {
        mixed3 f = make_mixed3(scale*force[index], scale*force[index+paddedNumAtoms], scale*force[index+paddedNumAtoms*2]);
        mixed invMass = velm[index].w;
        err += (f.x*f.x + f.y*f.y + f.z*f.z)*invMass;
        index += blockDim.x*gridDim.x;
    }
    error[threadIdx.x] = err;
    __syncthreads();

    // Sum the errors from all threads.

    for (unsigned int offset = 1; offset < blockDim.x; offset *= 2) {
        if (threadIdx.x+offset < blockDim.x && (threadIdx.x&(2*offset-1)) == 0)
            error[threadIdx.x] += error[threadIdx.x+offset];
        __syncthreads();
    }
    if (blockIdx.x*blockDim.x+threadIdx.x == 0) {
        // Select the new step size.

        mixed totalError = SQRT(error[0]/(numAtoms*3));
        mixed newStepSize = SQRT(errorTol/totalError);
        mixed oldStepSize = dt[0].y;
        if (oldStepSize > 0.0f)
            newStepSize = min(newStepSize, oldStepSize*2.0f); // For safety, limit how quickly dt can increase.
        if (newStepSize > oldStepSize && newStepSize < 1.1f*oldStepSize)
            newStepSize = oldStepSize; // Keeping dt constant between steps improves the behavior of the integrator.
        if (newStepSize > maxStepSize)
            newStepSize = maxStepSize;
        dt[0].y = newStepSize;

        // Recalculate the integration parameters.

        mixed vscale = EXP(-newStepSize/tau);
        mixed fscale = (1-vscale)*tau;
        mixed noisescale = SQRT(2*kT/tau)*SQRT(0.5f*(1-vscale*vscale)*tau);
        params[VelScale] = vscale;
        params[ForceScale] = fscale;
        params[NoiseScale] = noisescale;
    }
    __syncthreads();
    if (threadIdx.x < MaxParams)
        paramBuffer[threadIdx.x] = params[threadIdx.x];
}
