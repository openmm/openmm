#include "hip/hip_runtime.h"
extern "C" __global__ void computeFloatSum(const float* __restrict__ sumBuffer, float* result) {
    __shared__ float tempBuffer[WORK_GROUP_SIZE];
    const unsigned int thread = threadIdx.x;
    float sum = 0;
    for (unsigned int index = thread; index < SUM_BUFFER_SIZE; index += blockDim.x)
        sum += sumBuffer[index];
    tempBuffer[thread] = sum;
    for (int i = 1; i < WORK_GROUP_SIZE; i *= 2) {
        __syncthreads();
        if (thread%(i*2) == 0 && thread+i < WORK_GROUP_SIZE)
            tempBuffer[thread] += tempBuffer[thread+i];
    }
    if (thread == 0)
        result[SUM_OUTPUT_INDEX] = tempBuffer[0];
}

extern "C" __global__ void computeDoubleSum(const double* __restrict__ sumBuffer, double* result) {
    __shared__ double tempBuffer[WORK_GROUP_SIZE];
    const unsigned int thread = threadIdx.x;
    double sum = 0;
    for (unsigned int index = thread; index < SUM_BUFFER_SIZE; index += blockDim.x)
        sum += sumBuffer[index];
    tempBuffer[thread] = sum;
    for (int i = 1; i < WORK_GROUP_SIZE; i *= 2) {
        __syncthreads();
        if (thread%(i*2) == 0 && thread+i < WORK_GROUP_SIZE)
            tempBuffer[thread] += tempBuffer[thread+i];
    }
    if (thread == 0)
        result[SUM_OUTPUT_INDEX] = tempBuffer[0];
}

extern "C" __global__ void applyPositionDeltas(real4* __restrict__ posq, real4* __restrict__ posqCorrection, mixed4* __restrict__ posDelta) {
    for (unsigned int index = blockIdx.x*blockDim.x+threadIdx.x; index < NUM_ATOMS; index += blockDim.x*gridDim.x) {
#ifdef USE_MIXED_PRECISION
        real4 pos1 = posq[index];
        real4 pos2 = posqCorrection[index];
        mixed4 pos = make_mixed4(pos1.x+(mixed)pos2.x, pos1.y+(mixed)pos2.y, pos1.z+(mixed)pos2.z, pos1.w);
#else
        real4 pos = posq[index];
#endif
        pos.x += posDelta[index].x;
        pos.y += posDelta[index].y;
        pos.z += posDelta[index].z;
#ifdef USE_MIXED_PRECISION
        posq[index] = make_real4((real) pos.x, (real) pos.y, (real) pos.z, (real) pos.w);
        posqCorrection[index] = make_real4(pos.x-(real) pos.x, pos.y-(real) pos.y, pos.z-(real) pos.z, 0);
#else
        posq[index] = pos;
#endif
        posDelta[index] = make_mixed4(0, 0, 0, 0);
    }
}

extern "C" __global__ void generateRandomNumbers(int numValues, float4* __restrict__ random, uint4* __restrict__ seed) {
    uint4 state = seed[blockIdx.x*blockDim.x+threadIdx.x];
    unsigned int carry = 0;
    for (int index = blockIdx.x*blockDim.x+threadIdx.x; index < numValues; index += blockDim.x*gridDim.x) {
        // Generate three uniform random numbers.

        state.x = state.x * 69069 + 1;
        state.y ^= state.y << 13;
        state.y ^= state.y >> 17;
        state.y ^= state.y << 5;
        unsigned int k = (state.z >> 2) + (state.w >> 3) + (carry >> 2);
        unsigned int m = state.w + state.w + state.z + carry;
        state.z = state.w;
        state.w = m;
        carry = k >> 30;
        float x1 = (float)max(state.x + state.y + state.w, 0x00000001u) / (float)0xffffffff;
        state.x = state.x * 69069 + 1;
        state.y ^= state.y << 13;
        state.y ^= state.y >> 17;
        state.y ^= state.y << 5;
        k = (state.z >> 2) + (state.w >> 3) + (carry >> 2);
        m = state.w + state.w + state.z + carry;
        state.z = state.w;
        state.w = m;
        carry = k >> 30;
        float x2 = (float)max(state.x + state.y + state.w, 0x00000001u) / (float)0xffffffff;
        state.x = state.x * 69069 + 1;
        state.y ^= state.y << 13;
        state.y ^= state.y >> 17;
        state.y ^= state.y << 5;
        k = (state.z >> 2) + (state.w >> 3) + (carry >> 2);
        m = state.w + state.w + state.z + carry;
        state.z = state.w;
        state.w = m;
        carry = k >> 30;
        float x3 = (float)max(state.x + state.y + state.w, 0x00000001u) / (float)0xffffffff;

        // Record the values.

        random[index] = make_float4(x1, x2, x3, 0.0f);
    }
    seed[blockIdx.x*blockDim.x+threadIdx.x] = state;
}
