#include "hip/hip_runtime.h"
float4 exceptionParams = PARAMS[index];
real3 delta = make_real3(pos2.x-pos1.x, pos2.y-pos1.y, pos2.z-pos1.z);
#if APPLY_PERIODIC
APPLY_PERIODIC_TO_DELTA(delta)
#endif
real r2 = delta.x*delta.x + delta.y*delta.y + delta.z*delta.z;
real invR = RSQRT(r2);
real sig2 = invR*exceptionParams.y;
sig2 *= sig2;
real sig6 = sig2*sig2*sig2;
real dEdR = exceptionParams.z*(12.0f*sig6-6.0f)*sig6;
real tempEnergy = exceptionParams.z*(sig6-1.0f)*sig6;
dEdR += exceptionParams.x*invR;
dEdR *= invR*invR;
tempEnergy += exceptionParams.x*invR;
energy += tempEnergy;
delta *= dEdR;
real3 force1 = -delta;
real3 force2 = delta;
