#include "hip/hip_runtime.h"
#define STORE_DERIVATIVE_1(INDEX) atomicAdd(&derivBuffers[offset+(INDEX-1)*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (deriv##INDEX##_1*0x100000000)));
#define STORE_DERIVATIVE_2(INDEX) atomicAdd(&derivBuffers[offset+(INDEX-1)*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].deriv##INDEX*0x100000000)));

typedef struct {
    real4 posq;
    real3 force;
    ATOM_PARAMETER_DATA
#ifdef NEED_PADDING
    float padding;
#endif
} AtomData;

/**
 * Compute a force based on pair interactions.
 */
extern "C" __global__ void computeN2Energy(unsigned long long* __restrict__ forceBuffers, real* __restrict__ energyBuffer,
        const real4* __restrict__ posq, const unsigned int* __restrict__ exclusions, const ushort2* __restrict__ exclusionTiles,
#ifdef USE_CUTOFF
        const int* __restrict__ tiles, const unsigned int* __restrict__ interactionCount, real4 periodicBoxSize, real4 invPeriodicBoxSize, 
        unsigned int maxTiles, const real4* __restrict__ blockCenter, const real4* __restrict__ blockSize, const unsigned int* __restrict__ interactingAtoms
#else
        unsigned int numTiles
#endif
        PARAMETER_ARGUMENTS) {
    const unsigned int totalWarps = (blockDim.x*gridDim.x)/TILE_SIZE;
    const unsigned int warp = (blockIdx.x*blockDim.x+threadIdx.x)/TILE_SIZE;
    const unsigned int tgx = threadIdx.x & (TILE_SIZE-1);
    const unsigned int tbx = threadIdx.x - tgx;
    real energy = 0;
    __shared__ AtomData localData[THREAD_BLOCK_SIZE];

    // First loop: process tiles that contain exclusions.
    
    const unsigned int firstExclusionTile = FIRST_EXCLUSION_TILE+warp*(LAST_EXCLUSION_TILE-FIRST_EXCLUSION_TILE)/totalWarps;
    const unsigned int lastExclusionTile = FIRST_EXCLUSION_TILE+(warp+1)*(LAST_EXCLUSION_TILE-FIRST_EXCLUSION_TILE)/totalWarps;
    for (int pos = firstExclusionTile; pos < lastExclusionTile; pos++) {
        const ushort2 tileIndices = exclusionTiles[pos];
        const unsigned int x = tileIndices.x;
        const unsigned int y = tileIndices.y;
        real3 force = make_real3(0);
        DECLARE_ATOM1_DERIVATIVES
        unsigned int atom1 = x*TILE_SIZE + tgx;
        real4 posq1 = posq[atom1];
        LOAD_ATOM1_PARAMETERS
#ifdef USE_EXCLUSIONS
        unsigned int excl = exclusions[pos*TILE_SIZE+tgx];
#endif
        if (x == y) {
            // This tile is on the diagonal.

            const unsigned int localAtomIndex = threadIdx.x;
            localData[localAtomIndex].posq = posq1;
            LOAD_LOCAL_PARAMETERS_FROM_1
            for (unsigned int j = 0; j < TILE_SIZE; j++) {
                int atom2 = tbx+j;
                real4 posq2 = localData[atom2].posq;
                real3 delta = make_real3(posq2.x-posq1.x, posq2.y-posq1.y, posq2.z-posq1.z);
#ifdef USE_PERIODIC
                delta.x -= floor(delta.x*invPeriodicBoxSize.x+0.5f)*periodicBoxSize.x;
                delta.y -= floor(delta.y*invPeriodicBoxSize.y+0.5f)*periodicBoxSize.y;
                delta.z -= floor(delta.z*invPeriodicBoxSize.z+0.5f)*periodicBoxSize.z;
#endif
                real r2 = delta.x*delta.x + delta.y*delta.y + delta.z*delta.z;
#ifdef USE_CUTOFF
                if (r2 < CUTOFF_SQUARED) {
#endif
                    real invR = RSQRT(r2);
                    real r = RECIP(invR);
                    LOAD_ATOM2_PARAMETERS
                    atom2 = y*TILE_SIZE+j;
                    real dEdR = 0;
                    real tempEnergy = 0;
#ifdef USE_EXCLUSIONS
                    bool isExcluded = !(excl & 0x1);
#endif
                    if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS && atom1 != atom2) {
                        COMPUTE_INTERACTION
                        dEdR /= -r;
                    }
                    energy += 0.5f*tempEnergy;
                    delta *= dEdR;
                    force.x -= delta.x;
                    force.y -= delta.y;
                    force.z -= delta.z;
#ifdef USE_CUTOFF
                }
#endif
#ifdef USE_EXCLUSIONS
                excl >>= 1;
#endif
            }
        }
        else {
            // This is an off-diagonal tile.

            const unsigned int localAtomIndex = threadIdx.x;
            unsigned int j = y*TILE_SIZE + tgx;
            localData[localAtomIndex].posq = posq[j];
            LOAD_LOCAL_PARAMETERS_FROM_GLOBAL
            localData[localAtomIndex].force = make_real3(0);
            CLEAR_LOCAL_DERIVATIVES
#ifdef USE_EXCLUSIONS
            excl = (excl >> tgx) | (excl << (TILE_SIZE - tgx));
#endif
            unsigned int tj = tgx;
            for (j = 0; j < TILE_SIZE; j++) {
                int atom2 = tbx+tj;
                real4 posq2 = localData[atom2].posq;
                real3 delta = make_real3(posq2.x-posq1.x, posq2.y-posq1.y, posq2.z-posq1.z);
#ifdef USE_PERIODIC
                delta.x -= floor(delta.x*invPeriodicBoxSize.x+0.5f)*periodicBoxSize.x;
                delta.y -= floor(delta.y*invPeriodicBoxSize.y+0.5f)*periodicBoxSize.y;
                delta.z -= floor(delta.z*invPeriodicBoxSize.z+0.5f)*periodicBoxSize.z;
#endif
                real r2 = delta.x*delta.x + delta.y*delta.y + delta.z*delta.z;
#ifdef USE_CUTOFF
                if (r2 < CUTOFF_SQUARED) {
#endif
                    real invR = RSQRT(r2);
                    real r = RECIP(invR);
                    LOAD_ATOM2_PARAMETERS
                    atom2 = y*TILE_SIZE+tj;
                    real dEdR = 0;
                    real tempEnergy = 0;
#ifdef USE_EXCLUSIONS
                    bool isExcluded = !(excl & 0x1);
#endif
                    if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS) {
                        COMPUTE_INTERACTION
                        dEdR /= -r;
                    }
                    energy += tempEnergy;
                    delta *= dEdR;
                    force.x -= delta.x;
                    force.y -= delta.y;
                    force.z -= delta.z;
                    atom2 = tbx+tj;
                    localData[atom2].force.x += delta.x;
                    localData[atom2].force.y += delta.y;
                    localData[atom2].force.z += delta.z;
                    RECORD_DERIVATIVE_2
#ifdef USE_CUTOFF
                }
#endif
#ifdef USE_EXCLUSIONS
                excl >>= 1;
#endif
                tj = (tj + 1) & (TILE_SIZE - 1);
            }
        }

        // Write results.

        unsigned int offset = x*TILE_SIZE + tgx;
        atomicAdd(&forceBuffers[offset], static_cast<unsigned long long>((long long) (force.x*0x100000000)));
        atomicAdd(&forceBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (force.y*0x100000000)));
        atomicAdd(&forceBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (force.z*0x100000000)));
        STORE_DERIVATIVES_1
        if (x != y) {
            offset = y*TILE_SIZE + tgx;
            atomicAdd(&forceBuffers[offset], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.x*0x100000000)));
            atomicAdd(&forceBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.y*0x100000000)));
            atomicAdd(&forceBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.z*0x100000000)));
            STORE_DERIVATIVES_2
        }
    }

    // Second loop: tiles without exclusions, either from the neighbor list (with cutoff) or just enumerating all
    // of them (no cutoff).

#ifdef USE_CUTOFF
    unsigned int numTiles = interactionCount[0];
    int pos = warp*(numTiles > maxTiles ? NUM_BLOCKS*(NUM_BLOCKS+1)/2 : numTiles)/totalWarps;
    int end = (warp+1)*(numTiles > maxTiles ? NUM_BLOCKS*(NUM_BLOCKS+1)/2 : numTiles)/totalWarps;
#else
    int pos = warp*numTiles/totalWarps;
    int end = (warp+1)*numTiles/totalWarps;
#endif
    int skipBase = 0;
    int currentSkipIndex = tbx;
    __shared__ int atomIndices[THREAD_BLOCK_SIZE];
    __shared__ volatile int skipTiles[THREAD_BLOCK_SIZE];
    skipTiles[threadIdx.x] = -1;
    
    while (pos < end) {
        const bool isExcluded = false;
        real3 force = make_real3(0);
        DECLARE_ATOM1_DERIVATIVES
        bool includeTile = true;
        
        // Extract the coordinates of this tile.
        
        unsigned int x, y;
        bool singlePeriodicCopy = false;
#ifdef USE_CUTOFF
        if (numTiles <= maxTiles) {
            x = tiles[pos];
            real4 blockSizeX = blockSize[x];
            singlePeriodicCopy = (0.5f*periodicBoxSize.x-blockSizeX.x >= CUTOFF &&
                                  0.5f*periodicBoxSize.y-blockSizeX.y >= CUTOFF &&
                                  0.5f*periodicBoxSize.z-blockSizeX.z >= CUTOFF);
        }
        else
#endif
        {
            y = (unsigned int) floor(NUM_BLOCKS+0.5f-SQRT((NUM_BLOCKS+0.5f)*(NUM_BLOCKS+0.5f)-2*pos));
            x = (pos-y*NUM_BLOCKS+y*(y+1)/2);
            if (x < y || x >= NUM_BLOCKS) { // Occasionally happens due to roundoff error.
                y += (x < y ? -1 : 1);
                x = (pos-y*NUM_BLOCKS+y*(y+1)/2);
            }

            // Skip over tiles that have exclusions, since they were already processed.

            while (skipTiles[tbx+TILE_SIZE-1] < pos) {
                if (skipBase+tgx < NUM_TILES_WITH_EXCLUSIONS) {
                    ushort2 tile = exclusionTiles[skipBase+tgx];
                    skipTiles[threadIdx.x] = tile.x + tile.y*NUM_BLOCKS - tile.y*(tile.y+1)/2;
                }
                else
                    skipTiles[threadIdx.x] = end;
                skipBase += TILE_SIZE;            
                currentSkipIndex = tbx;
            }
            while (skipTiles[currentSkipIndex] < pos)
                currentSkipIndex++;
            includeTile = (skipTiles[currentSkipIndex] != pos);
        }
        if (includeTile) {
            unsigned int atom1 = x*TILE_SIZE + tgx;

            // Load atom data for this tile.

            real4 posq1 = posq[atom1];
            LOAD_ATOM1_PARAMETERS
            const unsigned int localAtomIndex = threadIdx.x;
#ifdef USE_CUTOFF
            unsigned int j = (numTiles <= maxTiles ? interactingAtoms[pos*TILE_SIZE+tgx] : y*TILE_SIZE + tgx);
#else
            unsigned int j = y*TILE_SIZE + tgx;
#endif
            atomIndices[threadIdx.x] = j;
            if (j < PADDED_NUM_ATOMS) {
                localData[localAtomIndex].posq = posq[j];
                LOAD_LOCAL_PARAMETERS_FROM_GLOBAL
                localData[localAtomIndex].force = make_real3(0);
                CLEAR_LOCAL_DERIVATIVES
            }
#ifdef USE_PERIODIC
            if (singlePeriodicCopy) {
                // The box is small enough that we can just translate all the atoms into a single periodic
                // box, then skip having to apply periodic boundary conditions later.

                real4 blockCenterX = blockCenter[x];
                posq1.x -= floor((posq1.x-blockCenterX.x)*invPeriodicBoxSize.x+0.5f)*periodicBoxSize.x;
                posq1.y -= floor((posq1.y-blockCenterX.y)*invPeriodicBoxSize.y+0.5f)*periodicBoxSize.y;
                posq1.z -= floor((posq1.z-blockCenterX.z)*invPeriodicBoxSize.z+0.5f)*periodicBoxSize.z;
                localData[threadIdx.x].posq.x -= floor((localData[threadIdx.x].posq.x-blockCenterX.x)*invPeriodicBoxSize.x+0.5f)*periodicBoxSize.x;
                localData[threadIdx.x].posq.y -= floor((localData[threadIdx.x].posq.y-blockCenterX.y)*invPeriodicBoxSize.y+0.5f)*periodicBoxSize.y;
                localData[threadIdx.x].posq.z -= floor((localData[threadIdx.x].posq.z-blockCenterX.z)*invPeriodicBoxSize.z+0.5f)*periodicBoxSize.z;
                unsigned int tj = tgx;
                for (j = 0; j < TILE_SIZE; j++) {
                    int atom2 = tbx+tj;
                    real4 posq2 = localData[atom2].posq;
                    real3 delta = make_real3(posq2.x-posq1.x, posq2.y-posq1.y, posq2.z-posq1.z);
                    real r2 = delta.x*delta.x + delta.y*delta.y + delta.z*delta.z;
#ifdef USE_CUTOFF
                    if (r2 < CUTOFF_SQUARED) {
#endif
                        real invR = RSQRT(r2);
                        real r = RECIP(invR);
                        LOAD_ATOM2_PARAMETERS
                        atom2 = atomIndices[tbx+tj];
                        real dEdR = 0;
                        real tempEnergy = 0;
                        if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS) {
                            COMPUTE_INTERACTION
                            dEdR /= -r;
                        }
                        energy += tempEnergy;
                        delta *= dEdR;
                        force.x -= delta.x;
                        force.y -= delta.y;
                        force.z -= delta.z;
                        atom2 = tbx+tj;
                        localData[atom2].force.x += delta.x;
                        localData[atom2].force.y += delta.y;
                        localData[atom2].force.z += delta.z;
                        RECORD_DERIVATIVE_2
#ifdef USE_CUTOFF
                    }
#endif
                    tj = (tj + 1) & (TILE_SIZE - 1);
                }
            }
            else
#endif
            {
                // We need to apply periodic boundary conditions separately for each interaction.

                unsigned int tj = tgx;
                for (j = 0; j < TILE_SIZE; j++) {
                    int atom2 = tbx+tj;
                    real4 posq2 = localData[atom2].posq;
                    real3 delta = make_real3(posq2.x-posq1.x, posq2.y-posq1.y, posq2.z-posq1.z);
#ifdef USE_PERIODIC
                    delta.x -= floor(delta.x*invPeriodicBoxSize.x+0.5f)*periodicBoxSize.x;
                    delta.y -= floor(delta.y*invPeriodicBoxSize.y+0.5f)*periodicBoxSize.y;
                    delta.z -= floor(delta.z*invPeriodicBoxSize.z+0.5f)*periodicBoxSize.z;
#endif
                    real r2 = delta.x*delta.x + delta.y*delta.y + delta.z*delta.z;
#ifdef USE_CUTOFF
                    if (r2 < CUTOFF_SQUARED) {
#endif
                        real invR = RSQRT(r2);
                        real r = RECIP(invR);
                        LOAD_ATOM2_PARAMETERS
                        atom2 = atomIndices[tbx+tj];
                        real dEdR = 0;
                        real tempEnergy = 0;
                        if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS) {
                            COMPUTE_INTERACTION
                            dEdR /= -r;
                        }
                        energy += tempEnergy;
                        delta *= dEdR;
                        force.x -= delta.x;
                        force.y -= delta.y;
                        force.z -= delta.z;
                        atom2 = tbx+tj;
                        localData[atom2].force.x += delta.x;
                        localData[atom2].force.y += delta.y;
                        localData[atom2].force.z += delta.z;
                        RECORD_DERIVATIVE_2
#ifdef USE_CUTOFF
                    }
#endif
                    tj = (tj + 1) & (TILE_SIZE - 1);
                }
            }
        
            // Write results.

            atomicAdd(&forceBuffers[atom1], static_cast<unsigned long long>((long long) (force.x*0x100000000)));
            atomicAdd(&forceBuffers[atom1+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (force.y*0x100000000)));
            atomicAdd(&forceBuffers[atom1+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (force.z*0x100000000)));
            unsigned int offset = atom1;
            STORE_DERIVATIVES_1
#ifdef USE_CUTOFF
            unsigned int atom2 = atomIndices[threadIdx.x];
#else
            unsigned int atom2 = y*TILE_SIZE + tgx;
#endif
            if (atom2 < PADDED_NUM_ATOMS) {
                atomicAdd(&forceBuffers[atom2], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.x*0x100000000)));
                atomicAdd(&forceBuffers[atom2+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.y*0x100000000)));
                atomicAdd(&forceBuffers[atom2+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.z*0x100000000)));
                offset = atom2;
                STORE_DERIVATIVES_2
            }
        }
        pos++;
    }
    energyBuffer[blockIdx.x*blockDim.x+threadIdx.x] += energy;
}
