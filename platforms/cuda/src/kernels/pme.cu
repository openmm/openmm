#include "hip/hip_runtime.h"
extern "C" __global__ void findAtomGridIndex(const real4* __restrict__ posq, int2* __restrict__ pmeAtomGridIndex,
            real4 periodicBoxSize, real4 invPeriodicBoxSize, real4 periodicBoxVecX, real4 periodicBoxVecY, real4 periodicBoxVecZ,
            real3 recipBoxVecX, real3 recipBoxVecY, real3 recipBoxVecZ) {
    // Compute the index of the grid point each atom is associated with.
    
    for (int atom = blockIdx.x*blockDim.x+threadIdx.x; atom < NUM_ATOMS; atom += blockDim.x*gridDim.x) {
        real4 pos = posq[atom];
        APPLY_PERIODIC_TO_POS(pos)
        real3 t = make_real3(pos.x*recipBoxVecX.x+pos.y*recipBoxVecY.x+pos.z*recipBoxVecZ.x,
                             pos.y*recipBoxVecY.y+pos.z*recipBoxVecZ.y,
                             pos.z*recipBoxVecZ.z);
        t.x = (t.x-floor(t.x))*GRID_SIZE_X;
        t.y = (t.y-floor(t.y))*GRID_SIZE_Y;
        t.z = (t.z-floor(t.z))*GRID_SIZE_Z;
        int3 gridIndex = make_int3(((int) t.x) % GRID_SIZE_X,
                                   ((int) t.y) % GRID_SIZE_Y,
                                   ((int) t.z) % GRID_SIZE_Z);
        pmeAtomGridIndex[atom] = make_int2(atom, gridIndex.x*GRID_SIZE_Y*GRID_SIZE_Z+gridIndex.y*GRID_SIZE_Z+gridIndex.z);
    }
}

extern "C" __global__ void gridSpreadCharge(const real4* __restrict__ posq, real* __restrict__ originalPmeGrid,
        real4 periodicBoxSize, real4 invPeriodicBoxSize, real4 periodicBoxVecX, real4 periodicBoxVecY, real4 periodicBoxVecZ,
        real3 recipBoxVecX, real3 recipBoxVecY, real3 recipBoxVecZ, const int2* __restrict__ pmeAtomGridIndex
#ifdef CHARGE_FROM_SIGEPS
        , const float2* __restrict__ sigmaEpsilon
#else
        , const real* __restrict__ charges
#endif
        ) {
    // To improve memory efficiency, we divide indices along the z axis into
    // PME_ORDER blocks, where the data for each block is stored together.  We
    // can ensure that all threads write to the same block at the same time,
    // which leads to better coalescing of writes.
    
    __shared__ int zindexTable[GRID_SIZE_Z+PME_ORDER];
    int blockSize = (int) ceil(GRID_SIZE_Z/(real) PME_ORDER);
    for (int i = threadIdx.x; i < GRID_SIZE_Z+PME_ORDER; i += blockDim.x) {
        int zindex = i % GRID_SIZE_Z;
	int block = zindex % PME_ORDER;
        zindexTable[i] = zindex/PME_ORDER + block*GRID_SIZE_X*GRID_SIZE_Y*blockSize;
    }
    __syncthreads();
    
    // Process the atoms in spatially sorted order.  This improves efficiency when writing
    // the grid values.
    
    real3 data[PME_ORDER];
    const real scale = RECIP(PME_ORDER-1);
    for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < NUM_ATOMS; i += blockDim.x*gridDim.x) {
        int atom = pmeAtomGridIndex[i].x;
        real4 pos = posq[atom];
#ifdef CHARGE_FROM_SIGEPS
        const float2 sigEps = sigmaEpsilon[atom];
        const real charge = 8*sigEps.x*sigEps.x*sigEps.x*sigEps.y;
#else
        const real charge = (CHARGE)*EPSILON_FACTOR;
#endif
        if (charge == 0)
            continue;
        APPLY_PERIODIC_TO_POS(pos)
        real3 t = make_real3(pos.x*recipBoxVecX.x+pos.y*recipBoxVecY.x+pos.z*recipBoxVecZ.x,
                             pos.y*recipBoxVecY.y+pos.z*recipBoxVecZ.y,
                             pos.z*recipBoxVecZ.z);
        t.x = (t.x-floor(t.x))*GRID_SIZE_X;
        t.y = (t.y-floor(t.y))*GRID_SIZE_Y;
        t.z = (t.z-floor(t.z))*GRID_SIZE_Z;
        int3 gridIndex = make_int3(((int) t.x) % GRID_SIZE_X,
                                   ((int) t.y) % GRID_SIZE_Y,
                                   ((int) t.z) % GRID_SIZE_Z);

        // Since we need the full set of thetas, it's faster to compute them here than load them
        // from global memory.
        
        real3 dr = make_real3(t.x-(int) t.x, t.y-(int) t.y, t.z-(int) t.z);
        data[PME_ORDER-1] = make_real3(0);
        data[1] = dr;
        data[0] = make_real3(1)-dr;
        for (int j = 3; j < PME_ORDER; j++) {
            real div = RECIP(j-1);
            data[j-1] = div*dr*data[j-2];
            for (int k = 1; k < (j-1); k++)
                data[j-k-1] = div*((dr+make_real3(k))*data[j-k-2] + (make_real3(j-k)-dr)*data[j-k-1]);
            data[0] = div*(make_real3(1)-dr)*data[0];
        }
        data[PME_ORDER-1] = scale*dr*data[PME_ORDER-2];
        for (int j = 1; j < (PME_ORDER-1); j++)
            data[PME_ORDER-j-1] = scale*((dr+make_real3(j))*data[PME_ORDER-j-2] + (make_real3(PME_ORDER-j)-dr)*data[PME_ORDER-j-1]);
        data[0] = scale*(make_real3(1)-dr)*data[0];
        
        // Spread the charge from this atom onto each grid point.

	int izoffset = (PME_ORDER-(gridIndex.z%PME_ORDER)) % PME_ORDER;
        for (int ix = 0; ix < PME_ORDER; ix++) {
            int xbase = gridIndex.x+ix;
            xbase -= (xbase >= GRID_SIZE_X ? GRID_SIZE_X : 0);
            xbase = xbase*GRID_SIZE_Y;
            real dx = charge*data[ix].x;
            for (int iy = 0; iy < PME_ORDER; iy++) {
                int ybase = gridIndex.y+iy;
                ybase -= (ybase >= GRID_SIZE_Y ? GRID_SIZE_Y : 0);
                ybase = (xbase+ybase)*blockSize;
                real dxdy = dx*data[iy].y;
                for (int i = 0; i < PME_ORDER; i++) {
		    int iz = (i+izoffset) % PME_ORDER;
                    int zindex = gridIndex.z+iz;
                    int index = ybase + zindexTable[zindex];
                    real add = dxdy*data[iz].z;
#if defined(USE_DOUBLE_PRECISION) || defined(USE_DETERMINISTIC_FORCES)
                    unsigned long long * ulonglong_p = (unsigned long long *) originalPmeGrid;
                    atomicAdd(&ulonglong_p[index],  static_cast<unsigned long long>((long long) (add*0x100000000)));
#else
                    atomicAdd(&originalPmeGrid[index], add);
#endif
                }
            }
        }
    }
}

extern "C" __global__ void finishSpreadCharge(
#if defined(USE_DOUBLE_PRECISION) || defined(USE_DETERMINISTIC_FORCES)
        const long long* __restrict__ grid1,
#else
        const real* __restrict__ grid1,
#endif
        real* __restrict__ grid2) {
    // During charge spreading, we shuffled the order of indices along the z
    // axis to make memory access more efficient.  We now need to unshuffle
    // them.  If the values were accumulated as fixed point, we also need to
    // convert them to floating point.

    __shared__ int zindexTable[GRID_SIZE_Z];
    int blockSize = (int) ceil(GRID_SIZE_Z/(real) PME_ORDER);
    for (int i = threadIdx.x; i < GRID_SIZE_Z; i += blockDim.x) {
	int block = i % PME_ORDER;
        zindexTable[i] = i/PME_ORDER + block*GRID_SIZE_X*GRID_SIZE_Y*blockSize;
    }
    __syncthreads();
    const unsigned int gridSize = GRID_SIZE_X*GRID_SIZE_Y*GRID_SIZE_Z;
    real scale = 1/(real) 0x100000000;
    for (int index = blockIdx.x*blockDim.x+threadIdx.x; index < gridSize; index += blockDim.x*gridDim.x) {
        int zindex = index%GRID_SIZE_Z;
        int loadIndex = zindexTable[zindex] + blockSize*(int) (index/GRID_SIZE_Z);
#if defined(USE_DOUBLE_PRECISION) || defined(USE_DETERMINISTIC_FORCES)
        grid2[index] = scale*grid1[loadIndex];
#else
        grid2[index] = grid1[loadIndex];
#endif
    }
}

// convolutes on the halfcomplex_pmeGrid, which is of size NX*NY*(NZ/2+1) as F(Q) is conjugate symmetric
extern "C" __global__ void 
reciprocalConvolution(real2* __restrict__ halfcomplex_pmeGrid, mixed* __restrict__ energyBuffer, 
                      const real* __restrict__ pmeBsplineModuliX, const real* __restrict__ pmeBsplineModuliY, const real* __restrict__ pmeBsplineModuliZ, 
                      real4 periodicBoxSize, real3 recipBoxVecX, real3 recipBoxVecY, real3 recipBoxVecZ) {
    // R2C stores into a half complex matrix where the last dimension is cut by half
    const unsigned int gridSize = GRID_SIZE_X*GRID_SIZE_Y*(GRID_SIZE_Z/2+1);
#ifdef USE_LJPME
    const real recipScaleFactor = -2*M_PI*SQRT(M_PI)*RECIP(6*periodicBoxSize.x*periodicBoxSize.y*periodicBoxSize.z);
    real bfac = M_PI / EWALD_ALPHA;
    real fac1 = 2*M_PI*M_PI*M_PI*SQRT(M_PI);
    real fac2 = EWALD_ALPHA*EWALD_ALPHA*EWALD_ALPHA;
    real fac3 = -2*EWALD_ALPHA*M_PI*M_PI;
#else
    const real recipScaleFactor = RECIP(M_PI*periodicBoxSize.x*periodicBoxSize.y*periodicBoxSize.z);
#endif

    for (int index = blockIdx.x*blockDim.x+threadIdx.x; index < gridSize; index += blockDim.x*gridDim.x) {
        // real indices
        int kx = index/(GRID_SIZE_Y*(GRID_SIZE_Z/2+1));
        int remainder = index-kx*GRID_SIZE_Y*(GRID_SIZE_Z/2+1);
        int ky = remainder/(GRID_SIZE_Z/2+1);
        int kz = remainder-ky*(GRID_SIZE_Z/2+1);
        int mx = (kx < (GRID_SIZE_X+1)/2) ? kx : (kx-GRID_SIZE_X);
        int my = (ky < (GRID_SIZE_Y+1)/2) ? ky : (ky-GRID_SIZE_Y);
        int mz = (kz < (GRID_SIZE_Z+1)/2) ? kz : (kz-GRID_SIZE_Z);
        real mhx = mx*recipBoxVecX.x;
        real mhy = mx*recipBoxVecY.x+my*recipBoxVecY.y;
        real mhz = mx*recipBoxVecZ.x+my*recipBoxVecZ.y+mz*recipBoxVecZ.z;
        real bx = pmeBsplineModuliX[kx];
        real by = pmeBsplineModuliY[ky];
        real bz = pmeBsplineModuliZ[kz];
        real2 grid = halfcomplex_pmeGrid[index];
        real m2 = mhx*mhx+mhy*mhy+mhz*mhz;
#ifdef USE_LJPME
        real denom = recipScaleFactor/(bx*by*bz);
        real m = SQRT(m2);
        real m3 = m*m2;
        real b = bfac*m;
        real expfac = -b*b;
        real expterm = EXP(expfac);
        real erfcterm = ERFC(b);
        real eterm = (fac1*erfcterm*m3 + expterm*(fac2 + fac3*m2)) * denom;
        halfcomplex_pmeGrid[index] = make_real2(grid.x*eterm, grid.y*eterm);
#else
        real denom = m2*bx*by*bz;
        real eterm = recipScaleFactor*EXP(-RECIP_EXP_FACTOR*m2)/denom;
        if (kx != 0 || ky != 0 || kz != 0) {
            halfcomplex_pmeGrid[index] = make_real2(grid.x*eterm, grid.y*eterm);
        }
#endif
    }
}


extern "C" __global__ void
gridEvaluateEnergy(real2* __restrict__ halfcomplex_pmeGrid, mixed* __restrict__ energyBuffer,
                      const real* __restrict__ pmeBsplineModuliX, const real* __restrict__ pmeBsplineModuliY, const real* __restrict__ pmeBsplineModuliZ,
                      real4 periodicBoxSize, real3 recipBoxVecX, real3 recipBoxVecY, real3 recipBoxVecZ) {
    // R2C stores into a half complex matrix where the last dimension is cut by half
    const unsigned int gridSize = GRID_SIZE_X*GRID_SIZE_Y*GRID_SIZE_Z;
 #ifdef USE_LJPME
    const real recipScaleFactor = -2*M_PI*SQRT(M_PI)*RECIP(6*periodicBoxSize.x*periodicBoxSize.y*periodicBoxSize.z);
    real bfac = M_PI / EWALD_ALPHA;
    real fac1 = 2*M_PI*M_PI*M_PI*SQRT(M_PI);
    real fac2 = EWALD_ALPHA*EWALD_ALPHA*EWALD_ALPHA;
    real fac3 = -2*EWALD_ALPHA*M_PI*M_PI;
#else
    const real recipScaleFactor = RECIP(M_PI*periodicBoxSize.x*periodicBoxSize.y*periodicBoxSize.z);
#endif

    mixed energy = 0;
    for (int index = blockIdx.x*blockDim.x+threadIdx.x; index < gridSize; index += blockDim.x*gridDim.x) {
        // real indices
        int kx = index/(GRID_SIZE_Y*(GRID_SIZE_Z));
        int remainder = index-kx*GRID_SIZE_Y*(GRID_SIZE_Z);
        int ky = remainder/(GRID_SIZE_Z);
        int kz = remainder-ky*(GRID_SIZE_Z);
        int mx = (kx < (GRID_SIZE_X+1)/2) ? kx : (kx-GRID_SIZE_X);
        int my = (ky < (GRID_SIZE_Y+1)/2) ? ky : (ky-GRID_SIZE_Y);
        int mz = (kz < (GRID_SIZE_Z+1)/2) ? kz : (kz-GRID_SIZE_Z);
        real mhx = mx*recipBoxVecX.x;
        real mhy = mx*recipBoxVecY.x+my*recipBoxVecY.y;
        real mhz = mx*recipBoxVecZ.x+my*recipBoxVecZ.y+mz*recipBoxVecZ.z;
        real m2 = mhx*mhx+mhy*mhy+mhz*mhz;
        real bx = pmeBsplineModuliX[kx];
        real by = pmeBsplineModuliY[ky];
        real bz = pmeBsplineModuliZ[kz];
#ifdef USE_LJPME
        real denom = recipScaleFactor/(bx*by*bz);
        real m = SQRT(m2);
        real m3 = m*m2;
        real b = bfac*m;
        real expfac = -b*b;
        real expterm = EXP(expfac);
        real erfcterm = ERFC(b);
        real eterm = (fac1*erfcterm*m3 + expterm*(fac2 + fac3*m2)) * denom;
#else
        real denom = m2*bx*by*bz;
        real eterm = recipScaleFactor*EXP(-RECIP_EXP_FACTOR*m2)/denom;
#endif

        if (kz >= (GRID_SIZE_Z/2+1)) {
            kx = ((kx == 0) ? kx : GRID_SIZE_X-kx);
            ky = ((ky == 0) ? ky : GRID_SIZE_Y-ky);
            kz = GRID_SIZE_Z-kz;
        } 
        int indexInHalfComplexGrid = kz + ky*(GRID_SIZE_Z/2+1)+kx*(GRID_SIZE_Y*(GRID_SIZE_Z/2+1));
        real2 grid = halfcomplex_pmeGrid[indexInHalfComplexGrid];
#ifndef USE_LJPME
        if (kx != 0 || ky != 0 || kz != 0)
#endif
            energy += eterm*(grid.x*grid.x + grid.y*grid.y);
    }
#if defined(USE_PME_STREAM) && !defined(USE_LJPME)
    energyBuffer[blockIdx.x*blockDim.x+threadIdx.x] = 0.5f*energy;
#else
    energyBuffer[blockIdx.x*blockDim.x+threadIdx.x] += 0.5f*energy;
#endif
}

extern "C" __global__
void gridInterpolateForce(const real4* __restrict__ posq, unsigned long long* __restrict__ forceBuffers, const real* __restrict__ originalPmeGrid,
        real4 periodicBoxSize, real4 invPeriodicBoxSize, real4 periodicBoxVecX, real4 periodicBoxVecY, real4 periodicBoxVecZ,
        real3 recipBoxVecX, real3 recipBoxVecY, real3 recipBoxVecZ, const int2* __restrict__ pmeAtomGridIndex
#ifdef CHARGE_FROM_SIGEPS
        , const float2* __restrict__ sigmaEpsilon
#else
        , const real* __restrict__ charges
#endif
        ) {
    real3 data[PME_ORDER];
    real3 ddata[PME_ORDER];
    const real scale = RECIP(PME_ORDER-1);
    
    // Process the atoms in spatially sorted order.  This improves cache performance when loading
    // the grid values.
    
    for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < NUM_ATOMS; i += blockDim.x*gridDim.x) {
        int atom = pmeAtomGridIndex[i].x;
        real3 force = make_real3(0);
        real4 pos = posq[atom];
        APPLY_PERIODIC_TO_POS(pos)
        real3 t = make_real3(pos.x*recipBoxVecX.x+pos.y*recipBoxVecY.x+pos.z*recipBoxVecZ.x,
                             pos.y*recipBoxVecY.y+pos.z*recipBoxVecZ.y,
                             pos.z*recipBoxVecZ.z);
        t.x = (t.x-floor(t.x))*GRID_SIZE_X;
        t.y = (t.y-floor(t.y))*GRID_SIZE_Y;
        t.z = (t.z-floor(t.z))*GRID_SIZE_Z;
        int3 gridIndex = make_int3(((int) t.x) % GRID_SIZE_X,
                                   ((int) t.y) % GRID_SIZE_Y,
                                   ((int) t.z) % GRID_SIZE_Z);

        // Since we need the full set of thetas, it's faster to compute them here than load them
        // from global memory.
        
        real3 dr = make_real3(t.x-(int) t.x, t.y-(int) t.y, t.z-(int) t.z);
        data[PME_ORDER-1] = make_real3(0);
        data[1] = dr;
        data[0] = make_real3(1)-dr;
        for (int j = 3; j < PME_ORDER; j++) {
            real div = RECIP(j-1);
            data[j-1] = div*dr*data[j-2];
            for (int k = 1; k < (j-1); k++)
                data[j-k-1] = div*((dr+make_real3(k))*data[j-k-2] + (make_real3(j-k)-dr)*data[j-k-1]);
            data[0] = div*(make_real3(1)-dr)*data[0];
        }
        ddata[0] = -data[0];
        for (int j = 1; j < PME_ORDER; j++)
            ddata[j] = data[j-1]-data[j];
        data[PME_ORDER-1] = scale*dr*data[PME_ORDER-2];
        for (int j = 1; j < (PME_ORDER-1); j++)
            data[PME_ORDER-j-1] = scale*((dr+make_real3(j))*data[PME_ORDER-j-2] + (make_real3(PME_ORDER-j)-dr)*data[PME_ORDER-j-1]);
        data[0] = scale*(make_real3(1)-dr)*data[0];
        
        // Compute the force on this atom.
         
        for (int ix = 0; ix < PME_ORDER; ix++) {
            int xbase = gridIndex.x+ix;
            xbase -= (xbase >= GRID_SIZE_X ? GRID_SIZE_X : 0);
            xbase = xbase*GRID_SIZE_Y*GRID_SIZE_Z;
            real dx = data[ix].x;
            real ddx = ddata[ix].x;
            
            for (int iy = 0; iy < PME_ORDER; iy++) {
                int ybase = gridIndex.y+iy;
                ybase -= (ybase >= GRID_SIZE_Y ? GRID_SIZE_Y : 0);
                ybase = xbase + ybase*GRID_SIZE_Z;
                real dy = data[iy].y;
                real ddy = ddata[iy].y;
                
                for (int iz = 0; iz < PME_ORDER; iz++) {
                    int zindex = gridIndex.z+iz;
                    zindex -= (zindex >= GRID_SIZE_Z ? GRID_SIZE_Z : 0);
                    int index = ybase + zindex;
                    real gridvalue = originalPmeGrid[index];
                    force.x += ddx*dy*data[iz].z*gridvalue;
                    force.y += dx*ddy*data[iz].z*gridvalue;
                    force.z += dx*dy*ddata[iz].z*gridvalue;
                }
            }
        }
#ifdef CHARGE_FROM_SIGEPS
        const float2 sigEps = sigmaEpsilon[atom];
        real q = 8*sigEps.x*sigEps.x*sigEps.x*sigEps.y;
#else
        real q = CHARGE*EPSILON_FACTOR;
#endif
        real forceX = -q*(force.x*GRID_SIZE_X*recipBoxVecX.x);
        real forceY = -q*(force.x*GRID_SIZE_X*recipBoxVecY.x+force.y*GRID_SIZE_Y*recipBoxVecY.y);
        real forceZ = -q*(force.x*GRID_SIZE_X*recipBoxVecZ.x+force.y*GRID_SIZE_Y*recipBoxVecZ.y+force.z*GRID_SIZE_Z*recipBoxVecZ.z);
        atomicAdd(&forceBuffers[atom], static_cast<unsigned long long>((long long) (forceX*0x100000000)));
        atomicAdd(&forceBuffers[atom+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (forceY*0x100000000)));
        atomicAdd(&forceBuffers[atom+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (forceZ*0x100000000)));
    }
}

extern "C" __global__
void addForces(const real4* __restrict__ forces, unsigned long long* __restrict__ forceBuffers) {
    for (int atom = blockIdx.x*blockDim.x+threadIdx.x; atom < NUM_ATOMS; atom += blockDim.x*gridDim.x) {
        real4 f = forces[atom];
        forceBuffers[atom] += static_cast<unsigned long long>((long long) (f.x*0x100000000));
        forceBuffers[atom+PADDED_NUM_ATOMS] += static_cast<unsigned long long>((long long) (f.y*0x100000000));
        forceBuffers[atom+2*PADDED_NUM_ATOMS] += static_cast<unsigned long long>((long long) (f.z*0x100000000));
    }
}

extern "C" __global__
void addEnergy(const mixed* __restrict__ pmeEnergyBuffer, mixed* __restrict__ energyBuffer, int bufferSize) {
    for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < bufferSize; i += blockDim.x*gridDim.x)
        energyBuffer[i] += pmeEnergyBuffer[i];
}
